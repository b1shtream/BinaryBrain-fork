#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"




//////////////////////////////
// forward
//////////////////////////////

__global__ void kernal_fp32_MaxPooling_Forward(
			float const *x_buf,
			float       *y_buf,
   	        int			filter_h_size,
	        int 		filter_w_size,
            int         input_w_size,
            int         input_h_size,
            int         output_w_size,
            int         output_h_size,
            int         c_size,
			int         frame_size,
			int         frame_stride
		)
{
	int frame_base = threadIdx.x;
	int frame_step = blockDim.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.x;
    int c = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (y >= output_h_size || x >= output_w_size) {
        return;
    }

    for ( int frame = frame_base; frame < frame_size; frame += frame_step ) {
        // �ő�l�T��
        float max_val = -1.0e7f;
        for (int fy = 0; fy < filter_h_size; ++fy) {
            int iy = y * filter_h_size + fy;
            if ( iy < input_h_size ) {
                for (int fx = 0; fx < filter_w_size; ++fx) {
                    int ix = x * filter_w_size + fx;
                    if ( ix < input_w_size ) {
                        float sig = x_buf[((c * input_h_size + iy) * input_w_size + ix) * frame_stride + frame];
                        max_val = max(max_val, sig);
                    }
                }
            }
        }

        // �o��
        y_buf[((c * output_h_size + y) * output_w_size + x) * frame_stride + frame] = max_val;
    }
}


BBCU_DLL_EXPORT int bbcu_fp32_MaxPooling_Forward
		(
			float const *	dev_x_buf,
			float*			dev_y_buf,
   	        int			    filter_h_size,
	        int 		    filter_w_size,
            int             input_w_size,
            int             input_h_size,
            int             output_w_size,
            int             output_h_size,
            int             c_size,
			int             frame_size,
			int             frame_stride,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

	dim3	block(32, 32, 1);
	dim3	grid;
    grid.x = output_h_size;
    grid.y = (output_w_size + (block.y-1)) / block.y;
    grid.z = c_size;
	block.x = max(block.x, frame_size);
	block.y = max(block.y, output_w_size);

	kernal_fp32_MaxPooling_Forward<<<grid, block, 0, streamId>>>(
			dev_x_buf,
            dev_y_buf,
   	        filter_h_size,
	        filter_w_size,
            input_w_size,
            input_h_size,
            output_w_size,
            output_h_size,
            c_size,
			frame_size,
			frame_stride
		);
	BB_CUDA_CHECK_LAST_ERROR();

	return 0;
}



//////////////////////////////
// backward
//////////////////////////////

__global__ void kernal_fp32_MaxPooling_Backward(
			float const *x_buf,
			float const *y_buf,
			float const *dy_buf,
			float       *dx_buf,
   	        int			filter_h_size,
	        int 		filter_w_size,
            int         input_w_size,
            int         input_h_size,
            int         output_w_size,
            int         output_h_size,
            int         c_size,
			int         frame_size,
			int         frame_stride
		)
{
	int frame_base = threadIdx.x;
	int frame_step = blockDim.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.x;
    int c = blockIdx.z * blockDim.z + threadIdx.z;

    if (y >= output_h_size || x >= output_w_size) {
        return;
    }
    
    // �ő�l�ӏ��̂ݓ`�d
    for ( int frame = frame_base; frame < frame_size; frame += frame_step ) {
        float out_sig = y_buf[((c * output_h_size + y) * output_w_size + x) * frame_stride + frame];
        float grad    = dy_buf[((c * output_h_size + y) * output_w_size + x) * frame_stride + frame];
        for (int fy = 0; fy < filter_h_size; ++fy) {
            int iy = y * filter_h_size + fy;
            if ( iy < input_h_size ) {
                for (int fx = 0; fx < filter_w_size; ++fx) {
                    int ix = x * filter_w_size + fx;
                    if ( ix < input_w_size ) {
                        float in_sig  = x_buf[((c * input_h_size + iy) * input_w_size + ix) * frame_stride + frame];
                        dx_buf[((c * input_h_size + iy) * input_w_size + ix) * frame_stride + frame] = (in_sig == out_sig) ? grad : 0;
                    }
                }
            }
        }
    }
}


BBCU_DLL_EXPORT int bbcu_fp32_MaxPooling_Backward
		(
			float const     *dev_x_buf,
			float const     *dev_y_buf,
			float const     *dev_dy_buf,
			float           *dev_dx_buf,
            int			    filter_h_size,
	        int 		    filter_w_size,
            int             input_w_size,
            int             input_h_size,
            int             output_w_size,
            int             output_h_size,
            int             c_size,
			int             frame_size,
			int             frame_stride,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

	dim3	block(32, 32, 1);
	dim3	grid;
    grid.x = output_h_size;
    grid.y = (output_w_size + (block.y-1)) / block.y;
    grid.z = c_size;
	block.x = max(block.x, frame_size);
	block.y = max(block.y, output_w_size);

	kernal_fp32_MaxPooling_Backward<<<grid, block, 0, streamId>>>(
			dev_x_buf,
            dev_y_buf,
			dev_dy_buf,
			dev_dx_buf,
   	        filter_h_size,
	        filter_w_size,
            input_w_size,
            input_h_size,
            output_w_size,
            output_h_size,
            c_size,
			frame_size,
			frame_stride
		);
	BB_CUDA_CHECK_LAST_ERROR();

	return 0;
}

