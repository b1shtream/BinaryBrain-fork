#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"


#include "Common.cuh"
#include "StochasticLut.cuh"


//#define BINARY_BIAS     (0.125/2)
//#define BINARY_BIAS     0.125
#define BINARY_BIAS     0.2
#define BINARY_ZERO     (0.5 - BINARY_BIAS)
#define BINARY_ONE      (0.5 + BINARY_BIAS)

//#define BINARY_ZERO     0.0
//#define BINARY_ONE      1.0


// -------------------------------------------------
//  Forward
// -------------------------------------------------

#if 0
template<int MAX_NODE_UNIT=32>
__device__ float device_fp32_SparseBinaryLut6_NodeForward
        (
            int             node_id,
            float           xp[6],
            float   const   W[64][MAX_NODE_UNIT]
        )
{
    float   xn[6];
    for ( int i = 0; i < 6; ++i) {
        xn[i] = 1.0 - xp[i];
    }

    float x0_00 = xn[1] * xn[0];
    float x0_01 = xn[1] * xp[0];
    float x0_10 = xp[1] * xn[0];
    float x0_11 = xp[1] * xp[0];
    float x1_00 = xn[3] * xn[2];
    float x1_01 = xn[3] * xp[2];
    float x1_10 = xp[3] * xn[2];
    float x1_11 = xp[3] * xp[2];
    float x2_00 = xn[5] * xn[4];
    float x2_01 = xn[5] * xp[4];
    float x2_10 = xp[5] * xn[4];
    float x2_11 = xp[5] * xp[4];

    float y = 0;
    float x2_00_x1_00 = x2_00 * x1_00;
    y += W[0 ][node_id] * x2_00_x1_00 * x0_00;
    y += W[1 ][node_id] * x2_00_x1_00 * x0_01;
    y += W[2 ][node_id] * x2_00_x1_00 * x0_10;
    y += W[3 ][node_id] * x2_00_x1_00 * x0_11;
    float x2_00_x1_01 = x2_00 * x1_01;
    y += W[4 ][node_id] * x2_00_x1_01 * x0_00;
    y += W[5 ][node_id] * x2_00_x1_01 * x0_01;
    y += W[6 ][node_id] * x2_00_x1_01 * x0_10;
    y += W[7 ][node_id] * x2_00_x1_01 * x0_11;
    float x2_00_x1_10 = x2_00 * x1_10;
    y += W[8 ][node_id] * x2_00_x1_10 * x0_00;
    y += W[9 ][node_id] * x2_00_x1_10 * x0_01;
    y += W[10][node_id] * x2_00_x1_10 * x0_10;
    y += W[11][node_id] * x2_00_x1_10 * x0_11;
    float x2_00_x1_11 = x2_00 * x1_11;
    y += W[12][node_id] * x2_00_x1_11 * x0_00;
    y += W[13][node_id] * x2_00_x1_11 * x0_01;
    y += W[14][node_id] * x2_00_x1_11 * x0_10;
    y += W[15][node_id] * x2_00_x1_11 * x0_11;
    float x2_01_x1_00 = x2_01 * x1_00;
    y += W[16][node_id] * x2_01_x1_00 * x0_00;
    y += W[17][node_id] * x2_01_x1_00 * x0_01;
    y += W[18][node_id] * x2_01_x1_00 * x0_10;
    y += W[19][node_id] * x2_01_x1_00 * x0_11;
    float x2_01_x1_01 = x2_01 * x1_01;
    y += W[20][node_id] * x2_01_x1_01 * x0_00;
    y += W[21][node_id] * x2_01_x1_01 * x0_01;
    y += W[22][node_id] * x2_01_x1_01 * x0_10;
    y += W[23][node_id] * x2_01_x1_01 * x0_11;
    float x2_01_x1_10 = x2_01 * x1_10;
    y += W[24][node_id] * x2_01_x1_10 * x0_00;
    y += W[25][node_id] * x2_01_x1_10 * x0_01;
    y += W[26][node_id] * x2_01_x1_10 * x0_10;
    y += W[27][node_id] * x2_01_x1_10 * x0_11;
    float x2_01_x1_11 = x2_01 * x1_11;
    y += W[28][node_id] * x2_01_x1_11 * x0_00;
    y += W[29][node_id] * x2_01_x1_11 * x0_01;
    y += W[30][node_id] * x2_01_x1_11 * x0_10;
    y += W[31][node_id] * x2_01_x1_11 * x0_11;
    float x2_10_x1_00 = x2_10 * x1_00;
    y += W[32][node_id] * x2_10_x1_00 * x0_00;
    y += W[33][node_id] * x2_10_x1_00 * x0_01;
    y += W[34][node_id] * x2_10_x1_00 * x0_10;
    y += W[35][node_id] * x2_10_x1_00 * x0_11;
    float x2_10_x1_01 = x2_10 * x1_01;
    y += W[36][node_id] * x2_10_x1_01 * x0_00;
    y += W[37][node_id] * x2_10_x1_01 * x0_01;
    y += W[38][node_id] * x2_10_x1_01 * x0_10;
    y += W[39][node_id] * x2_10_x1_01 * x0_11;
    float x2_10_x1_10 = x2_10 * x1_10;
    y += W[40][node_id] * x2_10_x1_10 * x0_00;
    y += W[41][node_id] * x2_10_x1_10 * x0_01;
    y += W[42][node_id] * x2_10_x1_10 * x0_10;
    y += W[43][node_id] * x2_10_x1_10 * x0_11;
    float x2_10_x1_11 = x2_10 * x1_11;
    y += W[44][node_id] * x2_10_x1_11 * x0_00;
    y += W[45][node_id] * x2_10_x1_11 * x0_01;
    y += W[46][node_id] * x2_10_x1_11 * x0_10;
    y += W[47][node_id] * x2_10_x1_11 * x0_11;
    float x2_11_x1_00 = x2_11 * x1_00;
    y += W[48][node_id] * x2_11_x1_00 * x0_00;
    y += W[49][node_id] * x2_11_x1_00 * x0_01;
    y += W[50][node_id] * x2_11_x1_00 * x0_10;
    y += W[51][node_id] * x2_11_x1_00 * x0_11;
    float x2_11_x1_01 = x2_11 * x1_01;
    y += W[52][node_id] * x2_11_x1_01 * x0_00;
    y += W[53][node_id] * x2_11_x1_01 * x0_01;
    y += W[54][node_id] * x2_11_x1_01 * x0_10;
    y += W[55][node_id] * x2_11_x1_01 * x0_11;
    float x2_11_x1_10 = x2_11 * x1_10;
    y += W[56][node_id] * x2_11_x1_10 * x0_00;
    y += W[57][node_id] * x2_11_x1_10 * x0_01;
    y += W[58][node_id] * x2_11_x1_10 * x0_10;
    y += W[59][node_id] * x2_11_x1_10 * x0_11;
    float x2_11_x1_11 = x2_11 * x1_11;
    y += W[60][node_id] * x2_11_x1_11 * x0_00;
    y += W[61][node_id] * x2_11_x1_11 * x0_01;
    y += W[62][node_id] * x2_11_x1_11 * x0_10;
    y += W[63][node_id] * x2_11_x1_11 * x0_11;

    // clamp
    y = max(0.0, y);
    y = min(1.0, y);

    return y;
}
#endif



template<int MAX_FRAME_UNIT=32, int MAX_NODE_UNIT=32>
__global__ void kernal_bit_fp32_SparseBinaryLut6_ForwardTraining
        (
            int   const     *x_buf,
            int             *y_buf,
            int   const     *input_index,
            float const     *W_buf,
            float           *mean_buf,
            float           *rstd_buf,
            float           *running_mean_buf,
            float           *running_var_buf,
            float           gamma,
            float           beta,
            float           momentum,
            float           reciprocal_frame_size,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             lut_binarize
        )
{
//  int unit_id = ((threadIdx.x % MAX_FRAME_UNIT) & ~0x1f);
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];

    __shared__  float       W[64][MAX_NODE_UNIT];
                int   const *x_ptr[6];
                int         *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < 6; ++i ) {
            x_ptr[i] = &x_buf[frame_stride * input_index[6*node + i]];
        }
                     
        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    // ���ςƕ��U�v��
    float s1 = 0, c1 = 0, y1, t1;
    float s2 = 0, c2 = 0, y2, t2;
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            // Forward�v�Z
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);
            float x[6];
            for ( int i = 0; i < 6; ++i) {
                x[i] = (x_ptr[i][unit] & bit) ? BINARY_ONE : BINARY_ZERO;
            }
//          float y = device_fp32_SparseBinaryLut6_NodeForward<MAX_NODE_UNIT>(node_id, x, W);
            float y = StochasticLut<6, float, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

//          printf("[SparseBinaryLut6] node=%d frame=%d lut_y=%f\n", node, frame, y);

            // �W�v
            y1 = y - c1;
            t1 = s1 + y1;
            c1 = (t1 - s1) - y1;
            s1 = t1;

            y2 = (y * y) - c2;
            t2 = s2 + y2;
            c2 = (t2 - s2) - y2;
            s2 = t2;
        }
    }

    s1 = device_fp32_LocalSum(s1, sbuf[node_id]);
    s2 = device_fp32_LocalSum(s2, sbuf[node_id]);
    float mean = s1 * reciprocal_frame_size;
    float var = max(1.0e-7f, (s2 * reciprocal_frame_size) - (mean * mean));
  
    float rstd = rsqrt(var);

    // ��������
    if (id == 0) {
        if ( node < node_size ) {
            running_mean_buf[node] = running_mean_buf[node] * momentum + mean * (1.0f - momentum);
            running_var_buf[node]  = running_var_buf[node] * momentum + var * (1.0f - momentum);
            mean_buf[node] = mean;
            rstd_buf[node] = rstd;
//          printf("[SparseBinaryLut6] node=%d mean=%f rstd=%f\n", node, mean, rstd);
        }
    }

    // ���K��
    int loop_size = ((frame_size + blockDim.x - 1) & ~(blockDim.x - 1));
    for ( int frame = id; frame < loop_size; frame += id_step) {
        int unit     = (frame >> 5);
        int bit      = (frame & 0x1f);
        int bit_mask = (1 << bit);

        int y_mask = 0;
        if ( node < node_size && frame < frame_size) {
            // Forward�v�Z
            float x[6];
            for ( int i = 0; i < 6; ++i) {
                x[i] = (x_ptr[i][unit] & bit_mask) ? BINARY_ONE : BINARY_ZERO;
            }
//          float y = device_fp32_SparseBinaryLut6_NodeForward<MAX_NODE_UNIT>(node_id, x, W);
            float y = StochasticLut<6, float, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

            y = (y - mean) * rstd;
            y = y * gamma + beta;
//          printf("[SparseBinaryLut6] frame=%d node=%d y=%f\n", frame, node, y);

            if ( y > 0.5 ) {
                y_mask = bit_mask;
            }
        }

//      y_mask = device_int_LocalOr(y_mask, bit, (int *)&sbuf[node_id][unit_id]);
        y_mask = device_int_ShuffleOr(y_mask);

        if ( bit == 0 ) {
            if ( node < node_size && frame < frame_size ) {
                y_ptr[unit] = y_mask;
            }
        }
    }
}


BBCU_DLL_EXPORT int bbcu_bit_fp32_SparseBinaryLut6_ForwardTraining
        (
            int   const     *dev_x_buf,
            int             *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            float           *mean_buf,
            float           *rstd_buf,
            float           *running_mean_buf,
            float           *running_var_buf,
            float           gamma,
            float           beta,
            float           momentum,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             lut_binarize,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 256;
    unsigned int const MAX_FRAME_UNIT = 256;
    unsigned int const MAX_NODE_UNIT  = 8;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size && block.x > 32 ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size                  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  )                { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size && block.x > 32) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_bit_fp32_SparseBinaryLut6_ForwardTraining<MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            mean_buf,
            rstd_buf,
            running_mean_buf,
            running_var_buf,
            gamma,
            beta,
            momentum,
            1.0f / (float)frame_size,
            node_size,
            frame_size,
            frame_stride,
            lut_binarize
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}


// -------------------------------------------------
//  Forward Inference
// -------------------------------------------------


template<int MAX_FRAME_UNIT=32, int MAX_NODE_UNIT=32>
__global__ void kernal_bit_fp32_SparseBinaryLut6_ForwardInference
        (
            int   const     *x_buf,
            int             *y_buf,
            int   const     *input_index,
            float const     *W_buf,
            float const     *running_mean_buf,
            float const     *running_var_buf,
            float           gamma,
            float           beta,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             lut_binarize
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       W[64][MAX_NODE_UNIT];
                int   const *x_ptr[6];
                int         *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < 6; ++i ) {
            x_ptr[i] = &x_buf[frame_stride * input_index[6*node + i]];
        }
                     
        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    if ( node < node_size ) {
        float mean  = running_mean_buf[node];
        float var   = running_var_buf[node];
        float rstd = 1.0 / (sqrt(var) + 1.0e-7);

        int loop_size = ((frame_size + blockDim.x - 1) & ~(blockDim.x - 1));
        for ( int frame = id; frame < loop_size; frame += id_step) {
            int unit     = (frame >> 5);
            int bit      = (frame & 0x1f);
            int bit_mask = (1 << bit);

            int y_mask = 0;
            if ( node < node_size && frame < frame_size) {
                // Forward�v�Z
                float x[6];
                for ( int i = 0; i < 6; ++i) {
                    x[i] = (x_ptr[i][unit] & bit_mask) ? BINARY_ONE : BINARY_ZERO;
                }
//              float y = device_fp32_SparseBinaryLut6_NodeForward<MAX_NODE_UNIT>(node_id, x, W);
                float y = StochasticLut<6, float, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

                y = ((y - mean) * rstd) * gamma + beta;

                if ( y > 0.5 ) {
                    y_mask = bit_mask;
                }
            }

            y_mask = device_int_ShuffleOr(y_mask);

            if ( bit == 0 ) {
                if ( node < node_size && frame < frame_size ) {
                    y_ptr[unit] = y_mask;
                }
            }
        }
    }
}


BBCU_DLL_EXPORT int bbcu_bit_fp32_SparseBinaryLut6_ForwardInference
        (
            int   const     *dev_x_buf,
            int             *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            float const     *running_mean_buf,
            float const     *running_var_buf,
            float           gamma,
            float           beta,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             lut_binarize,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 256;
    unsigned int const MAX_FRAME_UNIT = 256;
    unsigned int const MAX_NODE_UNIT  = 8;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size && block.x > 32 ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size                  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  )                { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size && block.x > 32) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_bit_fp32_SparseBinaryLut6_ForwardInference<MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            running_mean_buf,
            running_var_buf,
            gamma,
            beta,
            node_size,
            frame_size,
            frame_stride,
            lut_binarize
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



// -------------------------------------------------
//  Backward
// -------------------------------------------------

#if 0
template<int MAX_NODE_UNIT=32>
__device__ void device_fp32_SparseBinaryLut6_NodeBackward
        (
            int             node_id,
            float   const   xp[6],
            float           dy,
            float           *dx_ptr,
            float   const   W[64][MAX_NODE_UNIT],
            float           dW[64],
            int             frame_stride
        )
{
    float   xn[6];
    for (int i = 0; i < 6; ++i) {
        xn[i] = 1.0 - xp[i];
    }

    float x0_00 = xn[1] * xn[0];
    float x0_01 = xn[1] * xp[0];
    float x0_10 = xp[1] * xn[0];
    float x0_11 = xp[1] * xp[0];
    float x1_00 = xn[3] * xn[2];
    float x1_01 = xn[3] * xp[2];
    float x1_10 = xp[3] * xn[2];
    float x1_11 = xp[3] * xp[2];
    float x2_00 = xn[5] * xn[4];
    float x2_01 = xn[5] * xp[4];
    float x2_10 = xp[5] * xn[4];
    float x2_11 = xp[5] * xp[4];

    float  x2_00_x1_00 =  x2_00 * x1_00;
    float  x2_00_x1_01 =  x2_00 * x1_01;
    float  x2_00_x1_10 =  x2_00 * x1_10;
    float  x2_00_x1_11 =  x2_00 * x1_11;
    float  x2_01_x1_00 =  x2_01 * x1_00;
    float  x2_01_x1_01 =  x2_01 * x1_01;
    float  x2_01_x1_10 =  x2_01 * x1_10;
    float  x2_01_x1_11 =  x2_01 * x1_11;
    float  x2_10_x1_00 =  x2_10 * x1_00;
    float  x2_10_x1_01 =  x2_10 * x1_01;
    float  x2_10_x1_10 =  x2_10 * x1_10;
    float  x2_10_x1_11 =  x2_10 * x1_11;
    float  x2_11_x1_00 =  x2_11 * x1_00;
    float  x2_11_x1_01 =  x2_11 * x1_01;
    float  x2_11_x1_10 =  x2_11 * x1_10;
    float  x2_11_x1_11 =  x2_11 * x1_11;

    dW[ 0] += x2_00_x1_00 * x0_00 * dy;
    dW[ 1] += x2_00_x1_00 * x0_01 * dy;
    dW[ 2] += x2_00_x1_00 * x0_10 * dy;
    dW[ 3] += x2_00_x1_00 * x0_11 * dy;
    dW[ 4] += x2_00_x1_01 * x0_00 * dy;
    dW[ 5] += x2_00_x1_01 * x0_01 * dy;
    dW[ 6] += x2_00_x1_01 * x0_10 * dy;
    dW[ 7] += x2_00_x1_01 * x0_11 * dy;
    dW[ 8] += x2_00_x1_10 * x0_00 * dy;
    dW[ 9] += x2_00_x1_10 * x0_01 * dy;
    dW[10] += x2_00_x1_10 * x0_10 * dy;
    dW[11] += x2_00_x1_10 * x0_11 * dy;
    dW[12] += x2_00_x1_11 * x0_00 * dy;
    dW[13] += x2_00_x1_11 * x0_01 * dy;
    dW[14] += x2_00_x1_11 * x0_10 * dy;
    dW[15] += x2_00_x1_11 * x0_11 * dy;
    dW[16] += x2_01_x1_00 * x0_00 * dy;
    dW[17] += x2_01_x1_00 * x0_01 * dy;
    dW[18] += x2_01_x1_00 * x0_10 * dy;
    dW[19] += x2_01_x1_00 * x0_11 * dy;
    dW[20] += x2_01_x1_01 * x0_00 * dy;
    dW[21] += x2_01_x1_01 * x0_01 * dy;
    dW[22] += x2_01_x1_01 * x0_10 * dy;
    dW[23] += x2_01_x1_01 * x0_11 * dy;
    dW[24] += x2_01_x1_10 * x0_00 * dy;
    dW[25] += x2_01_x1_10 * x0_01 * dy;
    dW[26] += x2_01_x1_10 * x0_10 * dy;
    dW[27] += x2_01_x1_10 * x0_11 * dy;
    dW[28] += x2_01_x1_11 * x0_00 * dy;
    dW[29] += x2_01_x1_11 * x0_01 * dy;
    dW[30] += x2_01_x1_11 * x0_10 * dy;
    dW[31] += x2_01_x1_11 * x0_11 * dy;
    dW[32] += x2_10_x1_00 * x0_00 * dy;
    dW[33] += x2_10_x1_00 * x0_01 * dy;
    dW[34] += x2_10_x1_00 * x0_10 * dy;
    dW[35] += x2_10_x1_00 * x0_11 * dy;
    dW[36] += x2_10_x1_01 * x0_00 * dy;
    dW[37] += x2_10_x1_01 * x0_01 * dy;
    dW[38] += x2_10_x1_01 * x0_10 * dy;
    dW[39] += x2_10_x1_01 * x0_11 * dy;
    dW[40] += x2_10_x1_10 * x0_00 * dy;
    dW[41] += x2_10_x1_10 * x0_01 * dy;
    dW[42] += x2_10_x1_10 * x0_10 * dy;
    dW[43] += x2_10_x1_10 * x0_11 * dy;
    dW[44] += x2_10_x1_11 * x0_00 * dy;
    dW[45] += x2_10_x1_11 * x0_01 * dy;
    dW[46] += x2_10_x1_11 * x0_10 * dy;
    dW[47] += x2_10_x1_11 * x0_11 * dy;
    dW[48] += x2_11_x1_00 * x0_00 * dy;
    dW[49] += x2_11_x1_00 * x0_01 * dy;
    dW[50] += x2_11_x1_00 * x0_10 * dy;
    dW[51] += x2_11_x1_00 * x0_11 * dy;
    dW[52] += x2_11_x1_01 * x0_00 * dy;
    dW[53] += x2_11_x1_01 * x0_01 * dy;
    dW[54] += x2_11_x1_01 * x0_10 * dy;
    dW[55] += x2_11_x1_01 * x0_11 * dy;
    dW[56] += x2_11_x1_10 * x0_00 * dy;
    dW[57] += x2_11_x1_10 * x0_01 * dy;
    dW[58] += x2_11_x1_10 * x0_10 * dy;
    dW[59] += x2_11_x1_10 * x0_11 * dy;
    dW[60] += x2_11_x1_11 * x0_00 * dy;
    dW[61] += x2_11_x1_11 * x0_01 * dy;
    dW[62] += x2_11_x1_11 * x0_10 * dy;
    dW[63] += x2_11_x1_11 * x0_11 * dy;

    float  x2_00_x0_00 =  x2_00 * x0_00;
    float  x2_00_x0_01 =  x2_00 * x0_01;
    float  x2_00_x0_10 =  x2_00 * x0_10;
    float  x2_00_x0_11 =  x2_00 * x0_11;
    float  x2_01_x0_00 =  x2_01 * x0_00;
    float  x2_01_x0_01 =  x2_01 * x0_01;
    float  x2_01_x0_10 =  x2_01 * x0_10;
    float  x2_01_x0_11 =  x2_01 * x0_11;
    float  x2_10_x0_00 =  x2_10 * x0_00;
    float  x2_10_x0_01 =  x2_10 * x0_01;
    float  x2_10_x0_10 =  x2_10 * x0_10;
    float  x2_10_x0_11 =  x2_10 * x0_11;
    float  x2_11_x0_00 =  x2_11 * x0_00;
    float  x2_11_x0_01 =  x2_11 * x0_01;
    float  x2_11_x0_10 =  x2_11 * x0_10;
    float  x2_11_x0_11 =  x2_11 * x0_11;

    float  x1_00_x0_00 =  x1_00 * x0_00;
    float  x1_00_x0_01 =  x1_00 * x0_01;
    float  x1_00_x0_10 =  x1_00 * x0_10;
    float  x1_00_x0_11 =  x1_00 * x0_11;
    float  x1_01_x0_00 =  x1_01 * x0_00;
    float  x1_01_x0_01 =  x1_01 * x0_01;
    float  x1_01_x0_10 =  x1_01 * x0_10;
    float  x1_01_x0_11 =  x1_01 * x0_11;
    float  x1_10_x0_00 =  x1_10 * x0_00;
    float  x1_10_x0_01 =  x1_10 * x0_01;
    float  x1_10_x0_10 =  x1_10 * x0_10;
    float  x1_10_x0_11 =  x1_10 * x0_11;
    float  x1_11_x0_00 =  x1_11 * x0_00;
    float  x1_11_x0_01 =  x1_11 * x0_01;
    float  x1_11_x0_10 =  x1_11 * x0_10;
    float  x1_11_x0_11 =  x1_11 * x0_11;


    float dxi;
    float dx0_00 = 0;
    float dx0_01 = 0;
    float dx0_10 = 0;
    float dx0_11 = 0;
    float dx1_00 = 0;
    float dx1_01 = 0;
    float dx1_10 = 0;
    float dx1_11 = 0;
    float dx2_00 = 0;
    float dx2_01 = 0;
    float dx2_10 = 0;
    float dx2_11 = 0;
    dxi = W[ 0][node_id];  dx0_00 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_00_x0_00;
    dxi = W[ 1][node_id];  dx0_01 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_00_x0_01;
    dxi = W[ 2][node_id];  dx0_10 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_00_x0_10;
    dxi = W[ 3][node_id];  dx0_11 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_00_x0_11;
    dxi = W[ 4][node_id];  dx0_00 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_01_x0_00;
    dxi = W[ 5][node_id];  dx0_01 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_01_x0_01;
    dxi = W[ 6][node_id];  dx0_10 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_01_x0_10;
    dxi = W[ 7][node_id];  dx0_11 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_01_x0_11;
    dxi = W[ 8][node_id];  dx0_00 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_10_x0_00;
    dxi = W[ 9][node_id];  dx0_01 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_10_x0_01;
    dxi = W[10][node_id];  dx0_10 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_10_x0_10;
    dxi = W[11][node_id];  dx0_11 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_10_x0_11;
    dxi = W[12][node_id];  dx0_00 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_11_x0_00;
    dxi = W[13][node_id];  dx0_01 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_11_x0_01;
    dxi = W[14][node_id];  dx0_10 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_11_x0_10;
    dxi = W[15][node_id];  dx0_11 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_11_x0_11;
    dxi = W[16][node_id];  dx0_00 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_00_x0_00;
    dxi = W[17][node_id];  dx0_01 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_00_x0_01;
    dxi = W[18][node_id];  dx0_10 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_00_x0_10;
    dxi = W[19][node_id];  dx0_11 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_00_x0_11;
    dxi = W[20][node_id];  dx0_00 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_01_x0_00;
    dxi = W[21][node_id];  dx0_01 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_01_x0_01;
    dxi = W[22][node_id];  dx0_10 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_01_x0_10;
    dxi = W[23][node_id];  dx0_11 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_01_x0_11;
    dxi = W[24][node_id];  dx0_00 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_10_x0_00;
    dxi = W[25][node_id];  dx0_01 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_10_x0_01;
    dxi = W[26][node_id];  dx0_10 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_10_x0_10;
    dxi = W[27][node_id];  dx0_11 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_10_x0_11;
    dxi = W[28][node_id];  dx0_00 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_11_x0_00;
    dxi = W[29][node_id];  dx0_01 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_11_x0_01;
    dxi = W[30][node_id];  dx0_10 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_11_x0_10;
    dxi = W[31][node_id];  dx0_11 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_11_x0_11;
    dxi = W[32][node_id];  dx0_00 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_00_x0_00;
    dxi = W[33][node_id];  dx0_01 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_00_x0_01;
    dxi = W[34][node_id];  dx0_10 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_00_x0_10;
    dxi = W[35][node_id];  dx0_11 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_00_x0_11;
    dxi = W[36][node_id];  dx0_00 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_01_x0_00;
    dxi = W[37][node_id];  dx0_01 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_01_x0_01;
    dxi = W[38][node_id];  dx0_10 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_01_x0_10;
    dxi = W[39][node_id];  dx0_11 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_01_x0_11;
    dxi = W[40][node_id];  dx0_00 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_10_x0_00;
    dxi = W[41][node_id];  dx0_01 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_10_x0_01;
    dxi = W[42][node_id];  dx0_10 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_10_x0_10;
    dxi = W[43][node_id];  dx0_11 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_10_x0_11;
    dxi = W[44][node_id];  dx0_00 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_11_x0_00;
    dxi = W[45][node_id];  dx0_01 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_11_x0_01;
    dxi = W[46][node_id];  dx0_10 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_11_x0_10;
    dxi = W[47][node_id];  dx0_11 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_11_x0_11;
    dxi = W[48][node_id];  dx0_00 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_00_x0_00;
    dxi = W[49][node_id];  dx0_01 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_00_x0_01;
    dxi = W[50][node_id];  dx0_10 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_00_x0_10;
    dxi = W[51][node_id];  dx0_11 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_00_x0_11;
    dxi = W[52][node_id];  dx0_00 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_01_x0_00;
    dxi = W[53][node_id];  dx0_01 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_01_x0_01;
    dxi = W[54][node_id];  dx0_10 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_01_x0_10;
    dxi = W[55][node_id];  dx0_11 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_01_x0_11;
    dxi = W[56][node_id];  dx0_00 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_10_x0_00;
    dxi = W[57][node_id];  dx0_01 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_10_x0_01;
    dxi = W[58][node_id];  dx0_10 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_10_x0_10;
    dxi = W[59][node_id];  dx0_11 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_10_x0_11;
    dxi = W[60][node_id];  dx0_00 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_11_x0_00;
    dxi = W[61][node_id];  dx0_01 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_11_x0_01;
    dxi = W[62][node_id];  dx0_10 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_11_x0_10;
    dxi = W[63][node_id];  dx0_11 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_11_x0_11;
    
    float dxn;
    float dxp;
    float dx;
    dxn  = dx0_00 * xn[1];    dxn += dx0_10 * xp[1];
    dxp  = dx0_01 * xn[1];    dxp += dx0_11 * xp[1];
    dx = (dxp - dxn) * dy;
    if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
    dx_ptr[0 * frame_stride] = dx;

    dxn  = dx0_00 * xn[0];
    dxn += dx0_01 * xp[0];
    dxp  = dx0_10 * xn[0];
    dxp += dx0_11 * xp[0];
    dx = (dxp - dxn) * dy;
    if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
    dx_ptr[1 * frame_stride] = dx;

    dxn  = dx1_00 * xn[3];     
    dxp  = dx1_01 * xn[3];     
    dxn += dx1_10 * xp[3];     
    dxp += dx1_11 * xp[3];     
    dx = (dxp - dxn) * dy;
    if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
    dx_ptr[2 * frame_stride] = dx;

    dxn  = dx1_00 * xn[2];
    dxn += dx1_01 * xp[2];
    dxp  = dx1_10 * xn[2];
    dxp += dx1_11 * xp[2];
    dx = (dxp - dxn) * dy;
    if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
    dx_ptr[3 * frame_stride] = dx;

    dxn  = dx2_00 * xn[5];     
    dxp  = dx2_01 * xn[5];     
    dxn += dx2_10 * xp[5];     
    dxp += dx2_11 * xp[5];     
    dx = (dxp - dxn) * dy;
    if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
    dx_ptr[4 * frame_stride] = dx;

    dxn  = dx2_00 * xn[4];
    dxn += dx2_01 * xp[4];
    dxp  = dx2_10 * xn[4];
    dxp += dx2_11 * xp[4];
    dx = (dxp - dxn) * dy;
    if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
    dx_ptr[5 * frame_stride] = dx;
}
#endif


#if 1

template<int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_bit_fp32_SparseBinaryLut6_BackwardPhase0
        (
            int   const     *x_buf,
            float const     *dy_buf,
            int   const     *input_index,
            float const     *W_buf,
            float           *dW_buf,
            float const     *mean_buf,
            float const     *rstd_buf,
            float           *dmean_buf,
            float           *dvar_buf,
            float           gamma,
            float           beta,
            float           reciprocal_frame_size,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize
        )
{

    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  float       W[64][MAX_NODE_UNIT];
                int   const *x_ptr[6];
                float const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // init pointer
        for ( int i = 0; i < 6; ++i ) {
            int input_node = input_index[6*node + i];
            x_ptr[i]  = &x_buf[input_node * bin_frame_stride];
        }

        dy_ptr = &dy_buf[node * frame_stride];
    }

    __syncthreads();
    

    float mean;
    float rstd;
    if ( node < node_size ) {
        mean = mean_buf[node];
        rstd = rstd_buf[node];
    }
    float rstd2 = rstd * rstd;

    float dmeanx = 0;
    float dstd   = 0;
    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);
            
            // x ���Čv�Z
            float x_vec[6];
            for ( int i = 0; i < 6; ++i) {
                x_vec[i] = (x_ptr[i][unit] & bit) ? BINARY_ONE : BINARY_ZERO;
            }
//            float x = device_fp32_SparseBinaryLut6_NodeForward<MAX_NODE_UNIT>(node_id, x_vec, W);
            float x = StochasticLut<6, float, MAX_NODE_UNIT>::NodeForward(node_id, x_vec, W);
            float tanh_x = ((x - mean) * rstd) * gamma + beta;
            
            // hard-tanh
            float dy = dy_ptr[frame];
            if (tanh_x <= 0.0) { dy = 0.0; }
            if (tanh_x >= 1.0) { dy = 0.0; }

            // BatchNorm
            float xc = x - mean;
    //      float xn = xc * rstd;
            float dxn = gamma * dy;

    //      printf("[SparseBinaryLut6 bw] frame=%d node=%d x=%f dy=%f\n", frame, node, x, dy);

            dstd   += -(dxn * xc * rstd2);
            dmeanx += -(dxn * rstd);
        }
    }

    dstd   = device_fp32_LocalSum(dstd,   sbuf[node_id]);
    dmeanx = device_fp32_LocalSum(dmeanx, sbuf[node_id]);

    float dvar  = dstd * rstd;
    float dmean = (dmeanx - (mean * dvar)) * reciprocal_frame_size;

    if ( node < node_size ) {
        if ( id == 0 ) {
            dvar_buf[node]  = dvar;
            dmean_buf[node] = dmean;
        }
    }  
}

template<int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_bit_fp32_SparseBinaryLut6_BackwardPhase1
        (
            int   const     *x_buf,
            float const     *dy_buf,
            float           *dx_buf,
            int   const     *input_index,
            float const     *W_buf,
            float           *dW_buf,
            float const     *mean_buf,
            float const     *rstd_buf,
            float const     *dmean_buf,
            float const     *dvar_buf,
            float           gamma,
            float           beta,
            float           reciprocal_frame_size,
            int             node_size,
            int             frame_size,
            int             x_frame_stride,
            int             dy_frame_stride,
            int             dx_frame_stride,
            int             lut_binarize
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  float       dW_prev[64][MAX_NODE_UNIT];
    __shared__  float       W[64][MAX_NODE_UNIT];
                float       dW[64];
                int   const *x_ptr[6];
                float const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < 64; ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < 64; i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * 64 + i];
        }

        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // init pointer
        for ( int i = 0; i < 6; ++i ) {
            int input_node = input_index[6*node + i];
            x_ptr[i]  = &x_buf[input_node * x_frame_stride];
        }

        dy_ptr = &dy_buf[node * dy_frame_stride];
    }
    
    float   mean;
    float   rstd;
    float   dmean;
    float   dvar;
    if ( node < node_size ) {
        mean  = mean_buf[node];
        rstd  = rstd_buf[node];
        dmean = dmean_buf[node];
        dvar  = dvar_buf[node];
    }

    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);
            
            // x ���Čv�Z
            float x_vec[6];
            for ( int i = 0; i < 6; ++i) {
                x_vec[i] = (x_ptr[i][unit] & bit) ? BINARY_ONE : BINARY_ZERO;
            }
//          float x = device_fp32_SparseBinaryLut6_NodeForward<MAX_NODE_UNIT>(node_id, x_vec, W);
            float x = StochasticLut<6, float, MAX_NODE_UNIT>::NodeForward(node_id, x_vec, W);
            float tanh_x = ((x - mean) * rstd) * gamma + beta;

            // hard-tanh
            float dy = dy_ptr[frame];
            if (tanh_x <= 0.0) { dy = 0.0; }
            if (tanh_x >= 1.0) { dy = 0.0; }

            float dxn = dy * gamma;
            float dxc = dxn * rstd;
            float dx  = dxc + dmean + (x * dvar * reciprocal_frame_size);

//          device_fp32_SparseBinaryLut6_NodeBackward<MAX_NODE_UNIT>(node_id, x_vec, dx, &dx_buf[node*6*dx_frame_stride + frame], W, dW, dx_frame_stride);
            StochasticLut<6, float, MAX_NODE_UNIT>::NodeBackward(node_id, x_vec, dx, &dx_buf[node*6*dx_frame_stride + frame], W, dW, dx_frame_stride);
        }
    }

    for ( int i = 0; i < 64; ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < 64; ++i) {
                dW_buf[node*64 + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}


__global__ void kernal_fp32_SparseBinaryLut6_BackwardMarge(
            const float*    src_buf,
            float*          dst_buf,
            const int*      input_index,
            int             node_size,
            int             frame_size,
            int             src_frame_stride,
            int             dst_frame_stride
        )
{
    int frame = blockDim.x * blockIdx.x + threadIdx.x;

    for ( int node = 0; node < node_size; ++node ) {
        if ( frame < frame_size ) {
            for ( int n = 0; n < 6; ++n ) {
                int in_idx = input_index[node*6 + n];
                float*       dst_buf_ptr = &dst_buf[dst_frame_stride * in_idx];
                float        prev_data = dst_buf_ptr[frame];
                const float* src_buf_ptr = &src_buf[(6 * node + n) * src_frame_stride];
                
                dst_buf_ptr[frame] = prev_data + src_buf_ptr[frame];
            }
        }
        __syncthreads();
    }
}




BBCU_DLL_EXPORT int bbcu_bit_fp32_SparseBinaryLut6_Backward
        (
            int   const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            int   const     *dev_reverse_index,
            float const     *dev_W,
            float           *dev_dW,
            float const     *dev_mean_buf,
            float const     *dev_rstd_buf,
            float           *dev_dmean_tmp,
            float           *dev_dvar_tmp,
            float           gamma,
            float           beta,
            int             reverse_index_stride,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             x_frame_stride,
            int             tmp_frame_size,
            int             tmp_frame_stride,
            int             lut_binarize,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size && frame_size > 32 ) { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size              ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size              ) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size && frame_size > 32 ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);
        kernal_bit_fp32_SparseBinaryLut6_BackwardPhase0<MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>
            (
                dev_x_buf,
                dev_dy_buf,
                dev_input_index,
                dev_W,
                dev_dW,
                dev_mean_buf,
                dev_rstd_buf,
                dev_dmean_tmp,
                dev_dvar_tmp,
                gamma,
                beta,
                1.0f / frame_size,
                output_node_size,
                frame_size,
                frame_stride,
                x_frame_stride,
                lut_binarize
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    
//  BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

    int frame_offset = 0;
    do {
        int unit_frame_size = frame_size - frame_offset;
        if (unit_frame_size > tmp_frame_size) {
            unit_frame_size = tmp_frame_size;
        }

        {
            unsigned int const THREAD_SIZE    = 256;
            unsigned int const MAX_FRAME_UNIT = 256;
            unsigned int const MAX_NODE_UNIT  = 16;

    #if 0
            dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
            while ( (int)block.x / 2 >= unit_frame_size && unit_frame_size > 32 ) { block.x /= 2; block.y *= 2; }
            while ( (int)block.y / 2 >= output_node_size                        ) { block.y /= 2; }
    #else
            dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
            while ( (int)block.y / 2 >= output_node_size                        ) { block.y /= 2; block.x *= 2;}
            while ( (int)block.x / 2 >= unit_frame_size && unit_frame_size > 32 ) { block.x /= 2; }
    #endif

            block.x = std::min(block.x, MAX_FRAME_UNIT);
            block.y = std::min(block.y, MAX_NODE_UNIT);
            dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);
            kernal_bit_fp32_SparseBinaryLut6_BackwardPhase1<MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>
                (
                    dev_x_buf  + (frame_offset / 32),
                    dev_dy_buf + frame_offset,
                    dev_dx_tmp,
                    dev_input_index,
                    dev_W,
                    dev_dW,
                    dev_mean_buf,
                    dev_rstd_buf,
                    dev_dmean_tmp,
                    dev_dvar_tmp,
                    gamma,
                    beta,
                    1.0f / frame_size,
                    output_node_size,
                    unit_frame_size,
                    x_frame_stride,
                    frame_stride,
                    tmp_frame_stride,
                    lut_binarize
                );
            BB_CUDA_CHECK_LAST_ERROR();
        }

#if 1
        {
            unsigned int const THREAD_SIZE    = 1024;
            unsigned int const MAX_FRAME_UNIT = 1024;
            unsigned int const MAX_NODE_UNIT  = 1024;

    #if 1
            dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
            while ( (int)block.x / 2 >= unit_frame_size ) { block.x /= 2; block.y *= 2; }
            while ( (int)block.y / 2 >= input_node_size ) { block.y /= 2; }
    #else
            dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
            while ( (int)block.y / 2 >= input_node_size ) { block.y /= 2; block.x *= 2;}
            while ( (int)block.x / 2 >= unit_frame_size ) { block.x /= 2; }
    #endif

            block.x = std::min(block.x, MAX_FRAME_UNIT);
            block.y = std::min(block.y, MAX_NODE_UNIT);
            dim3    grid((unit_frame_size + (block.x - 1)) / block.x, (input_node_size + (block.y - 1)) / block.y);

            kernal_BackwardMarge<float><<<grid, block>>>
                (
                    dev_dx_tmp,
                    dev_dx_buf + frame_offset,
                    dev_reverse_index,
                    reverse_index_stride,
                    input_node_size,
                    unit_frame_size,
                    tmp_frame_stride,
                    frame_stride
                );
            BB_CUDA_CHECK_LAST_ERROR();
        }
#else
        {
            int block_x = frame_size;
            while ( block_x > 1024 ) { block_x /= 2; }

            dim3    grid((frame_size + block_x - 1) /block_x, 1);
            dim3    block(block_x, 1, 1);
            kernal_fp32_SparseBinaryLut6_BackwardMarge<<<grid, block>>>
                (
                    dev_dx_tmp,
                    dev_dx_buf + frame_offset,
                    dev_input_index,
                    output_node_size,
                    unit_frame_size,
                    tmp_frame_stride,
                    frame_stride
                );
            BB_CUDA_CHECK_LAST_ERROR();
        }
#endif

        frame_offset += unit_frame_size;
    } while ( frame_offset < frame_size );

    return 0;
}    



#else


template<int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_bit_fp32_SparseBinaryLut6_Backward
        (
            int   const     *x_buf,
            float const     *dy_buf,
            float           *dx_buf,
            int   const     *input_index,
            float const     *W_buf,
            float           *dW_buf,
            float const     *mean_buf,
            float const     *rstd_buf,
            float           gamma,
            float           beta,
            float           reciprocal_frame_size,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize
        )
{

    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  float       dW_prev[64][MAX_NODE_UNIT];
    __shared__  float       W[64][MAX_NODE_UNIT];
                float       dW[64];
                int   const *x_ptr[6];
                float const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < 64; ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < 64; i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * 64 + i];
        }

        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // init pointer
        for ( int i = 0; i < 6; ++i ) {
            int input_node = input_index[6*node + i];
            x_ptr[i]  = &x_buf[input_node * bin_frame_stride];
        }

        dy_ptr = &dy_buf[node * frame_stride];
    }

    __syncthreads();
    

    float mean = mean_buf[node];
    float rstd = rstd_buf[node];
    float rstd2 = rstd * rstd;
    
    float dmeanx = 0;
    float dstd   = 0;
    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);
            
            // x ���Čv�Z
            float x_vec[6];
            for ( int i = 0; i < 6; ++i) {
                x_vec[i] = (x_ptr[i][unit] & bit) ? BINARY_ONE : BINARY_ZERO;
            }
            float x = device_fp32_SparseBinaryLut6_NodeForward<MAX_NODE_UNIT>(node_id, x_vec, W);
            float tanh_x = ((x - mean) * rstd) * gamma + beta;
            
            // hard-tanh
            float dy = dy_ptr[frame];
            if (tanh_x <= 0.0) { dy = 0.0; }
            if (tanh_x >= 1.0) { dy = 0.0; }

            // BatchNorm
            float xc = x - mean;
    //      float xn = xc * rstd;
            float dxn = gamma * dy;

    //      printf("[SparseBinaryLut6 bw] frame=%d node=%d x=%f dy=%f\n", frame, node, x, dy);

            dstd   += -(dxn * xc * rstd2);
            dmeanx += -(dxn * rstd);
        }
    }

    dstd   = device_fp32_LocalSum(dstd,   sbuf[node_id]);
    dmeanx = device_fp32_LocalSum(dmeanx, sbuf[node_id]);

    float dvar  = dstd * rstd;
    float dmean = (dmeanx - (mean * dvar)) * reciprocal_frame_size;

    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);
            
            // x ���Čv�Z
            float x_vec[6];
            for ( int i = 0; i < 6; ++i) {
                x_vec[i] = (x_ptr[i][unit] & bit) ? BINARY_ONE : BINARY_ZERO;
            }
            float x = device_fp32_SparseBinaryLut6_NodeForward<MAX_NODE_UNIT>(node_id, x_vec, W);
            float tanh_x = ((x - mean) * rstd) * gamma + beta;

            // hard-tanh
            float dy = dy_ptr[frame];
            if (tanh_x <= 0.0) { dy = 0.0; }
            if (tanh_x >= 1.0) { dy = 0.0; }

            float dxn = dy * gamma;
            float dxc = dxn * rstd;
            float dx  = dxc + dmean + (x * dvar * reciprocal_frame_size);

            device_fp32_SparseBinaryLut6_NodeBackward<MAX_NODE_UNIT>(node_id, x_vec, dx, &dx_buf[node*6*frame_stride + frame], W, dW, frame_stride);
        }
    }

    for ( int i = 0; i < 64; ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < 64; ++i) {
                dW_buf[node*64 + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}


__global__ void kernal_fp32_SparseBinaryLut6_BackwardMarge(
            const float*    src_buf,
            float*          dst_buf,
            const int*      input_index,
            int             node_size,
            int             frame_size,
            int             frame_stride
        )
{
    int frame = blockDim.x * blockIdx.x + threadIdx.x;

    for ( int node = 0; node < node_size; ++node ) {
        if ( frame < frame_size ) {
            for ( int n = 0; n < 6; ++n ) {
                int in_idx = input_index[node*6 + n];
                float*       dst_buf_ptr = &dst_buf[frame_stride * in_idx];
                float        prev_data = dst_buf_ptr[frame];
                const float* src_buf_ptr = &src_buf[(6 * node + n) * frame_stride];
                
                dst_buf_ptr[frame] = prev_data + src_buf_ptr[frame];
            }
        }
        __syncthreads();
    }
}



BBCU_DLL_EXPORT int bbcu_bit_fp32_SparseBinaryLut6_Backward(
            int   const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_W,
            float           *dev_dW,
            float const     *dev_mean_buf,
            float const     *dev_rstd_buf,
            float           *dev_dmean_tmp,
            float           *dev_dvar_tmp,
            float           gamma,
            float           beta,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             x_frame_stride,
            int             lut_binarize,
            hipStream_t    streamId
    )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size && frame_size > 32 ) { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size              ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size              ) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size && frame_size > 32 ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);
        kernal_bit_fp32_SparseBinaryLut6_Backward<MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>
            (
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_W,
                dev_dW,
                dev_mean_buf,
                dev_rstd_buf,
                gamma,
                beta,
                1.0f / frame_size,
                output_node_size,
                frame_size,
                frame_stride,
                x_frame_stride,
                lut_binarize
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = frame_size;
        while ( block_x > 1024 ) { block_x /= 2; }

        dim3    grid((frame_size + block_x - 1) /block_x, 1);
        dim3    block(block_x, 1, 1);
        kernal_fp32_SparseBinaryLut6_BackwardMarge<<<grid, block>>>
            (
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}


#endif


// end of file
