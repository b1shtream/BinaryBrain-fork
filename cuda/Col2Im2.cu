#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



//////////////////////////////
// forward
//////////////////////////////

__global__ void kernal_fp32_Col2Im_Forward(
			const float*	x_buf,
            float*			y_buf,          
			int				hw_size,
            int				c_size,
            int				output_frame_size,
            int				output_frame_stride,
            int				input_frame_stride
		)
{
    int output_frame = blockDim.x * blockIdx.x + threadIdx.x;
    int xy           = blockDim.y * blockIdx.y + threadIdx.y;
    int c            = blockDim.z * blockIdx.z + threadIdx.z;

    if (output_frame < output_frame_size && xy < hw_size ) {
        int output_node = c * hw_size + xy;
        int input_frame = output_frame * hw_size + xy;
        int input_node  = c;

        y_buf[output_node * output_frame_stride + output_frame] = x_buf[input_node * input_frame_stride + input_frame];
    }
}


CUBB_DLL_EXPORT int cubb_fp32_Col2Im_Forward
		(
			float const     *dev_x_buf,
            float           *dev_y_buf,
			int			    w_size,
			int			    h_size,
			int			    c_size,
            int			    input_frame_stride,
            int			    output_frame_size,
			int			    output_frame_stride,
            hipStream_t	streamId
		)
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());
    
    int     hw_size = h_size * w_size;
    
	dim3	block(32, 32, 1);
	dim3	grid((output_frame_size+31)/32, (hw_size+31)/32, c_size);
	
	kernal_fp32_Col2Im_Forward<<<grid, block, 0, streamId>>>(
			dev_x_buf,
            dev_y_buf,          
			hw_size,
            c_size,
            output_frame_size,
            output_frame_stride,
            input_frame_stride
		);
	BB_CUDA_CHECK_LAST_ERROR();

	return 0;
}


//////////////////////////////
// backward
//////////////////////////////

__global__ void kernal_fp32_Col2Im_Backward(
			const float*	dy_buf,
            float*			dx_buf,          
			int				hw_size,
            int				c_size,
            int				output_frame_size,
            int				output_frame_stride,
            int				input_frame_stride
		)
{
    int output_frame = blockDim.x * blockIdx.x + threadIdx.x;
    int xy           = blockDim.y * blockIdx.y + threadIdx.y;
    int c            = blockDim.z * blockIdx.z + threadIdx.z;

    if (output_frame < output_frame_size && xy < hw_size ) {
        int output_node = c * hw_size + xy;
        int input_frame = output_frame * hw_size + xy;
        int input_node  = c;

         dx_buf[input_node * input_frame_stride + input_frame] = dy_buf[output_node * output_frame_stride + output_frame];
    }
}

CUBB_DLL_EXPORT int cubb_fp32_Col2Im_Backward
		(
			float const     *dev_dy_buf,
            float           *dev_dx_buf,
			int			    w_size,
			int			    h_size,
			int			    c_size,
            int			    input_frame_stride,
            int			    output_frame_size,
			int			    output_frame_stride,
            hipStream_t	streamId
		)
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());
    
    int     hw_size = h_size * w_size;
    
	dim3	block(32, 32, 1);
	dim3	grid((output_frame_size+31)/32, (hw_size+31)/32, c_size);
	
	kernal_fp32_Col2Im_Backward<<<grid, block, 0, streamId>>>(
			dev_dy_buf,
            dev_dx_buf,          
			hw_size,
            c_size,
            output_frame_size,
            output_frame_stride,
            input_frame_stride
		);
	BB_CUDA_CHECK_LAST_ERROR();

	return 0;
}


