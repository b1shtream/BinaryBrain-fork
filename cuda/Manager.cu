#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



static bool bbcu_HostOnly = false;


CUBB_DLL_EXPORT void bbcu_SetHostOnly(bool hostOnly)
{
    bbcu_HostOnly = hostOnly;
}


CUBB_DLL_EXPORT bool bbcu_IsHostOnly(void)
{
    return bbcu_HostOnly;
}


CUBB_DLL_EXPORT bool bbcu_IsDeviceAvailable(void)
{
    return !bbcu_HostOnly;
}


// end of file
