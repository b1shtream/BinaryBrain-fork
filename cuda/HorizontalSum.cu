#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// kernel
__global__ void kernel_HorizontalSum(
			const float*	src,
			float*			dst,
			int				size)
{
	extern __shared__   float	buf[];

	// ������
	int y      = blockIdx.x;
	int x      = threadIdx.x;
	int x_step = blockDim.x;

	// �ǂݍ���
	float acc = 0;
	const float* src_ptr = &src[size * y];
	while (x < size) {
		acc += src_ptr[x];
		x += x_step;
	}
	buf[threadIdx.x] = acc;

	__syncthreads();

	x = threadIdx.x;
	int comb = 1;
	while ( comb < size ) {
		int next = comb * 2;
		int mask = next - 1;
		if ( (x & mask) == 0 ) {
			buf[x] += buf[x + comb];
		}
		comb = next;
		__syncthreads();
	}
	
	dst[y] = buf[0];
}


int bbcu_HorizontalSum
		(
			const float*	dev_src,
			float*			dev_dst,
			int				x_size,
			int				y_size,
			hipStream_t	streamId
		)
{
	int		unit_x = 512;

	dim3	grid(y_size);
	dim3	block(unit_x);
	
	kernel_HorizontalSum<<<grid, block, unit_x*sizeof(float), streamId>>>(
			dev_src,
			dev_dst,
			x_size);

	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
		return 1;
    }
	
	return 0;
}



int bbcu_eva_HorizontalSum
		(
			const float*	src,
			float*			dst,
			int				x_size,
			int				y_size
		)
{
	float*	dev_src;
	float*	dev_dst;

	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_src, y_size * x_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_dst, y_size * sizeof(float)));

	BB_CUDA_SAFE_CALL(hipMemcpy(dev_src, src, y_size * x_size * sizeof(float), hipMemcpyHostToDevice));

	bbcu_HorizontalSum(dev_src, dev_dst, x_size, y_size, 0);

	BB_CUDA_SAFE_CALL(hipMemcpy(dst, dev_dst, y_size * sizeof(float), hipMemcpyDeviceToHost));

	return 0;
}

