#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// kernel
__global__ void kernel_fp32_HorizontalSum(
			const float*	src,
			float*			dst,
			int				size)
{
	extern __shared__   float	buf[];

	// ������
	int y      = blockIdx.x;
	int x      = threadIdx.x;
	int x_step = blockDim.x;

	// �ǂݍ���
	float acc = 0;
	const float* src_ptr = &src[size * y];
	while (x < size) {
		acc += src_ptr[x];
		x += x_step;
	}
	buf[threadIdx.x] = acc;

	__syncthreads();

	x = threadIdx.x;
	int comb = 1;
	while ( comb < size ) {
		int next = comb * 2;
		int mask = next - 1;
		if ( (x & mask) == 0 ) {
			buf[x] += buf[x + comb];
		}
		comb = next;
		__syncthreads();
	}
	
	dst[y] = buf[0];
}


int bbcu_fp32_HorizontalSum
		(
			const float*	dev_src,
			float*			dev_dst,
			int				x_size,
			int				y_size,
			hipStream_t	streamId
		)
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

	int		unit_x = 512;

	dim3	grid(y_size);
	dim3	block(unit_x);
	
	kernel_fp32_HorizontalSum<<<grid, block, unit_x*sizeof(float), streamId>>>(
			dev_src,
			dev_dst,
			x_size);

	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
		return 1;
    }
	
	return 0;
}

