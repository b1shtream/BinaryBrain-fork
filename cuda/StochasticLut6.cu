#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// -------------------------------------------------
//  Forward
// -------------------------------------------------


__global__ void kernal_fp32_StochasticLut6_Forward(
            float const     *x_buf,
            float           *y_buf,
            int   const     *input_index,
            float const     *W_buf,
            int             frame_size,
            int             frame_stride,
            int             binary_mode
        )
{
    int node    = blockIdx.x;
    int id      = threadIdx.x;
    int id_step = blockDim.x;
    
    // read W
    __shared__ float    W[64];
    for ( int i = id; i < 64; i += id_step ) {
        W[i] = W_buf[node * 64 + i];
        if ( binary_mode ) {
            W[i] = W[i] > 0.5 ? 1.0 : 0.0;
        }
    }
    
    // read input index
    __shared__ float const  *x_ptr[6];
    for ( int i = id; i < 6; i += id_step ) {
        x_ptr[i] = &x_buf[frame_stride * input_index[6*node + i]];
    }
    float        *y_ptr = &y_buf[node * frame_stride];

    __syncthreads();

    for (int frame = id; frame < frame_size; frame += id_step) {
        float   xp[6], xn[6];
        for ( int i = 0; i < 6; ++i) {
            xp[i] = x_ptr[i][frame];
            xp[i] = min(1.0, max(0.0, xp[i]));
            xn[i] = 1.0 - xp[i];
        }

        float x0_00 = xn[1] * xn[0];
        float x0_01 = xn[1] * xp[0];
        float x0_10 = xp[1] * xn[0];
        float x0_11 = xp[1] * xp[0];
        float x1_00 = xn[3] * xn[2];
        float x1_01 = xn[3] * xp[2];
        float x1_10 = xp[3] * xn[2];
        float x1_11 = xp[3] * xp[2];
        float x2_00 = xn[5] * xn[4];
        float x2_01 = xn[5] * xp[4];
        float x2_10 = xp[5] * xn[4];
        float x2_11 = xp[5] * xp[4];

        float y = 0;
        y += W[0 ] * x2_00 * x1_00 * x0_00;
        y += W[1 ] * x2_00 * x1_00 * x0_01;
        y += W[2 ] * x2_00 * x1_00 * x0_10;
        y += W[3 ] * x2_00 * x1_00 * x0_11;
        y += W[4 ] * x2_00 * x1_01 * x0_00;
        y += W[5 ] * x2_00 * x1_01 * x0_01;
        y += W[6 ] * x2_00 * x1_01 * x0_10;
        y += W[7 ] * x2_00 * x1_01 * x0_11;
        y += W[8 ] * x2_00 * x1_10 * x0_00;
        y += W[9 ] * x2_00 * x1_10 * x0_01;
        y += W[10] * x2_00 * x1_10 * x0_10;
        y += W[11] * x2_00 * x1_10 * x0_11;
        y += W[12] * x2_00 * x1_11 * x0_00;
        y += W[13] * x2_00 * x1_11 * x0_01;
        y += W[14] * x2_00 * x1_11 * x0_10;
        y += W[15] * x2_00 * x1_11 * x0_11;
        y += W[16] * x2_01 * x1_00 * x0_00;
        y += W[17] * x2_01 * x1_00 * x0_01;
        y += W[18] * x2_01 * x1_00 * x0_10;
        y += W[19] * x2_01 * x1_00 * x0_11;
        y += W[20] * x2_01 * x1_01 * x0_00;
        y += W[21] * x2_01 * x1_01 * x0_01;
        y += W[22] * x2_01 * x1_01 * x0_10;
        y += W[23] * x2_01 * x1_01 * x0_11;
        y += W[24] * x2_01 * x1_10 * x0_00;
        y += W[25] * x2_01 * x1_10 * x0_01;
        y += W[26] * x2_01 * x1_10 * x0_10;
        y += W[27] * x2_01 * x1_10 * x0_11;
        y += W[28] * x2_01 * x1_11 * x0_00;
        y += W[29] * x2_01 * x1_11 * x0_01;
        y += W[30] * x2_01 * x1_11 * x0_10;
        y += W[31] * x2_01 * x1_11 * x0_11;
        y += W[32] * x2_10 * x1_00 * x0_00;
        y += W[33] * x2_10 * x1_00 * x0_01;
        y += W[34] * x2_10 * x1_00 * x0_10;
        y += W[35] * x2_10 * x1_00 * x0_11;
        y += W[36] * x2_10 * x1_01 * x0_00;
        y += W[37] * x2_10 * x1_01 * x0_01;
        y += W[38] * x2_10 * x1_01 * x0_10;
        y += W[39] * x2_10 * x1_01 * x0_11;
        y += W[40] * x2_10 * x1_10 * x0_00;
        y += W[41] * x2_10 * x1_10 * x0_01;
        y += W[42] * x2_10 * x1_10 * x0_10;
        y += W[43] * x2_10 * x1_10 * x0_11;
        y += W[44] * x2_10 * x1_11 * x0_00;
        y += W[45] * x2_10 * x1_11 * x0_01;
        y += W[46] * x2_10 * x1_11 * x0_10;
        y += W[47] * x2_10 * x1_11 * x0_11;
        y += W[48] * x2_11 * x1_00 * x0_00;
        y += W[49] * x2_11 * x1_00 * x0_01;
        y += W[50] * x2_11 * x1_00 * x0_10;
        y += W[51] * x2_11 * x1_00 * x0_11;
        y += W[52] * x2_11 * x1_01 * x0_00;
        y += W[53] * x2_11 * x1_01 * x0_01;
        y += W[54] * x2_11 * x1_01 * x0_10;
        y += W[55] * x2_11 * x1_01 * x0_11;
        y += W[56] * x2_11 * x1_10 * x0_00;
        y += W[57] * x2_11 * x1_10 * x0_01;
        y += W[58] * x2_11 * x1_10 * x0_10;
        y += W[59] * x2_11 * x1_10 * x0_11;
        y += W[60] * x2_11 * x1_11 * x0_00;
        y += W[61] * x2_11 * x1_11 * x0_01;
        y += W[62] * x2_11 * x1_11 * x0_10;
        y += W[63] * x2_11 * x1_11 * x0_11;
        
        // clamp
        y = max(0.0, y);
        y = min(1.0, y);
        
        y_ptr[frame] = y;
    }
}


int bbcu_fp32_StochasticLut6_Forward
        (
            const float     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             binary_mode,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    dim3    block(512);
    dim3    grid(node_size);
    while ( frame_size < (int)block.x / 2 ) {
        block.x /= 2;
    }
    
    kernal_fp32_StochasticLut6_Forward<<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            frame_size,
            frame_stride,
            binary_mode
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



// -------------------------------------------------
//  Backward
// -------------------------------------------------


__device__ __forceinline__ float device_fp32_LocalSum(float v, float *buf)
{
    buf[threadIdx.x] = v;
    __syncthreads();

    // �X���b�h�ԏW�v
    int comb = 1;
    while (comb < blockDim.x) {
        int next = comb * 2;
        int mask = next - 1;
        if ((threadIdx.x & mask) == 0) {
            buf[threadIdx.x] += buf[threadIdx.x + comb];
        }
        comb = next;
        __syncthreads();
    }

    float sum = buf[0];
    __syncthreads();
    
    return sum;
}



// kernel
template<int THREAD_SIZE=256>
__global__ void kernal_fp32_StochasticLut6_Backward
        (
            float const     *x_buf,
            float const     *dy_buf,
            float           *dx_buf,
            int   const     *input_index,
            float const     *W_buf,
            float           *dW_buf,
            int             frame_size,
            int             frame_stride,
            int             binary_mode
        )
{
    __shared__ float buf[THREAD_SIZE];

    int node    = blockIdx.x;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    // initialize dW
    float dW[64];
    for ( int i = 0; i < 64; ++i) {
        dW[i] = 0;
    }

    __shared__ float    dW_prev[64];
    for ( int i = id; i < 64; i += id_step ) {
        dW_prev[i] = dW_buf[node * 64 + i];
    }

    // read W
    __shared__ float    W[64];
    for ( int i = id; i < 64; i += id_step ) {
        W[i] = W_buf[node * 64 + i];
        if ( binary_mode ) {
            W[i] = W[i] > 0.5 ? 1.0 : 0.0;
        }
    }
    
    // init pointer
    __shared__  float const *x_ptr[6];
    for ( int i = id; i < 6; i += id_step ) {
        int input_node = input_index[6*node + i];
        x_ptr[i]  = &x_buf[frame_stride * input_node];
    }
    float const *dy_ptr = &dy_buf[node*frame_stride];

    __syncthreads();

    for ( int frame = id; frame < frame_size; frame += id_step ) {
        float xp[6], xn[6];
        for ( int i = 0; i < 6; ++i) {
            xp[i] = x_ptr[i][frame];
            xp[i] = min(1.0, max(0.0, xp[i]));
            xn[i] = 1.0 - xp[i];
        }

        float x0_00 = xn[1] * xn[0];
        float x0_01 = xn[1] * xp[0];
        float x0_10 = xp[1] * xn[0];
        float x0_11 = xp[1] * xp[0];
        float x1_00 = xn[3] * xn[2];
        float x1_01 = xn[3] * xp[2];
        float x1_10 = xp[3] * xn[2];
        float x1_11 = xp[3] * xp[2];
        float x2_00 = xn[5] * xn[4];
        float x2_01 = xn[5] * xp[4];
        float x2_10 = xp[5] * xn[4];
        float x2_11 = xp[5] * xp[4];

        float grad = dy_ptr[frame];

        dW[0]  += x2_00 * x1_00 * x0_00 * grad;
        dW[1]  += x2_00 * x1_00 * x0_01 * grad;
        dW[2]  += x2_00 * x1_00 * x0_10 * grad;
        dW[3]  += x2_00 * x1_00 * x0_11 * grad;
        dW[4]  += x2_00 * x1_01 * x0_00 * grad;
        dW[5]  += x2_00 * x1_01 * x0_01 * grad;
        dW[6]  += x2_00 * x1_01 * x0_10 * grad;
        dW[7]  += x2_00 * x1_01 * x0_11 * grad;
        dW[8]  += x2_00 * x1_10 * x0_00 * grad;
        dW[9]  += x2_00 * x1_10 * x0_01 * grad;
        dW[10] += x2_00 * x1_10 * x0_10 * grad;
        dW[11] += x2_00 * x1_10 * x0_11 * grad;
        dW[12] += x2_00 * x1_11 * x0_00 * grad;
        dW[13] += x2_00 * x1_11 * x0_01 * grad;
        dW[14] += x2_00 * x1_11 * x0_10 * grad;
        dW[15] += x2_00 * x1_11 * x0_11 * grad;
        dW[16] += x2_01 * x1_00 * x0_00 * grad;
        dW[17] += x2_01 * x1_00 * x0_01 * grad;
        dW[18] += x2_01 * x1_00 * x0_10 * grad;
        dW[19] += x2_01 * x1_00 * x0_11 * grad;
        dW[20] += x2_01 * x1_01 * x0_00 * grad;
        dW[21] += x2_01 * x1_01 * x0_01 * grad;
        dW[22] += x2_01 * x1_01 * x0_10 * grad;
        dW[23] += x2_01 * x1_01 * x0_11 * grad;
        dW[24] += x2_01 * x1_10 * x0_00 * grad;
        dW[25] += x2_01 * x1_10 * x0_01 * grad;
        dW[26] += x2_01 * x1_10 * x0_10 * grad;
        dW[27] += x2_01 * x1_10 * x0_11 * grad;
        dW[28] += x2_01 * x1_11 * x0_00 * grad;
        dW[29] += x2_01 * x1_11 * x0_01 * grad;
        dW[30] += x2_01 * x1_11 * x0_10 * grad;
        dW[31] += x2_01 * x1_11 * x0_11 * grad;
        dW[32] += x2_10 * x1_00 * x0_00 * grad;
        dW[33] += x2_10 * x1_00 * x0_01 * grad;
        dW[34] += x2_10 * x1_00 * x0_10 * grad;
        dW[35] += x2_10 * x1_00 * x0_11 * grad;
        dW[36] += x2_10 * x1_01 * x0_00 * grad;
        dW[37] += x2_10 * x1_01 * x0_01 * grad;
        dW[38] += x2_10 * x1_01 * x0_10 * grad;
        dW[39] += x2_10 * x1_01 * x0_11 * grad;
        dW[40] += x2_10 * x1_10 * x0_00 * grad;
        dW[41] += x2_10 * x1_10 * x0_01 * grad;
        dW[42] += x2_10 * x1_10 * x0_10 * grad;
        dW[43] += x2_10 * x1_10 * x0_11 * grad;
        dW[44] += x2_10 * x1_11 * x0_00 * grad;
        dW[45] += x2_10 * x1_11 * x0_01 * grad;
        dW[46] += x2_10 * x1_11 * x0_10 * grad;
        dW[47] += x2_10 * x1_11 * x0_11 * grad;
        dW[48] += x2_11 * x1_00 * x0_00 * grad;
        dW[49] += x2_11 * x1_00 * x0_01 * grad;
        dW[50] += x2_11 * x1_00 * x0_10 * grad;
        dW[51] += x2_11 * x1_00 * x0_11 * grad;
        dW[52] += x2_11 * x1_01 * x0_00 * grad;
        dW[53] += x2_11 * x1_01 * x0_01 * grad;
        dW[54] += x2_11 * x1_01 * x0_10 * grad;
        dW[55] += x2_11 * x1_01 * x0_11 * grad;
        dW[56] += x2_11 * x1_10 * x0_00 * grad;
        dW[57] += x2_11 * x1_10 * x0_01 * grad;
        dW[58] += x2_11 * x1_10 * x0_10 * grad;
        dW[59] += x2_11 * x1_10 * x0_11 * grad;
        dW[60] += x2_11 * x1_11 * x0_00 * grad;
        dW[61] += x2_11 * x1_11 * x0_01 * grad;
        dW[62] += x2_11 * x1_11 * x0_10 * grad;
        dW[63] += x2_11 * x1_11 * x0_11 * grad;

        float dxi;
        float dx0_00 = 0;
        float dx0_01 = 0;
        float dx0_10 = 0;
        float dx0_11 = 0;
        float dx1_00 = 0;
        float dx1_01 = 0;
        float dx1_10 = 0;
        float dx1_11 = 0;
        float dx2_00 = 0;
        float dx2_01 = 0;
        float dx2_10 = 0;
        float dx2_11 = 0;
        dxi = W[ 0] * grad;  dx0_00 += dxi * x2_00 * x1_00;  dx1_00 += dxi * x2_00 * x0_00;  dx2_00 += dxi * x1_00 * x0_00;
        dxi = W[ 1] * grad;  dx0_01 += dxi * x2_00 * x1_00;  dx1_00 += dxi * x2_00 * x0_01;  dx2_00 += dxi * x1_00 * x0_01;
        dxi = W[ 2] * grad;  dx0_10 += dxi * x2_00 * x1_00;  dx1_00 += dxi * x2_00 * x0_10;  dx2_00 += dxi * x1_00 * x0_10;
        dxi = W[ 3] * grad;  dx0_11 += dxi * x2_00 * x1_00;  dx1_00 += dxi * x2_00 * x0_11;  dx2_00 += dxi * x1_00 * x0_11;
        dxi = W[ 4] * grad;  dx0_00 += dxi * x2_00 * x1_01;  dx1_01 += dxi * x2_00 * x0_00;  dx2_00 += dxi * x1_01 * x0_00;
        dxi = W[ 5] * grad;  dx0_01 += dxi * x2_00 * x1_01;  dx1_01 += dxi * x2_00 * x0_01;  dx2_00 += dxi * x1_01 * x0_01;
        dxi = W[ 6] * grad;  dx0_10 += dxi * x2_00 * x1_01;  dx1_01 += dxi * x2_00 * x0_10;  dx2_00 += dxi * x1_01 * x0_10;
        dxi = W[ 7] * grad;  dx0_11 += dxi * x2_00 * x1_01;  dx1_01 += dxi * x2_00 * x0_11;  dx2_00 += dxi * x1_01 * x0_11;
        dxi = W[ 8] * grad;  dx0_00 += dxi * x2_00 * x1_10;  dx1_10 += dxi * x2_00 * x0_00;  dx2_00 += dxi * x1_10 * x0_00;
        dxi = W[ 9] * grad;  dx0_01 += dxi * x2_00 * x1_10;  dx1_10 += dxi * x2_00 * x0_01;  dx2_00 += dxi * x1_10 * x0_01;
        dxi = W[10] * grad;  dx0_10 += dxi * x2_00 * x1_10;  dx1_10 += dxi * x2_00 * x0_10;  dx2_00 += dxi * x1_10 * x0_10;
        dxi = W[11] * grad;  dx0_11 += dxi * x2_00 * x1_10;  dx1_10 += dxi * x2_00 * x0_11;  dx2_00 += dxi * x1_10 * x0_11;
        dxi = W[12] * grad;  dx0_00 += dxi * x2_00 * x1_11;  dx1_11 += dxi * x2_00 * x0_00;  dx2_00 += dxi * x1_11 * x0_00;
        dxi = W[13] * grad;  dx0_01 += dxi * x2_00 * x1_11;  dx1_11 += dxi * x2_00 * x0_01;  dx2_00 += dxi * x1_11 * x0_01;
        dxi = W[14] * grad;  dx0_10 += dxi * x2_00 * x1_11;  dx1_11 += dxi * x2_00 * x0_10;  dx2_00 += dxi * x1_11 * x0_10;
        dxi = W[15] * grad;  dx0_11 += dxi * x2_00 * x1_11;  dx1_11 += dxi * x2_00 * x0_11;  dx2_00 += dxi * x1_11 * x0_11;
        dxi = W[16] * grad;  dx0_00 += dxi * x2_01 * x1_00;  dx1_00 += dxi * x2_01 * x0_00;  dx2_01 += dxi * x1_00 * x0_00;
        dxi = W[17] * grad;  dx0_01 += dxi * x2_01 * x1_00;  dx1_00 += dxi * x2_01 * x0_01;  dx2_01 += dxi * x1_00 * x0_01;
        dxi = W[18] * grad;  dx0_10 += dxi * x2_01 * x1_00;  dx1_00 += dxi * x2_01 * x0_10;  dx2_01 += dxi * x1_00 * x0_10;
        dxi = W[19] * grad;  dx0_11 += dxi * x2_01 * x1_00;  dx1_00 += dxi * x2_01 * x0_11;  dx2_01 += dxi * x1_00 * x0_11;
        dxi = W[20] * grad;  dx0_00 += dxi * x2_01 * x1_01;  dx1_01 += dxi * x2_01 * x0_00;  dx2_01 += dxi * x1_01 * x0_00;
        dxi = W[21] * grad;  dx0_01 += dxi * x2_01 * x1_01;  dx1_01 += dxi * x2_01 * x0_01;  dx2_01 += dxi * x1_01 * x0_01;
        dxi = W[22] * grad;  dx0_10 += dxi * x2_01 * x1_01;  dx1_01 += dxi * x2_01 * x0_10;  dx2_01 += dxi * x1_01 * x0_10;
        dxi = W[23] * grad;  dx0_11 += dxi * x2_01 * x1_01;  dx1_01 += dxi * x2_01 * x0_11;  dx2_01 += dxi * x1_01 * x0_11;
        dxi = W[24] * grad;  dx0_00 += dxi * x2_01 * x1_10;  dx1_10 += dxi * x2_01 * x0_00;  dx2_01 += dxi * x1_10 * x0_00;
        dxi = W[25] * grad;  dx0_01 += dxi * x2_01 * x1_10;  dx1_10 += dxi * x2_01 * x0_01;  dx2_01 += dxi * x1_10 * x0_01;
        dxi = W[26] * grad;  dx0_10 += dxi * x2_01 * x1_10;  dx1_10 += dxi * x2_01 * x0_10;  dx2_01 += dxi * x1_10 * x0_10;
        dxi = W[27] * grad;  dx0_11 += dxi * x2_01 * x1_10;  dx1_10 += dxi * x2_01 * x0_11;  dx2_01 += dxi * x1_10 * x0_11;
        dxi = W[28] * grad;  dx0_00 += dxi * x2_01 * x1_11;  dx1_11 += dxi * x2_01 * x0_00;  dx2_01 += dxi * x1_11 * x0_00;
        dxi = W[29] * grad;  dx0_01 += dxi * x2_01 * x1_11;  dx1_11 += dxi * x2_01 * x0_01;  dx2_01 += dxi * x1_11 * x0_01;
        dxi = W[30] * grad;  dx0_10 += dxi * x2_01 * x1_11;  dx1_11 += dxi * x2_01 * x0_10;  dx2_01 += dxi * x1_11 * x0_10;
        dxi = W[31] * grad;  dx0_11 += dxi * x2_01 * x1_11;  dx1_11 += dxi * x2_01 * x0_11;  dx2_01 += dxi * x1_11 * x0_11;
        dxi = W[32] * grad;  dx0_00 += dxi * x2_10 * x1_00;  dx1_00 += dxi * x2_10 * x0_00;  dx2_10 += dxi * x1_00 * x0_00;
        dxi = W[33] * grad;  dx0_01 += dxi * x2_10 * x1_00;  dx1_00 += dxi * x2_10 * x0_01;  dx2_10 += dxi * x1_00 * x0_01;
        dxi = W[34] * grad;  dx0_10 += dxi * x2_10 * x1_00;  dx1_00 += dxi * x2_10 * x0_10;  dx2_10 += dxi * x1_00 * x0_10;
        dxi = W[35] * grad;  dx0_11 += dxi * x2_10 * x1_00;  dx1_00 += dxi * x2_10 * x0_11;  dx2_10 += dxi * x1_00 * x0_11;
        dxi = W[36] * grad;  dx0_00 += dxi * x2_10 * x1_01;  dx1_01 += dxi * x2_10 * x0_00;  dx2_10 += dxi * x1_01 * x0_00;
        dxi = W[37] * grad;  dx0_01 += dxi * x2_10 * x1_01;  dx1_01 += dxi * x2_10 * x0_01;  dx2_10 += dxi * x1_01 * x0_01;
        dxi = W[38] * grad;  dx0_10 += dxi * x2_10 * x1_01;  dx1_01 += dxi * x2_10 * x0_10;  dx2_10 += dxi * x1_01 * x0_10;
        dxi = W[39] * grad;  dx0_11 += dxi * x2_10 * x1_01;  dx1_01 += dxi * x2_10 * x0_11;  dx2_10 += dxi * x1_01 * x0_11;
        dxi = W[40] * grad;  dx0_00 += dxi * x2_10 * x1_10;  dx1_10 += dxi * x2_10 * x0_00;  dx2_10 += dxi * x1_10 * x0_00;
        dxi = W[41] * grad;  dx0_01 += dxi * x2_10 * x1_10;  dx1_10 += dxi * x2_10 * x0_01;  dx2_10 += dxi * x1_10 * x0_01;
        dxi = W[42] * grad;  dx0_10 += dxi * x2_10 * x1_10;  dx1_10 += dxi * x2_10 * x0_10;  dx2_10 += dxi * x1_10 * x0_10;
        dxi = W[43] * grad;  dx0_11 += dxi * x2_10 * x1_10;  dx1_10 += dxi * x2_10 * x0_11;  dx2_10 += dxi * x1_10 * x0_11;
        dxi = W[44] * grad;  dx0_00 += dxi * x2_10 * x1_11;  dx1_11 += dxi * x2_10 * x0_00;  dx2_10 += dxi * x1_11 * x0_00;
        dxi = W[45] * grad;  dx0_01 += dxi * x2_10 * x1_11;  dx1_11 += dxi * x2_10 * x0_01;  dx2_10 += dxi * x1_11 * x0_01;
        dxi = W[46] * grad;  dx0_10 += dxi * x2_10 * x1_11;  dx1_11 += dxi * x2_10 * x0_10;  dx2_10 += dxi * x1_11 * x0_10;
        dxi = W[47] * grad;  dx0_11 += dxi * x2_10 * x1_11;  dx1_11 += dxi * x2_10 * x0_11;  dx2_10 += dxi * x1_11 * x0_11;
        dxi = W[48] * grad;  dx0_00 += dxi * x2_11 * x1_00;  dx1_00 += dxi * x2_11 * x0_00;  dx2_11 += dxi * x1_00 * x0_00;
        dxi = W[49] * grad;  dx0_01 += dxi * x2_11 * x1_00;  dx1_00 += dxi * x2_11 * x0_01;  dx2_11 += dxi * x1_00 * x0_01;
        dxi = W[50] * grad;  dx0_10 += dxi * x2_11 * x1_00;  dx1_00 += dxi * x2_11 * x0_10;  dx2_11 += dxi * x1_00 * x0_10;
        dxi = W[51] * grad;  dx0_11 += dxi * x2_11 * x1_00;  dx1_00 += dxi * x2_11 * x0_11;  dx2_11 += dxi * x1_00 * x0_11;
        dxi = W[52] * grad;  dx0_00 += dxi * x2_11 * x1_01;  dx1_01 += dxi * x2_11 * x0_00;  dx2_11 += dxi * x1_01 * x0_00;
        dxi = W[53] * grad;  dx0_01 += dxi * x2_11 * x1_01;  dx1_01 += dxi * x2_11 * x0_01;  dx2_11 += dxi * x1_01 * x0_01;
        dxi = W[54] * grad;  dx0_10 += dxi * x2_11 * x1_01;  dx1_01 += dxi * x2_11 * x0_10;  dx2_11 += dxi * x1_01 * x0_10;
        dxi = W[55] * grad;  dx0_11 += dxi * x2_11 * x1_01;  dx1_01 += dxi * x2_11 * x0_11;  dx2_11 += dxi * x1_01 * x0_11;
        dxi = W[56] * grad;  dx0_00 += dxi * x2_11 * x1_10;  dx1_10 += dxi * x2_11 * x0_00;  dx2_11 += dxi * x1_10 * x0_00;
        dxi = W[57] * grad;  dx0_01 += dxi * x2_11 * x1_10;  dx1_10 += dxi * x2_11 * x0_01;  dx2_11 += dxi * x1_10 * x0_01;
        dxi = W[58] * grad;  dx0_10 += dxi * x2_11 * x1_10;  dx1_10 += dxi * x2_11 * x0_10;  dx2_11 += dxi * x1_10 * x0_10;
        dxi = W[59] * grad;  dx0_11 += dxi * x2_11 * x1_10;  dx1_10 += dxi * x2_11 * x0_11;  dx2_11 += dxi * x1_10 * x0_11;
        dxi = W[60] * grad;  dx0_00 += dxi * x2_11 * x1_11;  dx1_11 += dxi * x2_11 * x0_00;  dx2_11 += dxi * x1_11 * x0_00;
        dxi = W[61] * grad;  dx0_01 += dxi * x2_11 * x1_11;  dx1_11 += dxi * x2_11 * x0_01;  dx2_11 += dxi * x1_11 * x0_01;
        dxi = W[62] * grad;  dx0_10 += dxi * x2_11 * x1_11;  dx1_11 += dxi * x2_11 * x0_10;  dx2_11 += dxi * x1_11 * x0_10;
        dxi = W[63] * grad;  dx0_11 += dxi * x2_11 * x1_11;  dx1_11 += dxi * x2_11 * x0_11;  dx2_11 += dxi * x1_11 * x0_11;
        
        float *dx_ptr = &dx_buf[(node*6)*frame_stride + frame];
        float dxn;
        float dxp;
        float dx;
        dxn  = dx0_00 * xn[1];    dxn += dx0_10 * xp[1];
        dxp  = dx0_01 * xn[1];    dxp += dx0_11 * xp[1];
        dx = dxp - dxn;
        if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
        dx_ptr[0 * frame_stride] = dx;

        dxn  = dx0_00 * xn[0];
        dxn += dx0_01 * xp[0];
        dxp  = dx0_10 * xn[0];
        dxp += dx0_11 * xp[0];
        dx = dxp - dxn;
        if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
        dx_ptr[1 * frame_stride] = dx;

        dxn  = dx1_00 * xn[3];     
        dxp  = dx1_01 * xn[3];     
        dxn += dx1_10 * xp[3];     
        dxp += dx1_11 * xp[3];     
        dx = dxp - dxn;
        if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
        dx_ptr[2 * frame_stride] = dx;

        dxn  = dx1_00 * xn[2];
        dxn += dx1_01 * xp[2];
        dxp  = dx1_10 * xn[2];
        dxp += dx1_11 * xp[2];
        dx = dxp - dxn;
        if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
        dx_ptr[3 * frame_stride] = dx;

        dxn  = dx2_00 * xn[5];     
        dxp  = dx2_01 * xn[5];     
        dxn += dx2_10 * xp[5];     
        dxp += dx2_11 * xp[5];     
        dx = dxp - dxn;
        if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
        dx_ptr[4 * frame_stride] = dx;

        dxn  = dx2_00 * xn[4];
        dxn += dx2_01 * xp[4];
        dxp  = dx2_10 * xn[4];
        dxp += dx2_11 * xp[4];
        dx = dxp - dxn;
        if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
        dx_ptr[5 * frame_stride] = dx;
    }

    for ( int i = 0; i < 64; ++i) {
        dW[i] = device_fp32_LocalSum(dW[i], buf);
    }
    if ( id == 0 ) {
        for ( int i = 0; i < 64; ++i) {
            dW_buf[node*64 + i] = dW[i] + dW_prev[i];
        }
    }
}


__global__ void kernal_fp32_StochasticLut6_BackwardMarge(
            const float*    src_buf,
            float*          dst_buf,
            const int*      input_index,
            int             node_size,
            int             frame_size,
            int             frame_stride
        )
{
    int frame = blockDim.x * blockIdx.x + threadIdx.x;

    for ( int node = 0; node < node_size; ++node ) {
        if ( frame < frame_size ) {
            for ( int n = 0; n < 6; ++n ) {
                int in_idx = input_index[node*6 + n];
                float*       dst_buf_ptr = &dst_buf[frame_stride * in_idx];
                float        prev_data = dst_buf_ptr[frame];
                const float* src_buf_ptr = &src_buf[(6 * node + n) * frame_stride];
                
                dst_buf_ptr[frame] = prev_data + src_buf_ptr[frame];
            }
        }
        __syncthreads();
    }
}


int bbcu_fp32_StochasticLut6_Backward(
            float const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_W,
            float           *dev_dW,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             binary_mode,
            hipStream_t    streamId
    )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        int const thread_size = 256;
        dim3    block(thread_size);
        dim3    grid(output_node_size);
        while ( frame_size < (int)block.x / 2 ) {
            block.x /= 2;
        }

        kernal_fp32_StochasticLut6_Backward<thread_size><<<grid, block, 0, streamId>>>(
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_W,
                dev_dW,
                frame_size,
                frame_stride,
                binary_mode
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = frame_size;
        while ( block_x > 1024 ) { block_x /= 2; }

        dim3    grid((frame_size + block_x - 1) /block_x, 1);
        dim3    block(block_x, 1, 1);
        kernal_fp32_StochasticLut6_BackwardMarge<<<grid, block>>>(
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}





// end of file
