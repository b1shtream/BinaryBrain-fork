#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"
#include "Common.cuh"
#include "StochasticLut.cuh"


// -------------------------------------------------
//  Forward
// -------------------------------------------------

// real type
template<int N=6, typename T=float, int MAX_NODE_UNIT=32>
__global__ void kernal_StochasticLut_Forward(
            T   const   *x_buf,
            T           *y_buf,
            int const   *input_index,
            T   const   *W_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         input_binary,
            int         lut_binarize,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  T       W[(1<<N)][MAX_NODE_UNIT];
                T const *x_ptr[N];
                T       *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < (1<<N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1<<N) + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < N; ++i ) {
            x_ptr[i] = &x_buf[frame_stride * input_index[N*node + i]];
        }

        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            T   x[N];
            if ( input_binary ) {
                for ( int i = 0; i < N; ++i) {
                    x[i] = 0.5 + ((x_ptr[i][frame] > 0.5) ? +unbinarize_bias : -unbinarize_bias);
                }
            }
            else {
                for ( int i = 0; i < N; ++i) {
                    x[i] = min(1.0, max(0.0, x_ptr[i][frame]));
                }
            }

            T   y = StochasticLut<N, T, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

#if 0
            T   xp[N], xn[N];
            for ( int i = 0; i < N; ++i) {
                T x_val = x_ptr[i][frame];
                if ( input_binary ) {
                    x_val = 0.5 + ((x_val > 0.5) ? +unbinarize_bias : -unbinarize_bias);
                }
                else {
                    x_val = min(1.0, max(0.0, x_val));
                }

                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            T x0_00 = xn[1] * xn[0];
            T x0_01 = xn[1] * xp[0];
            T x0_10 = xp[1] * xn[0];
            T x0_11 = xp[1] * xp[0];
            T x1_00 = xn[3] * xn[2];
            T x1_01 = xn[3] * xp[2];
            T x1_10 = xp[3] * xn[2];
            T x1_11 = xp[3] * xp[2];
            T x2_00 = xn[5] * xn[4];
            T x2_01 = xn[5] * xp[4];
            T x2_10 = xp[5] * xn[4];
            T x2_11 = xp[5] * xp[4];

            T y = 0;
            T x2_00_x1_00 = x2_00 * x1_00;
            y += W[0 ][node_id] * x2_00_x1_00 * x0_00;
            y += W[1 ][node_id] * x2_00_x1_00 * x0_01;
            y += W[2 ][node_id] * x2_00_x1_00 * x0_10;
            y += W[3 ][node_id] * x2_00_x1_00 * x0_11;
            T x2_00_x1_01 = x2_00 * x1_01;
            y += W[4 ][node_id] * x2_00_x1_01 * x0_00;
            y += W[5 ][node_id] * x2_00_x1_01 * x0_01;
            y += W[6 ][node_id] * x2_00_x1_01 * x0_10;
            y += W[7 ][node_id] * x2_00_x1_01 * x0_11;
            T x2_00_x1_10 = x2_00 * x1_10;
            y += W[8 ][node_id] * x2_00_x1_10 * x0_00;
            y += W[9 ][node_id] * x2_00_x1_10 * x0_01;
            y += W[10][node_id] * x2_00_x1_10 * x0_10;
            y += W[11][node_id] * x2_00_x1_10 * x0_11;
            T x2_00_x1_11 = x2_00 * x1_11;
            y += W[12][node_id] * x2_00_x1_11 * x0_00;
            y += W[13][node_id] * x2_00_x1_11 * x0_01;
            y += W[14][node_id] * x2_00_x1_11 * x0_10;
            y += W[15][node_id] * x2_00_x1_11 * x0_11;
            T x2_01_x1_00 = x2_01 * x1_00;
            y += W[16][node_id] * x2_01_x1_00 * x0_00;
            y += W[17][node_id] * x2_01_x1_00 * x0_01;
            y += W[18][node_id] * x2_01_x1_00 * x0_10;
            y += W[19][node_id] * x2_01_x1_00 * x0_11;
            T x2_01_x1_01 = x2_01 * x1_01;
            y += W[20][node_id] * x2_01_x1_01 * x0_00;
            y += W[21][node_id] * x2_01_x1_01 * x0_01;
            y += W[22][node_id] * x2_01_x1_01 * x0_10;
            y += W[23][node_id] * x2_01_x1_01 * x0_11;
            T x2_01_x1_10 = x2_01 * x1_10;
            y += W[24][node_id] * x2_01_x1_10 * x0_00;
            y += W[25][node_id] * x2_01_x1_10 * x0_01;
            y += W[26][node_id] * x2_01_x1_10 * x0_10;
            y += W[27][node_id] * x2_01_x1_10 * x0_11;
            T x2_01_x1_11 = x2_01 * x1_11;
            y += W[28][node_id] * x2_01_x1_11 * x0_00;
            y += W[29][node_id] * x2_01_x1_11 * x0_01;
            y += W[30][node_id] * x2_01_x1_11 * x0_10;
            y += W[31][node_id] * x2_01_x1_11 * x0_11;
            T x2_10_x1_00 = x2_10 * x1_00;
            y += W[32][node_id] * x2_10_x1_00 * x0_00;
            y += W[33][node_id] * x2_10_x1_00 * x0_01;
            y += W[34][node_id] * x2_10_x1_00 * x0_10;
            y += W[35][node_id] * x2_10_x1_00 * x0_11;
            T x2_10_x1_01 = x2_10 * x1_01;
            y += W[36][node_id] * x2_10_x1_01 * x0_00;
            y += W[37][node_id] * x2_10_x1_01 * x0_01;
            y += W[38][node_id] * x2_10_x1_01 * x0_10;
            y += W[39][node_id] * x2_10_x1_01 * x0_11;
            T x2_10_x1_10 = x2_10 * x1_10;
            y += W[40][node_id] * x2_10_x1_10 * x0_00;
            y += W[41][node_id] * x2_10_x1_10 * x0_01;
            y += W[42][node_id] * x2_10_x1_10 * x0_10;
            y += W[43][node_id] * x2_10_x1_10 * x0_11;
            T x2_10_x1_11 = x2_10 * x1_11;
            y += W[44][node_id] * x2_10_x1_11 * x0_00;
            y += W[45][node_id] * x2_10_x1_11 * x0_01;
            y += W[46][node_id] * x2_10_x1_11 * x0_10;
            y += W[47][node_id] * x2_10_x1_11 * x0_11;
            T x2_11_x1_00 = x2_11 * x1_00;
            y += W[48][node_id] * x2_11_x1_00 * x0_00;
            y += W[49][node_id] * x2_11_x1_00 * x0_01;
            y += W[50][node_id] * x2_11_x1_00 * x0_10;
            y += W[51][node_id] * x2_11_x1_00 * x0_11;
            T x2_11_x1_01 = x2_11 * x1_01;
            y += W[52][node_id] * x2_11_x1_01 * x0_00;
            y += W[53][node_id] * x2_11_x1_01 * x0_01;
            y += W[54][node_id] * x2_11_x1_01 * x0_10;
            y += W[55][node_id] * x2_11_x1_01 * x0_11;
            T x2_11_x1_10 = x2_11 * x1_10;
            y += W[56][node_id] * x2_11_x1_10 * x0_00;
            y += W[57][node_id] * x2_11_x1_10 * x0_01;
            y += W[58][node_id] * x2_11_x1_10 * x0_10;
            y += W[59][node_id] * x2_11_x1_10 * x0_11;
            T x2_11_x1_11 = x2_11 * x1_11;
            y += W[60][node_id] * x2_11_x1_11 * x0_00;
            y += W[61][node_id] * x2_11_x1_11 * x0_01;
            y += W[62][node_id] * x2_11_x1_11 * x0_10;
            y += W[63][node_id] * x2_11_x1_11 * x0_11;
#endif

            // clamp
            y = max(0.0, y);
            y = min(1.0, y);
        
            y_ptr[frame] = y;
        }
    }
}


int bbcu_fp32_StochasticLut6_Forward
        (
            const float     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             input_binary,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 512;
    unsigned int const MAX_NODE_UNIT  = 64;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_StochasticLut_Forward<6, float, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            node_size,
            frame_size,
            frame_stride,
            input_binary,
            lut_binarize,
            unbinarize_bias
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



// bit packing
template<int N=6, typename T=float, int MAX_NODE_UNIT=32>
__global__ void kernal_bit_StochasticLut_Forward(
            int const   *x_buf,
            T           *y_buf,
            int const   *input_index,
            T   const   *W_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         bin_frame_stride,
            int         binary_mode,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__ T    W[(1 << N)][MAX_NODE_UNIT];
    int   const     *x_ptr[N];
    T               *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( binary_mode ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < N; ++i ) {
            x_ptr[i] = &x_buf[bin_frame_stride * input_index[N*node + i]];
        }

        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            int bit_mask = (1 << (frame & 0x1f));
            int unit     = (frame >> 5);
            
            T   x[N];
            for ( int i = 0; i < N; ++i) {
                x[i] = 0.5 + ((x_ptr[i][unit] & bit_mask) ? +unbinarize_bias : -unbinarize_bias);
            }

            T   y = StochasticLut<N, T, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

#if 0
            T   xp[N], xn[N];
            for ( int i = 0; i < N; ++i) {
                T x_val =  0.5 + ((x_ptr[i][unit] & bit) ? +unbinarize_bias : -unbinarize_bias);
                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            T x0_00 = xn[1] * xn[0];
            T x0_01 = xn[1] * xp[0];
            T x0_10 = xp[1] * xn[0];
            T x0_11 = xp[1] * xp[0];
            T x1_00 = xn[3] * xn[2];
            T x1_01 = xn[3] * xp[2];
            T x1_10 = xp[3] * xn[2];
            T x1_11 = xp[3] * xp[2];
            T x2_00 = xn[5] * xn[4];
            T x2_01 = xn[5] * xp[4];
            T x2_10 = xp[5] * xn[4];
            T x2_11 = xp[5] * xp[4];

            T y = 0;
            T x2_00_x1_00 = x2_00 * x1_00;
            y += W[0 ][node_id] * x2_00_x1_00 * x0_00;
            y += W[1 ][node_id] * x2_00_x1_00 * x0_01;
            y += W[2 ][node_id] * x2_00_x1_00 * x0_10;
            y += W[3 ][node_id] * x2_00_x1_00 * x0_11;
            T x2_00_x1_01 = x2_00 * x1_01;
            y += W[4 ][node_id] * x2_00_x1_01 * x0_00;
            y += W[5 ][node_id] * x2_00_x1_01 * x0_01;
            y += W[6 ][node_id] * x2_00_x1_01 * x0_10;
            y += W[7 ][node_id] * x2_00_x1_01 * x0_11;
            T x2_00_x1_10 = x2_00 * x1_10;
            y += W[8 ][node_id] * x2_00_x1_10 * x0_00;
            y += W[9 ][node_id] * x2_00_x1_10 * x0_01;
            y += W[10][node_id] * x2_00_x1_10 * x0_10;
            y += W[11][node_id] * x2_00_x1_10 * x0_11;
            T x2_00_x1_11 = x2_00 * x1_11;
            y += W[12][node_id] * x2_00_x1_11 * x0_00;
            y += W[13][node_id] * x2_00_x1_11 * x0_01;
            y += W[14][node_id] * x2_00_x1_11 * x0_10;
            y += W[15][node_id] * x2_00_x1_11 * x0_11;
            T x2_01_x1_00 = x2_01 * x1_00;
            y += W[16][node_id] * x2_01_x1_00 * x0_00;
            y += W[17][node_id] * x2_01_x1_00 * x0_01;
            y += W[18][node_id] * x2_01_x1_00 * x0_10;
            y += W[19][node_id] * x2_01_x1_00 * x0_11;
            T x2_01_x1_01 = x2_01 * x1_01;
            y += W[20][node_id] * x2_01_x1_01 * x0_00;
            y += W[21][node_id] * x2_01_x1_01 * x0_01;
            y += W[22][node_id] * x2_01_x1_01 * x0_10;
            y += W[23][node_id] * x2_01_x1_01 * x0_11;
            T x2_01_x1_10 = x2_01 * x1_10;
            y += W[24][node_id] * x2_01_x1_10 * x0_00;
            y += W[25][node_id] * x2_01_x1_10 * x0_01;
            y += W[26][node_id] * x2_01_x1_10 * x0_10;
            y += W[27][node_id] * x2_01_x1_10 * x0_11;
            T x2_01_x1_11 = x2_01 * x1_11;
            y += W[28][node_id] * x2_01_x1_11 * x0_00;
            y += W[29][node_id] * x2_01_x1_11 * x0_01;
            y += W[30][node_id] * x2_01_x1_11 * x0_10;
            y += W[31][node_id] * x2_01_x1_11 * x0_11;
            T x2_10_x1_00 = x2_10 * x1_00;
            y += W[32][node_id] * x2_10_x1_00 * x0_00;
            y += W[33][node_id] * x2_10_x1_00 * x0_01;
            y += W[34][node_id] * x2_10_x1_00 * x0_10;
            y += W[35][node_id] * x2_10_x1_00 * x0_11;
            T x2_10_x1_01 = x2_10 * x1_01;
            y += W[36][node_id] * x2_10_x1_01 * x0_00;
            y += W[37][node_id] * x2_10_x1_01 * x0_01;
            y += W[38][node_id] * x2_10_x1_01 * x0_10;
            y += W[39][node_id] * x2_10_x1_01 * x0_11;
            T x2_10_x1_10 = x2_10 * x1_10;
            y += W[40][node_id] * x2_10_x1_10 * x0_00;
            y += W[41][node_id] * x2_10_x1_10 * x0_01;
            y += W[42][node_id] * x2_10_x1_10 * x0_10;
            y += W[43][node_id] * x2_10_x1_10 * x0_11;
            T x2_10_x1_11 = x2_10 * x1_11;
            y += W[44][node_id] * x2_10_x1_11 * x0_00;
            y += W[45][node_id] * x2_10_x1_11 * x0_01;
            y += W[46][node_id] * x2_10_x1_11 * x0_10;
            y += W[47][node_id] * x2_10_x1_11 * x0_11;
            T x2_11_x1_00 = x2_11 * x1_00;
            y += W[48][node_id] * x2_11_x1_00 * x0_00;
            y += W[49][node_id] * x2_11_x1_00 * x0_01;
            y += W[50][node_id] * x2_11_x1_00 * x0_10;
            y += W[51][node_id] * x2_11_x1_00 * x0_11;
            T x2_11_x1_01 = x2_11 * x1_01;
            y += W[52][node_id] * x2_11_x1_01 * x0_00;
            y += W[53][node_id] * x2_11_x1_01 * x0_01;
            y += W[54][node_id] * x2_11_x1_01 * x0_10;
            y += W[55][node_id] * x2_11_x1_01 * x0_11;
            T x2_11_x1_10 = x2_11 * x1_10;
            y += W[56][node_id] * x2_11_x1_10 * x0_00;
            y += W[57][node_id] * x2_11_x1_10 * x0_01;
            y += W[58][node_id] * x2_11_x1_10 * x0_10;
            y += W[59][node_id] * x2_11_x1_10 * x0_11;
            T x2_11_x1_11 = x2_11 * x1_11;
            y += W[60][node_id] * x2_11_x1_11 * x0_00;
            y += W[61][node_id] * x2_11_x1_11 * x0_01;
            y += W[62][node_id] * x2_11_x1_11 * x0_10;
            y += W[63][node_id] * x2_11_x1_11 * x0_11;
#endif

            // clamp
            y = max(0.0, y);
            y = min(1.0, y);

            y_ptr[frame] = y;
        }
    }
}


int bbcu_bit_fp32_StochasticLut6_Forward
        (
            int   const     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 512;
    unsigned int const MAX_NODE_UNIT  = 64;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_bit_StochasticLut_Forward<6, float, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            node_size,
            frame_size,
            frame_stride,
            bin_frame_stride,
            lut_binarize,
            unbinarize_bias
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}




// -------------------------------------------------
//  Backward
// -------------------------------------------------

// real type
template<int N=6, typename T=float, int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_StochasticLut_Backward
        (
            T   const   *x_buf,
            T   const   *dy_buf,
            T           *dx_buf,
            int const   *input_index,
            T   const   *W_buf,
            T           *dW_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         input_binary,
            int         lut_binarize,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  T       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  T       dW_prev[(1 << N)][MAX_NODE_UNIT];
    __shared__  T       W[(1 << N)][MAX_NODE_UNIT];
                T       dW[(1 << N)];
                T const *x_ptr[6];
                T const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < (1 << N); ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < (1 << N); i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * (1 << N) + i];
        }

        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
    
        // init pointer
        for ( int i = 0; i < 6; ++i ) {
            int input_node = input_index[6*node + i];
            x_ptr[i]  = &x_buf[input_node * frame_stride];
        }

        dy_ptr = &dy_buf[node * frame_stride];
    }

    __syncthreads();

    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            // read x
            T   x[N];
            if ( input_binary ) {
                for ( int i = 0; i < N; ++i) {
                    x[i] = 0.5 +((x_ptr[i][frame] > 0.5)  ? +unbinarize_bias : -unbinarize_bias);
                }
            }
            else {
                for ( int i = 0; i < N; ++i) {
                    x[i] = max(0.0, min(1.0, x_ptr[i][frame]));
                }
            }

            // read dy
            T   dy = dy_ptr[frame];

            // calc
            StochasticLut<N, T, MAX_NODE_UNIT>::NodeBackward(node_id, x, dy, &dx_buf[node*N*frame_stride + frame], W, dW, frame_stride);
#if 0
            T xp[6], xn[6];
            for ( int i = 0; i < 6; ++i) {
                T x_val = x_ptr[i][frame];
                if ( input_binary ) {
                    x_val = 0.5 + ((x_val > 0.5) ? +unbinarize_bias : -unbinarize_bias);
                }
                else {
                    x_val = min(1.0, max(0.0, x_val));
                }

                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            T x0_00 = xn[1] * xn[0];
            T x0_01 = xn[1] * xp[0];
            T x0_10 = xp[1] * xn[0];
            T x0_11 = xp[1] * xp[0];
            T x1_00 = xn[3] * xn[2];
            T x1_01 = xn[3] * xp[2];
            T x1_10 = xp[3] * xn[2];
            T x1_11 = xp[3] * xp[2];
            T x2_00 = xn[5] * xn[4];
            T x2_01 = xn[5] * xp[4];
            T x2_10 = xp[5] * xn[4];
            T x2_11 = xp[5] * xp[4];

            T grad = dy_ptr[frame];

            T  x2_00_x1_00 =  x2_00 * x1_00;
            T  x2_00_x1_01 =  x2_00 * x1_01;
            T  x2_00_x1_10 =  x2_00 * x1_10;
            T  x2_00_x1_11 =  x2_00 * x1_11;
            T  x2_01_x1_00 =  x2_01 * x1_00;
            T  x2_01_x1_01 =  x2_01 * x1_01;
            T  x2_01_x1_10 =  x2_01 * x1_10;
            T  x2_01_x1_11 =  x2_01 * x1_11;
            T  x2_10_x1_00 =  x2_10 * x1_00;
            T  x2_10_x1_01 =  x2_10 * x1_01;
            T  x2_10_x1_10 =  x2_10 * x1_10;
            T  x2_10_x1_11 =  x2_10 * x1_11;
            T  x2_11_x1_00 =  x2_11 * x1_00;
            T  x2_11_x1_01 =  x2_11 * x1_01;
            T  x2_11_x1_10 =  x2_11 * x1_10;
            T  x2_11_x1_11 =  x2_11 * x1_11;

            dW[ 0] += x2_00_x1_00 * x0_00 * grad;
            dW[ 1] += x2_00_x1_00 * x0_01 * grad;
            dW[ 2] += x2_00_x1_00 * x0_10 * grad;
            dW[ 3] += x2_00_x1_00 * x0_11 * grad;
            dW[ 4] += x2_00_x1_01 * x0_00 * grad;
            dW[ 5] += x2_00_x1_01 * x0_01 * grad;
            dW[ 6] += x2_00_x1_01 * x0_10 * grad;
            dW[ 7] += x2_00_x1_01 * x0_11 * grad;
            dW[ 8] += x2_00_x1_10 * x0_00 * grad;
            dW[ 9] += x2_00_x1_10 * x0_01 * grad;
            dW[10] += x2_00_x1_10 * x0_10 * grad;
            dW[11] += x2_00_x1_10 * x0_11 * grad;
            dW[12] += x2_00_x1_11 * x0_00 * grad;
            dW[13] += x2_00_x1_11 * x0_01 * grad;
            dW[14] += x2_00_x1_11 * x0_10 * grad;
            dW[15] += x2_00_x1_11 * x0_11 * grad;
            dW[16] += x2_01_x1_00 * x0_00 * grad;
            dW[17] += x2_01_x1_00 * x0_01 * grad;
            dW[18] += x2_01_x1_00 * x0_10 * grad;
            dW[19] += x2_01_x1_00 * x0_11 * grad;
            dW[20] += x2_01_x1_01 * x0_00 * grad;
            dW[21] += x2_01_x1_01 * x0_01 * grad;
            dW[22] += x2_01_x1_01 * x0_10 * grad;
            dW[23] += x2_01_x1_01 * x0_11 * grad;
            dW[24] += x2_01_x1_10 * x0_00 * grad;
            dW[25] += x2_01_x1_10 * x0_01 * grad;
            dW[26] += x2_01_x1_10 * x0_10 * grad;
            dW[27] += x2_01_x1_10 * x0_11 * grad;
            dW[28] += x2_01_x1_11 * x0_00 * grad;
            dW[29] += x2_01_x1_11 * x0_01 * grad;
            dW[30] += x2_01_x1_11 * x0_10 * grad;
            dW[31] += x2_01_x1_11 * x0_11 * grad;
            dW[32] += x2_10_x1_00 * x0_00 * grad;
            dW[33] += x2_10_x1_00 * x0_01 * grad;
            dW[34] += x2_10_x1_00 * x0_10 * grad;
            dW[35] += x2_10_x1_00 * x0_11 * grad;
            dW[36] += x2_10_x1_01 * x0_00 * grad;
            dW[37] += x2_10_x1_01 * x0_01 * grad;
            dW[38] += x2_10_x1_01 * x0_10 * grad;
            dW[39] += x2_10_x1_01 * x0_11 * grad;
            dW[40] += x2_10_x1_10 * x0_00 * grad;
            dW[41] += x2_10_x1_10 * x0_01 * grad;
            dW[42] += x2_10_x1_10 * x0_10 * grad;
            dW[43] += x2_10_x1_10 * x0_11 * grad;
            dW[44] += x2_10_x1_11 * x0_00 * grad;
            dW[45] += x2_10_x1_11 * x0_01 * grad;
            dW[46] += x2_10_x1_11 * x0_10 * grad;
            dW[47] += x2_10_x1_11 * x0_11 * grad;
            dW[48] += x2_11_x1_00 * x0_00 * grad;
            dW[49] += x2_11_x1_00 * x0_01 * grad;
            dW[50] += x2_11_x1_00 * x0_10 * grad;
            dW[51] += x2_11_x1_00 * x0_11 * grad;
            dW[52] += x2_11_x1_01 * x0_00 * grad;
            dW[53] += x2_11_x1_01 * x0_01 * grad;
            dW[54] += x2_11_x1_01 * x0_10 * grad;
            dW[55] += x2_11_x1_01 * x0_11 * grad;
            dW[56] += x2_11_x1_10 * x0_00 * grad;
            dW[57] += x2_11_x1_10 * x0_01 * grad;
            dW[58] += x2_11_x1_10 * x0_10 * grad;
            dW[59] += x2_11_x1_10 * x0_11 * grad;
            dW[60] += x2_11_x1_11 * x0_00 * grad;
            dW[61] += x2_11_x1_11 * x0_01 * grad;
            dW[62] += x2_11_x1_11 * x0_10 * grad;
            dW[63] += x2_11_x1_11 * x0_11 * grad;

            T  x2_00_x0_00 =  x2_00 * x0_00;
            T  x2_00_x0_01 =  x2_00 * x0_01;
            T  x2_00_x0_10 =  x2_00 * x0_10;
            T  x2_00_x0_11 =  x2_00 * x0_11;
            T  x2_01_x0_00 =  x2_01 * x0_00;
            T  x2_01_x0_01 =  x2_01 * x0_01;
            T  x2_01_x0_10 =  x2_01 * x0_10;
            T  x2_01_x0_11 =  x2_01 * x0_11;
            T  x2_10_x0_00 =  x2_10 * x0_00;
            T  x2_10_x0_01 =  x2_10 * x0_01;
            T  x2_10_x0_10 =  x2_10 * x0_10;
            T  x2_10_x0_11 =  x2_10 * x0_11;
            T  x2_11_x0_00 =  x2_11 * x0_00;
            T  x2_11_x0_01 =  x2_11 * x0_01;
            T  x2_11_x0_10 =  x2_11 * x0_10;
            T  x2_11_x0_11 =  x2_11 * x0_11;

            T  x1_00_x0_00 =  x1_00 * x0_00;
            T  x1_00_x0_01 =  x1_00 * x0_01;
            T  x1_00_x0_10 =  x1_00 * x0_10;
            T  x1_00_x0_11 =  x1_00 * x0_11;
            T  x1_01_x0_00 =  x1_01 * x0_00;
            T  x1_01_x0_01 =  x1_01 * x0_01;
            T  x1_01_x0_10 =  x1_01 * x0_10;
            T  x1_01_x0_11 =  x1_01 * x0_11;
            T  x1_10_x0_00 =  x1_10 * x0_00;
            T  x1_10_x0_01 =  x1_10 * x0_01;
            T  x1_10_x0_10 =  x1_10 * x0_10;
            T  x1_10_x0_11 =  x1_10 * x0_11;
            T  x1_11_x0_00 =  x1_11 * x0_00;
            T  x1_11_x0_01 =  x1_11 * x0_01;
            T  x1_11_x0_10 =  x1_11 * x0_10;
            T  x1_11_x0_11 =  x1_11 * x0_11;


            T dxi;
            T dx0_00 = 0;
            T dx0_01 = 0;
            T dx0_10 = 0;
            T dx0_11 = 0;
            T dx1_00 = 0;
            T dx1_01 = 0;
            T dx1_10 = 0;
            T dx1_11 = 0;
            T dx2_00 = 0;
            T dx2_01 = 0;
            T dx2_10 = 0;
            T dx2_11 = 0;
            dxi = W[ 0][node_id];  dx0_00 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_00_x0_00;
            dxi = W[ 1][node_id];  dx0_01 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_00_x0_01;
            dxi = W[ 2][node_id];  dx0_10 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_00_x0_10;
            dxi = W[ 3][node_id];  dx0_11 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_00_x0_11;
            dxi = W[ 4][node_id];  dx0_00 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_01_x0_00;
            dxi = W[ 5][node_id];  dx0_01 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_01_x0_01;
            dxi = W[ 6][node_id];  dx0_10 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_01_x0_10;
            dxi = W[ 7][node_id];  dx0_11 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_01_x0_11;
            dxi = W[ 8][node_id];  dx0_00 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_10_x0_00;
            dxi = W[ 9][node_id];  dx0_01 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_10_x0_01;
            dxi = W[10][node_id];  dx0_10 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_10_x0_10;
            dxi = W[11][node_id];  dx0_11 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_10_x0_11;
            dxi = W[12][node_id];  dx0_00 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_11_x0_00;
            dxi = W[13][node_id];  dx0_01 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_11_x0_01;
            dxi = W[14][node_id];  dx0_10 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_11_x0_10;
            dxi = W[15][node_id];  dx0_11 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_11_x0_11;
            dxi = W[16][node_id];  dx0_00 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_00_x0_00;
            dxi = W[17][node_id];  dx0_01 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_00_x0_01;
            dxi = W[18][node_id];  dx0_10 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_00_x0_10;
            dxi = W[19][node_id];  dx0_11 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_00_x0_11;
            dxi = W[20][node_id];  dx0_00 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_01_x0_00;
            dxi = W[21][node_id];  dx0_01 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_01_x0_01;
            dxi = W[22][node_id];  dx0_10 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_01_x0_10;
            dxi = W[23][node_id];  dx0_11 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_01_x0_11;
            dxi = W[24][node_id];  dx0_00 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_10_x0_00;
            dxi = W[25][node_id];  dx0_01 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_10_x0_01;
            dxi = W[26][node_id];  dx0_10 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_10_x0_10;
            dxi = W[27][node_id];  dx0_11 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_10_x0_11;
            dxi = W[28][node_id];  dx0_00 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_11_x0_00;
            dxi = W[29][node_id];  dx0_01 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_11_x0_01;
            dxi = W[30][node_id];  dx0_10 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_11_x0_10;
            dxi = W[31][node_id];  dx0_11 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_11_x0_11;
            dxi = W[32][node_id];  dx0_00 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_00_x0_00;
            dxi = W[33][node_id];  dx0_01 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_00_x0_01;
            dxi = W[34][node_id];  dx0_10 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_00_x0_10;
            dxi = W[35][node_id];  dx0_11 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_00_x0_11;
            dxi = W[36][node_id];  dx0_00 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_01_x0_00;
            dxi = W[37][node_id];  dx0_01 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_01_x0_01;
            dxi = W[38][node_id];  dx0_10 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_01_x0_10;
            dxi = W[39][node_id];  dx0_11 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_01_x0_11;
            dxi = W[40][node_id];  dx0_00 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_10_x0_00;
            dxi = W[41][node_id];  dx0_01 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_10_x0_01;
            dxi = W[42][node_id];  dx0_10 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_10_x0_10;
            dxi = W[43][node_id];  dx0_11 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_10_x0_11;
            dxi = W[44][node_id];  dx0_00 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_11_x0_00;
            dxi = W[45][node_id];  dx0_01 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_11_x0_01;
            dxi = W[46][node_id];  dx0_10 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_11_x0_10;
            dxi = W[47][node_id];  dx0_11 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_11_x0_11;
            dxi = W[48][node_id];  dx0_00 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_00_x0_00;
            dxi = W[49][node_id];  dx0_01 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_00_x0_01;
            dxi = W[50][node_id];  dx0_10 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_00_x0_10;
            dxi = W[51][node_id];  dx0_11 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_00_x0_11;
            dxi = W[52][node_id];  dx0_00 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_01_x0_00;
            dxi = W[53][node_id];  dx0_01 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_01_x0_01;
            dxi = W[54][node_id];  dx0_10 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_01_x0_10;
            dxi = W[55][node_id];  dx0_11 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_01_x0_11;
            dxi = W[56][node_id];  dx0_00 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_10_x0_00;
            dxi = W[57][node_id];  dx0_01 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_10_x0_01;
            dxi = W[58][node_id];  dx0_10 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_10_x0_10;
            dxi = W[59][node_id];  dx0_11 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_10_x0_11;
            dxi = W[60][node_id];  dx0_00 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_11_x0_00;
            dxi = W[61][node_id];  dx0_01 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_11_x0_01;
            dxi = W[62][node_id];  dx0_10 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_11_x0_10;
            dxi = W[63][node_id];  dx0_11 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_11_x0_11;
        
            T *dx_ptr = &dx_buf[(node*6)*frame_stride + frame];
            T dxn;
            T dxp;
            T dx;
            dxn  = dx0_00 * xn[1];    dxn += dx0_10 * xp[1];
            dxp  = dx0_01 * xn[1];    dxp += dx0_11 * xp[1];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[0 * frame_stride] = dx;

            dxn  = dx0_00 * xn[0];
            dxn += dx0_01 * xp[0];
            dxp  = dx0_10 * xn[0];
            dxp += dx0_11 * xp[0];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[1 * frame_stride] = dx;

            dxn  = dx1_00 * xn[3];     
            dxp  = dx1_01 * xn[3];     
            dxn += dx1_10 * xp[3];     
            dxp += dx1_11 * xp[3];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[2 * frame_stride] = dx;

            dxn  = dx1_00 * xn[2];
            dxn += dx1_01 * xp[2];
            dxp  = dx1_10 * xn[2];
            dxp += dx1_11 * xp[2];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[3 * frame_stride] = dx;

            dxn  = dx2_00 * xn[5];     
            dxp  = dx2_01 * xn[5];     
            dxn += dx2_10 * xp[5];     
            dxp += dx2_11 * xp[5];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[4 * frame_stride] = dx;

            dxn  = dx2_00 * xn[4];
            dxn += dx2_01 * xp[4];
            dxp  = dx2_10 * xn[4];
            dxp += dx2_11 * xp[4];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[5 * frame_stride] = dx;
#endif
        }
    }

    for ( int i = 0; i < (1 << N); ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < (1 << N); ++i) {
                dW_buf[node*(1 << N) + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}

/*
__global__ void kernal_fp32_StochasticLut6_BackwardMarge(
            const float*    src_buf,
            float*          dst_buf,
            const int*      input_index,
            int             node_size,
            int             frame_size,
            int             frame_stride
        )
{
    int frame = blockDim.x * blockIdx.x + threadIdx.x;

    for ( int node = 0; node < node_size; ++node ) {
        if ( frame < frame_size ) {
            for ( int n = 0; n < 6; ++n ) {
                int in_idx = input_index[node*6 + n];
                float*       dst_buf_ptr = &dst_buf[frame_stride * in_idx];
                float        prev_data   = dst_buf_ptr[frame];
                const float* src_buf_ptr = &src_buf[(6 * node + n) * frame_stride];
                
                dst_buf_ptr[frame] = prev_data + src_buf_ptr[frame];
            }
        }
        __syncthreads();
    }
}
*/

int bbcu_fp32_StochasticLut6_Backward(
            float const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_W,
            float           *dev_dW,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             input_binary,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
    )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

        kernal_StochasticLut_Backward<6, float, MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_W,
                dev_dW,
                output_node_size,
                frame_size,
                frame_stride,
                input_binary,
                lut_binarize,
                unbinarize_bias
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = frame_size;
        while ( block_x > 1024 ) { block_x /= 2; }

        dim3    grid((frame_size + block_x - 1) /block_x, 1);
        dim3    block(block_x, 1, 1);

        /*
        kernal_fp32_StochasticLut6_BackwardMarge<<<grid, block>>>(
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        */

        kernal_NodeIntegrate<6, float><<<grid, block>>>(
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}



// bit packing
template<int N=6, typename T=float, int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_bit_StochasticLut_Backward
        (
            int const   *x_buf,
            T   const   *dy_buf,
            T           *dx_buf,
            int const   *input_index,
            T   const   *W_buf,
            T           *dW_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         bin_frame_stride,
            int         lut_binarize,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  T       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  T       dW_prev[(1 << N)][MAX_NODE_UNIT];
    __shared__  T       W[(1 << N)][MAX_NODE_UNIT];
                T       dW[(1 << N)];
                int   const *x_ptr[N];
                T const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < (1 << N); ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < (1 << N); i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * (1 << N) + i];
        }

        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // init pointer
        for ( int i = 0; i < N; ++i ) {
            int input_node = input_index[N*node + i];
            x_ptr[i]  = &x_buf[input_node * bin_frame_stride];
        }

        dy_ptr = &dy_buf[node * frame_stride];
    }

    __syncthreads();

    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);

            // read x
            T   x[N];
            for ( int i = 0; i < N; ++i) {
                x[i] = 0.5 +((x_ptr[i][unit] & bit) ? +unbinarize_bias : -unbinarize_bias);
            }

            // read dy
            T   dy = dy_ptr[frame];

            // calc
            StochasticLut<N, T, MAX_NODE_UNIT>::NodeBackward(node_id, x, dy, &dx_buf[node*N*frame_stride + frame], W, dW, frame_stride);

#if 0
            T   xp[6], xn[6];
            for ( int i = 0; i < 6; ++i) {
                T x_val = 0.5 + ((x_ptr[i][unit] & bit) ? +unbinarize_bias : -unbinarize_bias);
                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            T x0_00 = xn[1] * xn[0];
            T x0_01 = xn[1] * xp[0];
            T x0_10 = xp[1] * xn[0];
            T x0_11 = xp[1] * xp[0];
            T x1_00 = xn[3] * xn[2];
            T x1_01 = xn[3] * xp[2];
            T x1_10 = xp[3] * xn[2];
            T x1_11 = xp[3] * xp[2];
            T x2_00 = xn[5] * xn[4];
            T x2_01 = xn[5] * xp[4];
            T x2_10 = xp[5] * xn[4];
            T x2_11 = xp[5] * xp[4];

            T grad = dy_ptr[frame];

            T  x2_00_x1_00 =  x2_00 * x1_00;
            T  x2_00_x1_01 =  x2_00 * x1_01;
            T  x2_00_x1_10 =  x2_00 * x1_10;
            T  x2_00_x1_11 =  x2_00 * x1_11;
            T  x2_01_x1_00 =  x2_01 * x1_00;
            T  x2_01_x1_01 =  x2_01 * x1_01;
            T  x2_01_x1_10 =  x2_01 * x1_10;
            T  x2_01_x1_11 =  x2_01 * x1_11;
            T  x2_10_x1_00 =  x2_10 * x1_00;
            T  x2_10_x1_01 =  x2_10 * x1_01;
            T  x2_10_x1_10 =  x2_10 * x1_10;
            T  x2_10_x1_11 =  x2_10 * x1_11;
            T  x2_11_x1_00 =  x2_11 * x1_00;
            T  x2_11_x1_01 =  x2_11 * x1_01;
            T  x2_11_x1_10 =  x2_11 * x1_10;
            T  x2_11_x1_11 =  x2_11 * x1_11;

            dW[ 0] += x2_00_x1_00 * x0_00 * grad;
            dW[ 1] += x2_00_x1_00 * x0_01 * grad;
            dW[ 2] += x2_00_x1_00 * x0_10 * grad;
            dW[ 3] += x2_00_x1_00 * x0_11 * grad;
            dW[ 4] += x2_00_x1_01 * x0_00 * grad;
            dW[ 5] += x2_00_x1_01 * x0_01 * grad;
            dW[ 6] += x2_00_x1_01 * x0_10 * grad;
            dW[ 7] += x2_00_x1_01 * x0_11 * grad;
            dW[ 8] += x2_00_x1_10 * x0_00 * grad;
            dW[ 9] += x2_00_x1_10 * x0_01 * grad;
            dW[10] += x2_00_x1_10 * x0_10 * grad;
            dW[11] += x2_00_x1_10 * x0_11 * grad;
            dW[12] += x2_00_x1_11 * x0_00 * grad;
            dW[13] += x2_00_x1_11 * x0_01 * grad;
            dW[14] += x2_00_x1_11 * x0_10 * grad;
            dW[15] += x2_00_x1_11 * x0_11 * grad;
            dW[16] += x2_01_x1_00 * x0_00 * grad;
            dW[17] += x2_01_x1_00 * x0_01 * grad;
            dW[18] += x2_01_x1_00 * x0_10 * grad;
            dW[19] += x2_01_x1_00 * x0_11 * grad;
            dW[20] += x2_01_x1_01 * x0_00 * grad;
            dW[21] += x2_01_x1_01 * x0_01 * grad;
            dW[22] += x2_01_x1_01 * x0_10 * grad;
            dW[23] += x2_01_x1_01 * x0_11 * grad;
            dW[24] += x2_01_x1_10 * x0_00 * grad;
            dW[25] += x2_01_x1_10 * x0_01 * grad;
            dW[26] += x2_01_x1_10 * x0_10 * grad;
            dW[27] += x2_01_x1_10 * x0_11 * grad;
            dW[28] += x2_01_x1_11 * x0_00 * grad;
            dW[29] += x2_01_x1_11 * x0_01 * grad;
            dW[30] += x2_01_x1_11 * x0_10 * grad;
            dW[31] += x2_01_x1_11 * x0_11 * grad;
            dW[32] += x2_10_x1_00 * x0_00 * grad;
            dW[33] += x2_10_x1_00 * x0_01 * grad;
            dW[34] += x2_10_x1_00 * x0_10 * grad;
            dW[35] += x2_10_x1_00 * x0_11 * grad;
            dW[36] += x2_10_x1_01 * x0_00 * grad;
            dW[37] += x2_10_x1_01 * x0_01 * grad;
            dW[38] += x2_10_x1_01 * x0_10 * grad;
            dW[39] += x2_10_x1_01 * x0_11 * grad;
            dW[40] += x2_10_x1_10 * x0_00 * grad;
            dW[41] += x2_10_x1_10 * x0_01 * grad;
            dW[42] += x2_10_x1_10 * x0_10 * grad;
            dW[43] += x2_10_x1_10 * x0_11 * grad;
            dW[44] += x2_10_x1_11 * x0_00 * grad;
            dW[45] += x2_10_x1_11 * x0_01 * grad;
            dW[46] += x2_10_x1_11 * x0_10 * grad;
            dW[47] += x2_10_x1_11 * x0_11 * grad;
            dW[48] += x2_11_x1_00 * x0_00 * grad;
            dW[49] += x2_11_x1_00 * x0_01 * grad;
            dW[50] += x2_11_x1_00 * x0_10 * grad;
            dW[51] += x2_11_x1_00 * x0_11 * grad;
            dW[52] += x2_11_x1_01 * x0_00 * grad;
            dW[53] += x2_11_x1_01 * x0_01 * grad;
            dW[54] += x2_11_x1_01 * x0_10 * grad;
            dW[55] += x2_11_x1_01 * x0_11 * grad;
            dW[56] += x2_11_x1_10 * x0_00 * grad;
            dW[57] += x2_11_x1_10 * x0_01 * grad;
            dW[58] += x2_11_x1_10 * x0_10 * grad;
            dW[59] += x2_11_x1_10 * x0_11 * grad;
            dW[60] += x2_11_x1_11 * x0_00 * grad;
            dW[61] += x2_11_x1_11 * x0_01 * grad;
            dW[62] += x2_11_x1_11 * x0_10 * grad;
            dW[63] += x2_11_x1_11 * x0_11 * grad;

            T  x2_00_x0_00 =  x2_00 * x0_00;
            T  x2_00_x0_01 =  x2_00 * x0_01;
            T  x2_00_x0_10 =  x2_00 * x0_10;
            T  x2_00_x0_11 =  x2_00 * x0_11;
            T  x2_01_x0_00 =  x2_01 * x0_00;
            T  x2_01_x0_01 =  x2_01 * x0_01;
            T  x2_01_x0_10 =  x2_01 * x0_10;
            T  x2_01_x0_11 =  x2_01 * x0_11;
            T  x2_10_x0_00 =  x2_10 * x0_00;
            T  x2_10_x0_01 =  x2_10 * x0_01;
            T  x2_10_x0_10 =  x2_10 * x0_10;
            T  x2_10_x0_11 =  x2_10 * x0_11;
            T  x2_11_x0_00 =  x2_11 * x0_00;
            T  x2_11_x0_01 =  x2_11 * x0_01;
            T  x2_11_x0_10 =  x2_11 * x0_10;
            T  x2_11_x0_11 =  x2_11 * x0_11;

            T  x1_00_x0_00 =  x1_00 * x0_00;
            T  x1_00_x0_01 =  x1_00 * x0_01;
            T  x1_00_x0_10 =  x1_00 * x0_10;
            T  x1_00_x0_11 =  x1_00 * x0_11;
            T  x1_01_x0_00 =  x1_01 * x0_00;
            T  x1_01_x0_01 =  x1_01 * x0_01;
            T  x1_01_x0_10 =  x1_01 * x0_10;
            T  x1_01_x0_11 =  x1_01 * x0_11;
            T  x1_10_x0_00 =  x1_10 * x0_00;
            T  x1_10_x0_01 =  x1_10 * x0_01;
            T  x1_10_x0_10 =  x1_10 * x0_10;
            T  x1_10_x0_11 =  x1_10 * x0_11;
            T  x1_11_x0_00 =  x1_11 * x0_00;
            T  x1_11_x0_01 =  x1_11 * x0_01;
            T  x1_11_x0_10 =  x1_11 * x0_10;
            T  x1_11_x0_11 =  x1_11 * x0_11;


            T dxi;
            T dx0_00 = 0;
            T dx0_01 = 0;
            T dx0_10 = 0;
            T dx0_11 = 0;
            T dx1_00 = 0;
            T dx1_01 = 0;
            T dx1_10 = 0;
            T dx1_11 = 0;
            T dx2_00 = 0;
            T dx2_01 = 0;
            T dx2_10 = 0;
            T dx2_11 = 0;
            dxi = W[ 0][node_id];  dx0_00 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_00_x0_00;
            dxi = W[ 1][node_id];  dx0_01 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_00_x0_01;
            dxi = W[ 2][node_id];  dx0_10 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_00_x0_10;
            dxi = W[ 3][node_id];  dx0_11 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_00_x0_11;
            dxi = W[ 4][node_id];  dx0_00 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_01_x0_00;
            dxi = W[ 5][node_id];  dx0_01 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_01_x0_01;
            dxi = W[ 6][node_id];  dx0_10 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_01_x0_10;
            dxi = W[ 7][node_id];  dx0_11 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_01_x0_11;
            dxi = W[ 8][node_id];  dx0_00 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_10_x0_00;
            dxi = W[ 9][node_id];  dx0_01 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_10_x0_01;
            dxi = W[10][node_id];  dx0_10 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_10_x0_10;
            dxi = W[11][node_id];  dx0_11 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_10_x0_11;
            dxi = W[12][node_id];  dx0_00 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_11_x0_00;
            dxi = W[13][node_id];  dx0_01 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_11_x0_01;
            dxi = W[14][node_id];  dx0_10 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_11_x0_10;
            dxi = W[15][node_id];  dx0_11 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_11_x0_11;
            dxi = W[16][node_id];  dx0_00 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_00_x0_00;
            dxi = W[17][node_id];  dx0_01 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_00_x0_01;
            dxi = W[18][node_id];  dx0_10 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_00_x0_10;
            dxi = W[19][node_id];  dx0_11 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_00_x0_11;
            dxi = W[20][node_id];  dx0_00 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_01_x0_00;
            dxi = W[21][node_id];  dx0_01 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_01_x0_01;
            dxi = W[22][node_id];  dx0_10 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_01_x0_10;
            dxi = W[23][node_id];  dx0_11 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_01_x0_11;
            dxi = W[24][node_id];  dx0_00 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_10_x0_00;
            dxi = W[25][node_id];  dx0_01 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_10_x0_01;
            dxi = W[26][node_id];  dx0_10 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_10_x0_10;
            dxi = W[27][node_id];  dx0_11 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_10_x0_11;
            dxi = W[28][node_id];  dx0_00 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_11_x0_00;
            dxi = W[29][node_id];  dx0_01 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_11_x0_01;
            dxi = W[30][node_id];  dx0_10 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_11_x0_10;
            dxi = W[31][node_id];  dx0_11 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_11_x0_11;
            dxi = W[32][node_id];  dx0_00 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_00_x0_00;
            dxi = W[33][node_id];  dx0_01 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_00_x0_01;
            dxi = W[34][node_id];  dx0_10 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_00_x0_10;
            dxi = W[35][node_id];  dx0_11 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_00_x0_11;
            dxi = W[36][node_id];  dx0_00 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_01_x0_00;
            dxi = W[37][node_id];  dx0_01 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_01_x0_01;
            dxi = W[38][node_id];  dx0_10 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_01_x0_10;
            dxi = W[39][node_id];  dx0_11 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_01_x0_11;
            dxi = W[40][node_id];  dx0_00 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_10_x0_00;
            dxi = W[41][node_id];  dx0_01 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_10_x0_01;
            dxi = W[42][node_id];  dx0_10 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_10_x0_10;
            dxi = W[43][node_id];  dx0_11 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_10_x0_11;
            dxi = W[44][node_id];  dx0_00 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_11_x0_00;
            dxi = W[45][node_id];  dx0_01 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_11_x0_01;
            dxi = W[46][node_id];  dx0_10 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_11_x0_10;
            dxi = W[47][node_id];  dx0_11 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_11_x0_11;
            dxi = W[48][node_id];  dx0_00 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_00_x0_00;
            dxi = W[49][node_id];  dx0_01 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_00_x0_01;
            dxi = W[50][node_id];  dx0_10 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_00_x0_10;
            dxi = W[51][node_id];  dx0_11 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_00_x0_11;
            dxi = W[52][node_id];  dx0_00 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_01_x0_00;
            dxi = W[53][node_id];  dx0_01 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_01_x0_01;
            dxi = W[54][node_id];  dx0_10 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_01_x0_10;
            dxi = W[55][node_id];  dx0_11 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_01_x0_11;
            dxi = W[56][node_id];  dx0_00 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_10_x0_00;
            dxi = W[57][node_id];  dx0_01 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_10_x0_01;
            dxi = W[58][node_id];  dx0_10 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_10_x0_10;
            dxi = W[59][node_id];  dx0_11 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_10_x0_11;
            dxi = W[60][node_id];  dx0_00 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_11_x0_00;
            dxi = W[61][node_id];  dx0_01 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_11_x0_01;
            dxi = W[62][node_id];  dx0_10 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_11_x0_10;
            dxi = W[63][node_id];  dx0_11 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_11_x0_11;
        
            T *dx_ptr = &dx_buf[(node*6)*frame_stride + frame];
            T dxn;
            T dxp;
            T dx;
            dxn  = dx0_00 * xn[1];    dxn += dx0_10 * xp[1];
            dxp  = dx0_01 * xn[1];    dxp += dx0_11 * xp[1];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[0 * frame_stride] = dx;

            dxn  = dx0_00 * xn[0];
            dxn += dx0_01 * xp[0];
            dxp  = dx0_10 * xn[0];
            dxp += dx0_11 * xp[0];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[1 * frame_stride] = dx;

            dxn  = dx1_00 * xn[3];     
            dxp  = dx1_01 * xn[3];     
            dxn += dx1_10 * xp[3];     
            dxp += dx1_11 * xp[3];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[2 * frame_stride] = dx;

            dxn  = dx1_00 * xn[2];
            dxn += dx1_01 * xp[2];
            dxp  = dx1_10 * xn[2];
            dxp += dx1_11 * xp[2];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[3 * frame_stride] = dx;

            dxn  = dx2_00 * xn[5];     
            dxp  = dx2_01 * xn[5];     
            dxn += dx2_10 * xp[5];     
            dxp += dx2_11 * xp[5];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[4 * frame_stride] = dx;

            dxn  = dx2_00 * xn[4];
            dxn += dx2_01 * xp[4];
            dxp  = dx2_10 * xn[4];
            dxp += dx2_11 * xp[4];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[5 * frame_stride] = dx;
#endif
        }
    }

    for ( int i = 0; i < (1 << N); ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < (1 << N); ++i) {
                dW_buf[node*(1 << N) + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}

int bbcu_bit_fp32_StochasticLut6_Backward(
            int   const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_W,
            float           *dev_dW,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
    )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

        kernal_bit_StochasticLut_Backward<6, float, MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_W,
                dev_dW,
                output_node_size,
                frame_size,
                frame_stride,
                bin_frame_stride,
                lut_binarize,
                unbinarize_bias
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = frame_size;
        while ( block_x > 1024 ) { block_x /= 2; }

        dim3    grid((frame_size + block_x - 1) /block_x, 1);
        dim3    block(block_x, 1, 1);
        /*
        kernal_fp32_StochasticLut6_BackwardMarge<<<grid, block>>>(
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        */
        
        kernal_NodeIntegrate<6, float><<<grid, block>>>(
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}




// end of file
