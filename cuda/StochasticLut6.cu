#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// -------------------------------------------------
//  Forward
// -------------------------------------------------


template<int MAX_NODE_UNIT=32>
__global__ void kernal_fp32_StochasticLut6_Forward(
            float const     *x_buf,
            float           *y_buf,
            int   const     *input_index,
            float const     *W_buf,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             input_binary,
            int             lut_binarize,
            float           param_min,
            float           param_max
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       W[64][MAX_NODE_UNIT];
                float const *x_ptr[6];
                float       *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < 6; ++i ) {
            x_ptr[i] = &x_buf[frame_stride * input_index[6*node + i]];
        }

        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            float   xp[6], xn[6];
            for ( int i = 0; i < 6; ++i) {
                float x_val = x_ptr[i][frame];
                if ( input_binary ) {
                    x_val = (x_val > 0.5) ? 0.7 : 0.3;
                }
                else {
                    x_val = min(1.0, max(0.0, x_val));
                }

                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            float x0_00 = xn[1] * xn[0];
            float x0_01 = xn[1] * xp[0];
            float x0_10 = xp[1] * xn[0];
            float x0_11 = xp[1] * xp[0];
            float x1_00 = xn[3] * xn[2];
            float x1_01 = xn[3] * xp[2];
            float x1_10 = xp[3] * xn[2];
            float x1_11 = xp[3] * xp[2];
            float x2_00 = xn[5] * xn[4];
            float x2_01 = xn[5] * xp[4];
            float x2_10 = xp[5] * xn[4];
            float x2_11 = xp[5] * xp[4];

            float y = 0;
            float x2_00_x1_00 = x2_00 * x1_00;
            y += W[0 ][node_id] * x2_00_x1_00 * x0_00;
            y += W[1 ][node_id] * x2_00_x1_00 * x0_01;
            y += W[2 ][node_id] * x2_00_x1_00 * x0_10;
            y += W[3 ][node_id] * x2_00_x1_00 * x0_11;
            float x2_00_x1_01 = x2_00 * x1_01;
            y += W[4 ][node_id] * x2_00_x1_01 * x0_00;
            y += W[5 ][node_id] * x2_00_x1_01 * x0_01;
            y += W[6 ][node_id] * x2_00_x1_01 * x0_10;
            y += W[7 ][node_id] * x2_00_x1_01 * x0_11;
            float x2_00_x1_10 = x2_00 * x1_10;
            y += W[8 ][node_id] * x2_00_x1_10 * x0_00;
            y += W[9 ][node_id] * x2_00_x1_10 * x0_01;
            y += W[10][node_id] * x2_00_x1_10 * x0_10;
            y += W[11][node_id] * x2_00_x1_10 * x0_11;
            float x2_00_x1_11 = x2_00 * x1_11;
            y += W[12][node_id] * x2_00_x1_11 * x0_00;
            y += W[13][node_id] * x2_00_x1_11 * x0_01;
            y += W[14][node_id] * x2_00_x1_11 * x0_10;
            y += W[15][node_id] * x2_00_x1_11 * x0_11;
            float x2_01_x1_00 = x2_01 * x1_00;
            y += W[16][node_id] * x2_01_x1_00 * x0_00;
            y += W[17][node_id] * x2_01_x1_00 * x0_01;
            y += W[18][node_id] * x2_01_x1_00 * x0_10;
            y += W[19][node_id] * x2_01_x1_00 * x0_11;
            float x2_01_x1_01 = x2_01 * x1_01;
            y += W[20][node_id] * x2_01_x1_01 * x0_00;
            y += W[21][node_id] * x2_01_x1_01 * x0_01;
            y += W[22][node_id] * x2_01_x1_01 * x0_10;
            y += W[23][node_id] * x2_01_x1_01 * x0_11;
            float x2_01_x1_10 = x2_01 * x1_10;
            y += W[24][node_id] * x2_01_x1_10 * x0_00;
            y += W[25][node_id] * x2_01_x1_10 * x0_01;
            y += W[26][node_id] * x2_01_x1_10 * x0_10;
            y += W[27][node_id] * x2_01_x1_10 * x0_11;
            float x2_01_x1_11 = x2_01 * x1_11;
            y += W[28][node_id] * x2_01_x1_11 * x0_00;
            y += W[29][node_id] * x2_01_x1_11 * x0_01;
            y += W[30][node_id] * x2_01_x1_11 * x0_10;
            y += W[31][node_id] * x2_01_x1_11 * x0_11;
            float x2_10_x1_00 = x2_10 * x1_00;
            y += W[32][node_id] * x2_10_x1_00 * x0_00;
            y += W[33][node_id] * x2_10_x1_00 * x0_01;
            y += W[34][node_id] * x2_10_x1_00 * x0_10;
            y += W[35][node_id] * x2_10_x1_00 * x0_11;
            float x2_10_x1_01 = x2_10 * x1_01;
            y += W[36][node_id] * x2_10_x1_01 * x0_00;
            y += W[37][node_id] * x2_10_x1_01 * x0_01;
            y += W[38][node_id] * x2_10_x1_01 * x0_10;
            y += W[39][node_id] * x2_10_x1_01 * x0_11;
            float x2_10_x1_10 = x2_10 * x1_10;
            y += W[40][node_id] * x2_10_x1_10 * x0_00;
            y += W[41][node_id] * x2_10_x1_10 * x0_01;
            y += W[42][node_id] * x2_10_x1_10 * x0_10;
            y += W[43][node_id] * x2_10_x1_10 * x0_11;
            float x2_10_x1_11 = x2_10 * x1_11;
            y += W[44][node_id] * x2_10_x1_11 * x0_00;
            y += W[45][node_id] * x2_10_x1_11 * x0_01;
            y += W[46][node_id] * x2_10_x1_11 * x0_10;
            y += W[47][node_id] * x2_10_x1_11 * x0_11;
            float x2_11_x1_00 = x2_11 * x1_00;
            y += W[48][node_id] * x2_11_x1_00 * x0_00;
            y += W[49][node_id] * x2_11_x1_00 * x0_01;
            y += W[50][node_id] * x2_11_x1_00 * x0_10;
            y += W[51][node_id] * x2_11_x1_00 * x0_11;
            float x2_11_x1_01 = x2_11 * x1_01;
            y += W[52][node_id] * x2_11_x1_01 * x0_00;
            y += W[53][node_id] * x2_11_x1_01 * x0_01;
            y += W[54][node_id] * x2_11_x1_01 * x0_10;
            y += W[55][node_id] * x2_11_x1_01 * x0_11;
            float x2_11_x1_10 = x2_11 * x1_10;
            y += W[56][node_id] * x2_11_x1_10 * x0_00;
            y += W[57][node_id] * x2_11_x1_10 * x0_01;
            y += W[58][node_id] * x2_11_x1_10 * x0_10;
            y += W[59][node_id] * x2_11_x1_10 * x0_11;
            float x2_11_x1_11 = x2_11 * x1_11;
            y += W[60][node_id] * x2_11_x1_11 * x0_00;
            y += W[61][node_id] * x2_11_x1_11 * x0_01;
            y += W[62][node_id] * x2_11_x1_11 * x0_10;
            y += W[63][node_id] * x2_11_x1_11 * x0_11;

            // clamp
            y = max(param_min, y);
            y = min(param_max, y);
        
            y_ptr[frame] = y;
        }

        __syncthreads();
    }
}


int bbcu_fp32_StochasticLut6_Forward
        (
            const float     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             input_binary,
            int             lut_binarize,
            float           param_min,
            float           param_max,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 512;
    unsigned int const MAX_NODE_UNIT  = 64;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_fp32_StochasticLut6_Forward<MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            node_size,
            frame_size,
            frame_stride,
            input_binary,
            lut_binarize,
            param_min,
            param_max
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



////////////////



template<int MAX_NODE_UNIT=32>
__global__ void kernal_bit_fp32_StochasticLut6_Forward(
            int   const     *x_buf,
            float           *y_buf,
            int   const     *input_index,
            float const     *W_buf,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             binary_mode,
            float           param_min,
            float           param_max
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__ float    W[64][MAX_NODE_UNIT];
    int   const         *x_ptr[6];
    float               *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( binary_mode ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < 6; ++i ) {
            x_ptr[i] = &x_buf[bin_frame_stride * input_index[6*node + i]];
        }

        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);

            float   xp[6], xn[6];
            for ( int i = 0; i < 6; ++i) {
                float x_val = (x_ptr[i][unit] & bit) ? 0.7 : 0.3;
                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            float x0_00 = xn[1] * xn[0];
            float x0_01 = xn[1] * xp[0];
            float x0_10 = xp[1] * xn[0];
            float x0_11 = xp[1] * xp[0];
            float x1_00 = xn[3] * xn[2];
            float x1_01 = xn[3] * xp[2];
            float x1_10 = xp[3] * xn[2];
            float x1_11 = xp[3] * xp[2];
            float x2_00 = xn[5] * xn[4];
            float x2_01 = xn[5] * xp[4];
            float x2_10 = xp[5] * xn[4];
            float x2_11 = xp[5] * xp[4];

            float y = 0;
            float x2_00_x1_00 = x2_00 * x1_00;
            y += W[0 ][node_id] * x2_00_x1_00 * x0_00;
            y += W[1 ][node_id] * x2_00_x1_00 * x0_01;
            y += W[2 ][node_id] * x2_00_x1_00 * x0_10;
            y += W[3 ][node_id] * x2_00_x1_00 * x0_11;
            float x2_00_x1_01 = x2_00 * x1_01;
            y += W[4 ][node_id] * x2_00_x1_01 * x0_00;
            y += W[5 ][node_id] * x2_00_x1_01 * x0_01;
            y += W[6 ][node_id] * x2_00_x1_01 * x0_10;
            y += W[7 ][node_id] * x2_00_x1_01 * x0_11;
            float x2_00_x1_10 = x2_00 * x1_10;
            y += W[8 ][node_id] * x2_00_x1_10 * x0_00;
            y += W[9 ][node_id] * x2_00_x1_10 * x0_01;
            y += W[10][node_id] * x2_00_x1_10 * x0_10;
            y += W[11][node_id] * x2_00_x1_10 * x0_11;
            float x2_00_x1_11 = x2_00 * x1_11;
            y += W[12][node_id] * x2_00_x1_11 * x0_00;
            y += W[13][node_id] * x2_00_x1_11 * x0_01;
            y += W[14][node_id] * x2_00_x1_11 * x0_10;
            y += W[15][node_id] * x2_00_x1_11 * x0_11;
            float x2_01_x1_00 = x2_01 * x1_00;
            y += W[16][node_id] * x2_01_x1_00 * x0_00;
            y += W[17][node_id] * x2_01_x1_00 * x0_01;
            y += W[18][node_id] * x2_01_x1_00 * x0_10;
            y += W[19][node_id] * x2_01_x1_00 * x0_11;
            float x2_01_x1_01 = x2_01 * x1_01;
            y += W[20][node_id] * x2_01_x1_01 * x0_00;
            y += W[21][node_id] * x2_01_x1_01 * x0_01;
            y += W[22][node_id] * x2_01_x1_01 * x0_10;
            y += W[23][node_id] * x2_01_x1_01 * x0_11;
            float x2_01_x1_10 = x2_01 * x1_10;
            y += W[24][node_id] * x2_01_x1_10 * x0_00;
            y += W[25][node_id] * x2_01_x1_10 * x0_01;
            y += W[26][node_id] * x2_01_x1_10 * x0_10;
            y += W[27][node_id] * x2_01_x1_10 * x0_11;
            float x2_01_x1_11 = x2_01 * x1_11;
            y += W[28][node_id] * x2_01_x1_11 * x0_00;
            y += W[29][node_id] * x2_01_x1_11 * x0_01;
            y += W[30][node_id] * x2_01_x1_11 * x0_10;
            y += W[31][node_id] * x2_01_x1_11 * x0_11;
            float x2_10_x1_00 = x2_10 * x1_00;
            y += W[32][node_id] * x2_10_x1_00 * x0_00;
            y += W[33][node_id] * x2_10_x1_00 * x0_01;
            y += W[34][node_id] * x2_10_x1_00 * x0_10;
            y += W[35][node_id] * x2_10_x1_00 * x0_11;
            float x2_10_x1_01 = x2_10 * x1_01;
            y += W[36][node_id] * x2_10_x1_01 * x0_00;
            y += W[37][node_id] * x2_10_x1_01 * x0_01;
            y += W[38][node_id] * x2_10_x1_01 * x0_10;
            y += W[39][node_id] * x2_10_x1_01 * x0_11;
            float x2_10_x1_10 = x2_10 * x1_10;
            y += W[40][node_id] * x2_10_x1_10 * x0_00;
            y += W[41][node_id] * x2_10_x1_10 * x0_01;
            y += W[42][node_id] * x2_10_x1_10 * x0_10;
            y += W[43][node_id] * x2_10_x1_10 * x0_11;
            float x2_10_x1_11 = x2_10 * x1_11;
            y += W[44][node_id] * x2_10_x1_11 * x0_00;
            y += W[45][node_id] * x2_10_x1_11 * x0_01;
            y += W[46][node_id] * x2_10_x1_11 * x0_10;
            y += W[47][node_id] * x2_10_x1_11 * x0_11;
            float x2_11_x1_00 = x2_11 * x1_00;
            y += W[48][node_id] * x2_11_x1_00 * x0_00;
            y += W[49][node_id] * x2_11_x1_00 * x0_01;
            y += W[50][node_id] * x2_11_x1_00 * x0_10;
            y += W[51][node_id] * x2_11_x1_00 * x0_11;
            float x2_11_x1_01 = x2_11 * x1_01;
            y += W[52][node_id] * x2_11_x1_01 * x0_00;
            y += W[53][node_id] * x2_11_x1_01 * x0_01;
            y += W[54][node_id] * x2_11_x1_01 * x0_10;
            y += W[55][node_id] * x2_11_x1_01 * x0_11;
            float x2_11_x1_10 = x2_11 * x1_10;
            y += W[56][node_id] * x2_11_x1_10 * x0_00;
            y += W[57][node_id] * x2_11_x1_10 * x0_01;
            y += W[58][node_id] * x2_11_x1_10 * x0_10;
            y += W[59][node_id] * x2_11_x1_10 * x0_11;
            float x2_11_x1_11 = x2_11 * x1_11;
            y += W[60][node_id] * x2_11_x1_11 * x0_00;
            y += W[61][node_id] * x2_11_x1_11 * x0_01;
            y += W[62][node_id] * x2_11_x1_11 * x0_10;
            y += W[63][node_id] * x2_11_x1_11 * x0_11;

            // clamp
            y = max(param_min, y);
            y = min(param_max, y);
            
//          printf("[StochasticLut6] node=%d frame=%d y=%f\n", node, frame, y);

            y_ptr[frame] = y;
        }
    }
}


int bbcu_bit_fp32_StochasticLut6_Forward
        (
            int   const     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize,
            float           param_min,
            float           param_max,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 512;
    unsigned int const MAX_NODE_UNIT  = 64;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_bit_fp32_StochasticLut6_Forward<MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            node_size,
            frame_size,
            frame_stride,
            bin_frame_stride,
            lut_binarize,
            param_min,
            param_max
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}




// -------------------------------------------------
//  Backward
// -------------------------------------------------


__device__ __forceinline__ float device_fp32_LocalSum(float v, float *buf)
{
    buf[threadIdx.x] = v;
    __syncthreads();

    // �X���b�h�ԏW�v
    int comb = 1;
    while (comb < blockDim.x) {
        int next = comb * 2;
        int mask = next - 1;
        if ((threadIdx.x & mask) == 0) {
            buf[threadIdx.x] += buf[threadIdx.x + comb];
        }
        comb = next;
        __syncthreads();
    }

    float sum = buf[0];
    __syncthreads();
    
    return sum;
}


// kernel
template<int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_fp32_StochasticLut6_Backward
        (
            float const     *x_buf,
            float const     *dy_buf,
            float           *dx_buf,
            int   const     *input_index,
            float const     *W_buf,
            float           *dW_buf,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             input_binary,
            int             lut_binarize
        )
{

    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  float       dW_prev[64][MAX_NODE_UNIT];
    __shared__  float       W[64][MAX_NODE_UNIT];
                float       dW[64];
                float const *x_ptr[6];
                float const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < 64; ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < 64; i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * 64 + i];
        }

        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
    
        // init pointer
        for ( int i = 0; i < 6; ++i ) {
            int input_node = input_index[6*node + i];
            x_ptr[i]  = &x_buf[input_node * frame_stride];
        }

        dy_ptr = &dy_buf[node * frame_stride];
    }

    __syncthreads();

    if ( node < node_size ) {
        for ( int frame = id; frame < frame_size; frame += id_step ) {
            float xp[6], xn[6];
            for ( int i = 0; i < 6; ++i) {
                float x_val = x_ptr[i][frame];
                if ( input_binary ) {
                    x_val = (x_val > 0.5) ? 0.7 : 0.3;
                }
                else {
                    x_val = min(1.0, max(0.0, x_val));
                }

                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            float x0_00 = xn[1] * xn[0];
            float x0_01 = xn[1] * xp[0];
            float x0_10 = xp[1] * xn[0];
            float x0_11 = xp[1] * xp[0];
            float x1_00 = xn[3] * xn[2];
            float x1_01 = xn[3] * xp[2];
            float x1_10 = xp[3] * xn[2];
            float x1_11 = xp[3] * xp[2];
            float x2_00 = xn[5] * xn[4];
            float x2_01 = xn[5] * xp[4];
            float x2_10 = xp[5] * xn[4];
            float x2_11 = xp[5] * xp[4];

            float grad = dy_ptr[frame];

            float  x2_00_x1_00 =  x2_00 * x1_00;
            float  x2_00_x1_01 =  x2_00 * x1_01;
            float  x2_00_x1_10 =  x2_00 * x1_10;
            float  x2_00_x1_11 =  x2_00 * x1_11;
            float  x2_01_x1_00 =  x2_01 * x1_00;
            float  x2_01_x1_01 =  x2_01 * x1_01;
            float  x2_01_x1_10 =  x2_01 * x1_10;
            float  x2_01_x1_11 =  x2_01 * x1_11;
            float  x2_10_x1_00 =  x2_10 * x1_00;
            float  x2_10_x1_01 =  x2_10 * x1_01;
            float  x2_10_x1_10 =  x2_10 * x1_10;
            float  x2_10_x1_11 =  x2_10 * x1_11;
            float  x2_11_x1_00 =  x2_11 * x1_00;
            float  x2_11_x1_01 =  x2_11 * x1_01;
            float  x2_11_x1_10 =  x2_11 * x1_10;
            float  x2_11_x1_11 =  x2_11 * x1_11;

            dW[ 0] += x2_00_x1_00 * x0_00 * grad;
            dW[ 1] += x2_00_x1_00 * x0_01 * grad;
            dW[ 2] += x2_00_x1_00 * x0_10 * grad;
            dW[ 3] += x2_00_x1_00 * x0_11 * grad;
            dW[ 4] += x2_00_x1_01 * x0_00 * grad;
            dW[ 5] += x2_00_x1_01 * x0_01 * grad;
            dW[ 6] += x2_00_x1_01 * x0_10 * grad;
            dW[ 7] += x2_00_x1_01 * x0_11 * grad;
            dW[ 8] += x2_00_x1_10 * x0_00 * grad;
            dW[ 9] += x2_00_x1_10 * x0_01 * grad;
            dW[10] += x2_00_x1_10 * x0_10 * grad;
            dW[11] += x2_00_x1_10 * x0_11 * grad;
            dW[12] += x2_00_x1_11 * x0_00 * grad;
            dW[13] += x2_00_x1_11 * x0_01 * grad;
            dW[14] += x2_00_x1_11 * x0_10 * grad;
            dW[15] += x2_00_x1_11 * x0_11 * grad;
            dW[16] += x2_01_x1_00 * x0_00 * grad;
            dW[17] += x2_01_x1_00 * x0_01 * grad;
            dW[18] += x2_01_x1_00 * x0_10 * grad;
            dW[19] += x2_01_x1_00 * x0_11 * grad;
            dW[20] += x2_01_x1_01 * x0_00 * grad;
            dW[21] += x2_01_x1_01 * x0_01 * grad;
            dW[22] += x2_01_x1_01 * x0_10 * grad;
            dW[23] += x2_01_x1_01 * x0_11 * grad;
            dW[24] += x2_01_x1_10 * x0_00 * grad;
            dW[25] += x2_01_x1_10 * x0_01 * grad;
            dW[26] += x2_01_x1_10 * x0_10 * grad;
            dW[27] += x2_01_x1_10 * x0_11 * grad;
            dW[28] += x2_01_x1_11 * x0_00 * grad;
            dW[29] += x2_01_x1_11 * x0_01 * grad;
            dW[30] += x2_01_x1_11 * x0_10 * grad;
            dW[31] += x2_01_x1_11 * x0_11 * grad;
            dW[32] += x2_10_x1_00 * x0_00 * grad;
            dW[33] += x2_10_x1_00 * x0_01 * grad;
            dW[34] += x2_10_x1_00 * x0_10 * grad;
            dW[35] += x2_10_x1_00 * x0_11 * grad;
            dW[36] += x2_10_x1_01 * x0_00 * grad;
            dW[37] += x2_10_x1_01 * x0_01 * grad;
            dW[38] += x2_10_x1_01 * x0_10 * grad;
            dW[39] += x2_10_x1_01 * x0_11 * grad;
            dW[40] += x2_10_x1_10 * x0_00 * grad;
            dW[41] += x2_10_x1_10 * x0_01 * grad;
            dW[42] += x2_10_x1_10 * x0_10 * grad;
            dW[43] += x2_10_x1_10 * x0_11 * grad;
            dW[44] += x2_10_x1_11 * x0_00 * grad;
            dW[45] += x2_10_x1_11 * x0_01 * grad;
            dW[46] += x2_10_x1_11 * x0_10 * grad;
            dW[47] += x2_10_x1_11 * x0_11 * grad;
            dW[48] += x2_11_x1_00 * x0_00 * grad;
            dW[49] += x2_11_x1_00 * x0_01 * grad;
            dW[50] += x2_11_x1_00 * x0_10 * grad;
            dW[51] += x2_11_x1_00 * x0_11 * grad;
            dW[52] += x2_11_x1_01 * x0_00 * grad;
            dW[53] += x2_11_x1_01 * x0_01 * grad;
            dW[54] += x2_11_x1_01 * x0_10 * grad;
            dW[55] += x2_11_x1_01 * x0_11 * grad;
            dW[56] += x2_11_x1_10 * x0_00 * grad;
            dW[57] += x2_11_x1_10 * x0_01 * grad;
            dW[58] += x2_11_x1_10 * x0_10 * grad;
            dW[59] += x2_11_x1_10 * x0_11 * grad;
            dW[60] += x2_11_x1_11 * x0_00 * grad;
            dW[61] += x2_11_x1_11 * x0_01 * grad;
            dW[62] += x2_11_x1_11 * x0_10 * grad;
            dW[63] += x2_11_x1_11 * x0_11 * grad;

            float  x2_00_x0_00 =  x2_00 * x0_00;
            float  x2_00_x0_01 =  x2_00 * x0_01;
            float  x2_00_x0_10 =  x2_00 * x0_10;
            float  x2_00_x0_11 =  x2_00 * x0_11;
            float  x2_01_x0_00 =  x2_01 * x0_00;
            float  x2_01_x0_01 =  x2_01 * x0_01;
            float  x2_01_x0_10 =  x2_01 * x0_10;
            float  x2_01_x0_11 =  x2_01 * x0_11;
            float  x2_10_x0_00 =  x2_10 * x0_00;
            float  x2_10_x0_01 =  x2_10 * x0_01;
            float  x2_10_x0_10 =  x2_10 * x0_10;
            float  x2_10_x0_11 =  x2_10 * x0_11;
            float  x2_11_x0_00 =  x2_11 * x0_00;
            float  x2_11_x0_01 =  x2_11 * x0_01;
            float  x2_11_x0_10 =  x2_11 * x0_10;
            float  x2_11_x0_11 =  x2_11 * x0_11;

            float  x1_00_x0_00 =  x1_00 * x0_00;
            float  x1_00_x0_01 =  x1_00 * x0_01;
            float  x1_00_x0_10 =  x1_00 * x0_10;
            float  x1_00_x0_11 =  x1_00 * x0_11;
            float  x1_01_x0_00 =  x1_01 * x0_00;
            float  x1_01_x0_01 =  x1_01 * x0_01;
            float  x1_01_x0_10 =  x1_01 * x0_10;
            float  x1_01_x0_11 =  x1_01 * x0_11;
            float  x1_10_x0_00 =  x1_10 * x0_00;
            float  x1_10_x0_01 =  x1_10 * x0_01;
            float  x1_10_x0_10 =  x1_10 * x0_10;
            float  x1_10_x0_11 =  x1_10 * x0_11;
            float  x1_11_x0_00 =  x1_11 * x0_00;
            float  x1_11_x0_01 =  x1_11 * x0_01;
            float  x1_11_x0_10 =  x1_11 * x0_10;
            float  x1_11_x0_11 =  x1_11 * x0_11;


            float dxi;
            float dx0_00 = 0;
            float dx0_01 = 0;
            float dx0_10 = 0;
            float dx0_11 = 0;
            float dx1_00 = 0;
            float dx1_01 = 0;
            float dx1_10 = 0;
            float dx1_11 = 0;
            float dx2_00 = 0;
            float dx2_01 = 0;
            float dx2_10 = 0;
            float dx2_11 = 0;
            dxi = W[ 0][node_id];  dx0_00 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_00_x0_00;
            dxi = W[ 1][node_id];  dx0_01 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_00_x0_01;
            dxi = W[ 2][node_id];  dx0_10 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_00_x0_10;
            dxi = W[ 3][node_id];  dx0_11 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_00_x0_11;
            dxi = W[ 4][node_id];  dx0_00 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_01_x0_00;
            dxi = W[ 5][node_id];  dx0_01 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_01_x0_01;
            dxi = W[ 6][node_id];  dx0_10 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_01_x0_10;
            dxi = W[ 7][node_id];  dx0_11 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_01_x0_11;
            dxi = W[ 8][node_id];  dx0_00 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_10_x0_00;
            dxi = W[ 9][node_id];  dx0_01 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_10_x0_01;
            dxi = W[10][node_id];  dx0_10 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_10_x0_10;
            dxi = W[11][node_id];  dx0_11 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_10_x0_11;
            dxi = W[12][node_id];  dx0_00 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_11_x0_00;
            dxi = W[13][node_id];  dx0_01 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_11_x0_01;
            dxi = W[14][node_id];  dx0_10 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_11_x0_10;
            dxi = W[15][node_id];  dx0_11 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_11_x0_11;
            dxi = W[16][node_id];  dx0_00 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_00_x0_00;
            dxi = W[17][node_id];  dx0_01 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_00_x0_01;
            dxi = W[18][node_id];  dx0_10 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_00_x0_10;
            dxi = W[19][node_id];  dx0_11 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_00_x0_11;
            dxi = W[20][node_id];  dx0_00 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_01_x0_00;
            dxi = W[21][node_id];  dx0_01 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_01_x0_01;
            dxi = W[22][node_id];  dx0_10 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_01_x0_10;
            dxi = W[23][node_id];  dx0_11 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_01_x0_11;
            dxi = W[24][node_id];  dx0_00 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_10_x0_00;
            dxi = W[25][node_id];  dx0_01 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_10_x0_01;
            dxi = W[26][node_id];  dx0_10 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_10_x0_10;
            dxi = W[27][node_id];  dx0_11 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_10_x0_11;
            dxi = W[28][node_id];  dx0_00 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_11_x0_00;
            dxi = W[29][node_id];  dx0_01 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_11_x0_01;
            dxi = W[30][node_id];  dx0_10 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_11_x0_10;
            dxi = W[31][node_id];  dx0_11 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_11_x0_11;
            dxi = W[32][node_id];  dx0_00 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_00_x0_00;
            dxi = W[33][node_id];  dx0_01 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_00_x0_01;
            dxi = W[34][node_id];  dx0_10 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_00_x0_10;
            dxi = W[35][node_id];  dx0_11 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_00_x0_11;
            dxi = W[36][node_id];  dx0_00 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_01_x0_00;
            dxi = W[37][node_id];  dx0_01 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_01_x0_01;
            dxi = W[38][node_id];  dx0_10 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_01_x0_10;
            dxi = W[39][node_id];  dx0_11 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_01_x0_11;
            dxi = W[40][node_id];  dx0_00 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_10_x0_00;
            dxi = W[41][node_id];  dx0_01 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_10_x0_01;
            dxi = W[42][node_id];  dx0_10 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_10_x0_10;
            dxi = W[43][node_id];  dx0_11 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_10_x0_11;
            dxi = W[44][node_id];  dx0_00 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_11_x0_00;
            dxi = W[45][node_id];  dx0_01 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_11_x0_01;
            dxi = W[46][node_id];  dx0_10 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_11_x0_10;
            dxi = W[47][node_id];  dx0_11 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_11_x0_11;
            dxi = W[48][node_id];  dx0_00 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_00_x0_00;
            dxi = W[49][node_id];  dx0_01 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_00_x0_01;
            dxi = W[50][node_id];  dx0_10 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_00_x0_10;
            dxi = W[51][node_id];  dx0_11 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_00_x0_11;
            dxi = W[52][node_id];  dx0_00 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_01_x0_00;
            dxi = W[53][node_id];  dx0_01 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_01_x0_01;
            dxi = W[54][node_id];  dx0_10 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_01_x0_10;
            dxi = W[55][node_id];  dx0_11 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_01_x0_11;
            dxi = W[56][node_id];  dx0_00 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_10_x0_00;
            dxi = W[57][node_id];  dx0_01 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_10_x0_01;
            dxi = W[58][node_id];  dx0_10 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_10_x0_10;
            dxi = W[59][node_id];  dx0_11 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_10_x0_11;
            dxi = W[60][node_id];  dx0_00 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_11_x0_00;
            dxi = W[61][node_id];  dx0_01 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_11_x0_01;
            dxi = W[62][node_id];  dx0_10 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_11_x0_10;
            dxi = W[63][node_id];  dx0_11 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_11_x0_11;
        
            float *dx_ptr = &dx_buf[(node*6)*frame_stride + frame];
            float dxn;
            float dxp;
            float dx;
            dxn  = dx0_00 * xn[1];    dxn += dx0_10 * xp[1];
            dxp  = dx0_01 * xn[1];    dxp += dx0_11 * xp[1];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[0 * frame_stride] = dx;

            dxn  = dx0_00 * xn[0];
            dxn += dx0_01 * xp[0];
            dxp  = dx0_10 * xn[0];
            dxp += dx0_11 * xp[0];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[1 * frame_stride] = dx;

            dxn  = dx1_00 * xn[3];     
            dxp  = dx1_01 * xn[3];     
            dxn += dx1_10 * xp[3];     
            dxp += dx1_11 * xp[3];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[2 * frame_stride] = dx;

            dxn  = dx1_00 * xn[2];
            dxn += dx1_01 * xp[2];
            dxp  = dx1_10 * xn[2];
            dxp += dx1_11 * xp[2];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[3 * frame_stride] = dx;

            dxn  = dx2_00 * xn[5];     
            dxp  = dx2_01 * xn[5];     
            dxn += dx2_10 * xp[5];     
            dxp += dx2_11 * xp[5];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[4 * frame_stride] = dx;

            dxn  = dx2_00 * xn[4];
            dxn += dx2_01 * xp[4];
            dxp  = dx2_10 * xn[4];
            dxp += dx2_11 * xp[4];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[5 * frame_stride] = dx;
        }
    }

    for ( int i = 0; i < 64; ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < 64; ++i) {
                dW_buf[node*64 + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}


__global__ void kernal_fp32_StochasticLut6_BackwardMarge(
            const float*    src_buf,
            float*          dst_buf,
            const int*      input_index,
            int             node_size,
            int             frame_size,
            int             frame_stride
        )
{
    int frame = blockDim.x * blockIdx.x + threadIdx.x;

    for ( int node = 0; node < node_size; ++node ) {
        if ( frame < frame_size ) {
            for ( int n = 0; n < 6; ++n ) {
                int in_idx = input_index[node*6 + n];
                float*       dst_buf_ptr = &dst_buf[frame_stride * in_idx];
                float        prev_data = dst_buf_ptr[frame];
                const float* src_buf_ptr = &src_buf[(6 * node + n) * frame_stride];
                
                dst_buf_ptr[frame] = prev_data + src_buf_ptr[frame];
            }
        }
        __syncthreads();
    }
}


int bbcu_fp32_StochasticLut6_Backward(
            float const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_W,
            float           *dev_dW,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             input_binary,
            int             lut_binarize,
            hipStream_t    streamId
    )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

        kernal_fp32_StochasticLut6_Backward<MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_W,
                dev_dW,
                output_node_size,
                frame_size,
                frame_stride,
                input_binary,
                lut_binarize
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = frame_size;
        while ( block_x > 1024 ) { block_x /= 2; }

        dim3    grid((frame_size + block_x - 1) /block_x, 1);
        dim3    block(block_x, 1, 1);
        kernal_fp32_StochasticLut6_BackwardMarge<<<grid, block>>>(
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}



////////////////////////////


template<int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_bit_fp32_StochasticLut6_Backward
        (
            int   const     *x_buf,
            float const     *dy_buf,
            float           *dx_buf,
            int   const     *input_index,
            float const     *W_buf,
            float           *dW_buf,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize
        )
{

    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  float       dW_prev[64][MAX_NODE_UNIT];
    __shared__  float       W[64][MAX_NODE_UNIT];
                float       dW[64];
                int   const *x_ptr[6];
                float const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < 64; ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < 64; i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * 64 + i];
        }

        // read W
        for ( int i = id; i < 64; i += id_step ) {
            W[i][node_id] = W_buf[node * 64 + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // init pointer
        for ( int i = 0; i < 6; ++i ) {
            int input_node = input_index[6*node + i];
            x_ptr[i]  = &x_buf[input_node * bin_frame_stride];
        }

        dy_ptr = &dy_buf[node * frame_stride];
    }

    __syncthreads();

    if ( node < node_size ) {
        for ( int frame = id; frame < frame_size; frame += id_step ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);

            float   xp[6], xn[6];
            for ( int i = 0; i < 6; ++i) {
                float x_val = (x_ptr[i][unit] & bit) ? 0.7 : 0.3;
                xp[i] = x_val;
                xn[i] = 1.0 - x_val;
            }

            float x0_00 = xn[1] * xn[0];
            float x0_01 = xn[1] * xp[0];
            float x0_10 = xp[1] * xn[0];
            float x0_11 = xp[1] * xp[0];
            float x1_00 = xn[3] * xn[2];
            float x1_01 = xn[3] * xp[2];
            float x1_10 = xp[3] * xn[2];
            float x1_11 = xp[3] * xp[2];
            float x2_00 = xn[5] * xn[4];
            float x2_01 = xn[5] * xp[4];
            float x2_10 = xp[5] * xn[4];
            float x2_11 = xp[5] * xp[4];

            float grad = dy_ptr[frame];

            float  x2_00_x1_00 =  x2_00 * x1_00;
            float  x2_00_x1_01 =  x2_00 * x1_01;
            float  x2_00_x1_10 =  x2_00 * x1_10;
            float  x2_00_x1_11 =  x2_00 * x1_11;
            float  x2_01_x1_00 =  x2_01 * x1_00;
            float  x2_01_x1_01 =  x2_01 * x1_01;
            float  x2_01_x1_10 =  x2_01 * x1_10;
            float  x2_01_x1_11 =  x2_01 * x1_11;
            float  x2_10_x1_00 =  x2_10 * x1_00;
            float  x2_10_x1_01 =  x2_10 * x1_01;
            float  x2_10_x1_10 =  x2_10 * x1_10;
            float  x2_10_x1_11 =  x2_10 * x1_11;
            float  x2_11_x1_00 =  x2_11 * x1_00;
            float  x2_11_x1_01 =  x2_11 * x1_01;
            float  x2_11_x1_10 =  x2_11 * x1_10;
            float  x2_11_x1_11 =  x2_11 * x1_11;

            dW[ 0] += x2_00_x1_00 * x0_00 * grad;
            dW[ 1] += x2_00_x1_00 * x0_01 * grad;
            dW[ 2] += x2_00_x1_00 * x0_10 * grad;
            dW[ 3] += x2_00_x1_00 * x0_11 * grad;
            dW[ 4] += x2_00_x1_01 * x0_00 * grad;
            dW[ 5] += x2_00_x1_01 * x0_01 * grad;
            dW[ 6] += x2_00_x1_01 * x0_10 * grad;
            dW[ 7] += x2_00_x1_01 * x0_11 * grad;
            dW[ 8] += x2_00_x1_10 * x0_00 * grad;
            dW[ 9] += x2_00_x1_10 * x0_01 * grad;
            dW[10] += x2_00_x1_10 * x0_10 * grad;
            dW[11] += x2_00_x1_10 * x0_11 * grad;
            dW[12] += x2_00_x1_11 * x0_00 * grad;
            dW[13] += x2_00_x1_11 * x0_01 * grad;
            dW[14] += x2_00_x1_11 * x0_10 * grad;
            dW[15] += x2_00_x1_11 * x0_11 * grad;
            dW[16] += x2_01_x1_00 * x0_00 * grad;
            dW[17] += x2_01_x1_00 * x0_01 * grad;
            dW[18] += x2_01_x1_00 * x0_10 * grad;
            dW[19] += x2_01_x1_00 * x0_11 * grad;
            dW[20] += x2_01_x1_01 * x0_00 * grad;
            dW[21] += x2_01_x1_01 * x0_01 * grad;
            dW[22] += x2_01_x1_01 * x0_10 * grad;
            dW[23] += x2_01_x1_01 * x0_11 * grad;
            dW[24] += x2_01_x1_10 * x0_00 * grad;
            dW[25] += x2_01_x1_10 * x0_01 * grad;
            dW[26] += x2_01_x1_10 * x0_10 * grad;
            dW[27] += x2_01_x1_10 * x0_11 * grad;
            dW[28] += x2_01_x1_11 * x0_00 * grad;
            dW[29] += x2_01_x1_11 * x0_01 * grad;
            dW[30] += x2_01_x1_11 * x0_10 * grad;
            dW[31] += x2_01_x1_11 * x0_11 * grad;
            dW[32] += x2_10_x1_00 * x0_00 * grad;
            dW[33] += x2_10_x1_00 * x0_01 * grad;
            dW[34] += x2_10_x1_00 * x0_10 * grad;
            dW[35] += x2_10_x1_00 * x0_11 * grad;
            dW[36] += x2_10_x1_01 * x0_00 * grad;
            dW[37] += x2_10_x1_01 * x0_01 * grad;
            dW[38] += x2_10_x1_01 * x0_10 * grad;
            dW[39] += x2_10_x1_01 * x0_11 * grad;
            dW[40] += x2_10_x1_10 * x0_00 * grad;
            dW[41] += x2_10_x1_10 * x0_01 * grad;
            dW[42] += x2_10_x1_10 * x0_10 * grad;
            dW[43] += x2_10_x1_10 * x0_11 * grad;
            dW[44] += x2_10_x1_11 * x0_00 * grad;
            dW[45] += x2_10_x1_11 * x0_01 * grad;
            dW[46] += x2_10_x1_11 * x0_10 * grad;
            dW[47] += x2_10_x1_11 * x0_11 * grad;
            dW[48] += x2_11_x1_00 * x0_00 * grad;
            dW[49] += x2_11_x1_00 * x0_01 * grad;
            dW[50] += x2_11_x1_00 * x0_10 * grad;
            dW[51] += x2_11_x1_00 * x0_11 * grad;
            dW[52] += x2_11_x1_01 * x0_00 * grad;
            dW[53] += x2_11_x1_01 * x0_01 * grad;
            dW[54] += x2_11_x1_01 * x0_10 * grad;
            dW[55] += x2_11_x1_01 * x0_11 * grad;
            dW[56] += x2_11_x1_10 * x0_00 * grad;
            dW[57] += x2_11_x1_10 * x0_01 * grad;
            dW[58] += x2_11_x1_10 * x0_10 * grad;
            dW[59] += x2_11_x1_10 * x0_11 * grad;
            dW[60] += x2_11_x1_11 * x0_00 * grad;
            dW[61] += x2_11_x1_11 * x0_01 * grad;
            dW[62] += x2_11_x1_11 * x0_10 * grad;
            dW[63] += x2_11_x1_11 * x0_11 * grad;

            float  x2_00_x0_00 =  x2_00 * x0_00;
            float  x2_00_x0_01 =  x2_00 * x0_01;
            float  x2_00_x0_10 =  x2_00 * x0_10;
            float  x2_00_x0_11 =  x2_00 * x0_11;
            float  x2_01_x0_00 =  x2_01 * x0_00;
            float  x2_01_x0_01 =  x2_01 * x0_01;
            float  x2_01_x0_10 =  x2_01 * x0_10;
            float  x2_01_x0_11 =  x2_01 * x0_11;
            float  x2_10_x0_00 =  x2_10 * x0_00;
            float  x2_10_x0_01 =  x2_10 * x0_01;
            float  x2_10_x0_10 =  x2_10 * x0_10;
            float  x2_10_x0_11 =  x2_10 * x0_11;
            float  x2_11_x0_00 =  x2_11 * x0_00;
            float  x2_11_x0_01 =  x2_11 * x0_01;
            float  x2_11_x0_10 =  x2_11 * x0_10;
            float  x2_11_x0_11 =  x2_11 * x0_11;

            float  x1_00_x0_00 =  x1_00 * x0_00;
            float  x1_00_x0_01 =  x1_00 * x0_01;
            float  x1_00_x0_10 =  x1_00 * x0_10;
            float  x1_00_x0_11 =  x1_00 * x0_11;
            float  x1_01_x0_00 =  x1_01 * x0_00;
            float  x1_01_x0_01 =  x1_01 * x0_01;
            float  x1_01_x0_10 =  x1_01 * x0_10;
            float  x1_01_x0_11 =  x1_01 * x0_11;
            float  x1_10_x0_00 =  x1_10 * x0_00;
            float  x1_10_x0_01 =  x1_10 * x0_01;
            float  x1_10_x0_10 =  x1_10 * x0_10;
            float  x1_10_x0_11 =  x1_10 * x0_11;
            float  x1_11_x0_00 =  x1_11 * x0_00;
            float  x1_11_x0_01 =  x1_11 * x0_01;
            float  x1_11_x0_10 =  x1_11 * x0_10;
            float  x1_11_x0_11 =  x1_11 * x0_11;


            float dxi;
            float dx0_00 = 0;
            float dx0_01 = 0;
            float dx0_10 = 0;
            float dx0_11 = 0;
            float dx1_00 = 0;
            float dx1_01 = 0;
            float dx1_10 = 0;
            float dx1_11 = 0;
            float dx2_00 = 0;
            float dx2_01 = 0;
            float dx2_10 = 0;
            float dx2_11 = 0;
            dxi = W[ 0][node_id];  dx0_00 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_00_x0_00;
            dxi = W[ 1][node_id];  dx0_01 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_00_x0_01;
            dxi = W[ 2][node_id];  dx0_10 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_00_x0_10;
            dxi = W[ 3][node_id];  dx0_11 += dxi * x2_00_x1_00;  dx1_00 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_00_x0_11;
            dxi = W[ 4][node_id];  dx0_00 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_01_x0_00;
            dxi = W[ 5][node_id];  dx0_01 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_01_x0_01;
            dxi = W[ 6][node_id];  dx0_10 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_01_x0_10;
            dxi = W[ 7][node_id];  dx0_11 += dxi * x2_00_x1_01;  dx1_01 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_01_x0_11;
            dxi = W[ 8][node_id];  dx0_00 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_10_x0_00;
            dxi = W[ 9][node_id];  dx0_01 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_10_x0_01;
            dxi = W[10][node_id];  dx0_10 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_10_x0_10;
            dxi = W[11][node_id];  dx0_11 += dxi * x2_00_x1_10;  dx1_10 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_10_x0_11;
            dxi = W[12][node_id];  dx0_00 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_00;  dx2_00 += dxi * x1_11_x0_00;
            dxi = W[13][node_id];  dx0_01 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_01;  dx2_00 += dxi * x1_11_x0_01;
            dxi = W[14][node_id];  dx0_10 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_10;  dx2_00 += dxi * x1_11_x0_10;
            dxi = W[15][node_id];  dx0_11 += dxi * x2_00_x1_11;  dx1_11 += dxi * x2_00_x0_11;  dx2_00 += dxi * x1_11_x0_11;
            dxi = W[16][node_id];  dx0_00 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_00_x0_00;
            dxi = W[17][node_id];  dx0_01 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_00_x0_01;
            dxi = W[18][node_id];  dx0_10 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_00_x0_10;
            dxi = W[19][node_id];  dx0_11 += dxi * x2_01_x1_00;  dx1_00 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_00_x0_11;
            dxi = W[20][node_id];  dx0_00 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_01_x0_00;
            dxi = W[21][node_id];  dx0_01 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_01_x0_01;
            dxi = W[22][node_id];  dx0_10 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_01_x0_10;
            dxi = W[23][node_id];  dx0_11 += dxi * x2_01_x1_01;  dx1_01 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_01_x0_11;
            dxi = W[24][node_id];  dx0_00 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_10_x0_00;
            dxi = W[25][node_id];  dx0_01 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_10_x0_01;
            dxi = W[26][node_id];  dx0_10 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_10_x0_10;
            dxi = W[27][node_id];  dx0_11 += dxi * x2_01_x1_10;  dx1_10 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_10_x0_11;
            dxi = W[28][node_id];  dx0_00 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_00;  dx2_01 += dxi * x1_11_x0_00;
            dxi = W[29][node_id];  dx0_01 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_01;  dx2_01 += dxi * x1_11_x0_01;
            dxi = W[30][node_id];  dx0_10 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_10;  dx2_01 += dxi * x1_11_x0_10;
            dxi = W[31][node_id];  dx0_11 += dxi * x2_01_x1_11;  dx1_11 += dxi * x2_01_x0_11;  dx2_01 += dxi * x1_11_x0_11;
            dxi = W[32][node_id];  dx0_00 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_00_x0_00;
            dxi = W[33][node_id];  dx0_01 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_00_x0_01;
            dxi = W[34][node_id];  dx0_10 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_00_x0_10;
            dxi = W[35][node_id];  dx0_11 += dxi * x2_10_x1_00;  dx1_00 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_00_x0_11;
            dxi = W[36][node_id];  dx0_00 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_01_x0_00;
            dxi = W[37][node_id];  dx0_01 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_01_x0_01;
            dxi = W[38][node_id];  dx0_10 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_01_x0_10;
            dxi = W[39][node_id];  dx0_11 += dxi * x2_10_x1_01;  dx1_01 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_01_x0_11;
            dxi = W[40][node_id];  dx0_00 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_10_x0_00;
            dxi = W[41][node_id];  dx0_01 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_10_x0_01;
            dxi = W[42][node_id];  dx0_10 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_10_x0_10;
            dxi = W[43][node_id];  dx0_11 += dxi * x2_10_x1_10;  dx1_10 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_10_x0_11;
            dxi = W[44][node_id];  dx0_00 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_00;  dx2_10 += dxi * x1_11_x0_00;
            dxi = W[45][node_id];  dx0_01 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_01;  dx2_10 += dxi * x1_11_x0_01;
            dxi = W[46][node_id];  dx0_10 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_10;  dx2_10 += dxi * x1_11_x0_10;
            dxi = W[47][node_id];  dx0_11 += dxi * x2_10_x1_11;  dx1_11 += dxi * x2_10_x0_11;  dx2_10 += dxi * x1_11_x0_11;
            dxi = W[48][node_id];  dx0_00 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_00_x0_00;
            dxi = W[49][node_id];  dx0_01 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_00_x0_01;
            dxi = W[50][node_id];  dx0_10 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_00_x0_10;
            dxi = W[51][node_id];  dx0_11 += dxi * x2_11_x1_00;  dx1_00 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_00_x0_11;
            dxi = W[52][node_id];  dx0_00 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_01_x0_00;
            dxi = W[53][node_id];  dx0_01 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_01_x0_01;
            dxi = W[54][node_id];  dx0_10 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_01_x0_10;
            dxi = W[55][node_id];  dx0_11 += dxi * x2_11_x1_01;  dx1_01 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_01_x0_11;
            dxi = W[56][node_id];  dx0_00 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_10_x0_00;
            dxi = W[57][node_id];  dx0_01 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_10_x0_01;
            dxi = W[58][node_id];  dx0_10 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_10_x0_10;
            dxi = W[59][node_id];  dx0_11 += dxi * x2_11_x1_10;  dx1_10 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_10_x0_11;
            dxi = W[60][node_id];  dx0_00 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_00;  dx2_11 += dxi * x1_11_x0_00;
            dxi = W[61][node_id];  dx0_01 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_01;  dx2_11 += dxi * x1_11_x0_01;
            dxi = W[62][node_id];  dx0_10 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_10;  dx2_11 += dxi * x1_11_x0_10;
            dxi = W[63][node_id];  dx0_11 += dxi * x2_11_x1_11;  dx1_11 += dxi * x2_11_x0_11;  dx2_11 += dxi * x1_11_x0_11;
        
            float *dx_ptr = &dx_buf[(node*6)*frame_stride + frame];
            float dxn;
            float dxp;
            float dx;
            dxn  = dx0_00 * xn[1];    dxn += dx0_10 * xp[1];
            dxp  = dx0_01 * xn[1];    dxp += dx0_11 * xp[1];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[0 * frame_stride] = dx;

            dxn  = dx0_00 * xn[0];
            dxn += dx0_01 * xp[0];
            dxp  = dx0_10 * xn[0];
            dxp += dx0_11 * xp[0];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[1 * frame_stride] = dx;

            dxn  = dx1_00 * xn[3];     
            dxp  = dx1_01 * xn[3];     
            dxn += dx1_10 * xp[3];     
            dxp += dx1_11 * xp[3];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[2 * frame_stride] = dx;

            dxn  = dx1_00 * xn[2];
            dxn += dx1_01 * xp[2];
            dxp  = dx1_10 * xn[2];
            dxp += dx1_11 * xp[2];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[3 * frame_stride] = dx;

            dxn  = dx2_00 * xn[5];     
            dxp  = dx2_01 * xn[5];     
            dxn += dx2_10 * xp[5];     
            dxp += dx2_11 * xp[5];     
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[4 * frame_stride] = dx;

            dxn  = dx2_00 * xn[4];
            dxn += dx2_01 * xp[4];
            dxp  = dx2_10 * xn[4];
            dxp += dx2_11 * xp[4];
            dx = (dxp - dxn) * grad;
            if ( xp[0] == 0.0 || xp[0] == 1.0 ) { dx = 0; }
            dx_ptr[5 * frame_stride] = dx;
        }
    }

    for ( int i = 0; i < 64; ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < 64; ++i) {
                dW_buf[node*64 + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}

int bbcu_bit_fp32_StochasticLut6_Backward(
            int   const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_W,
            float           *dev_dW,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize,
            hipStream_t    streamId
    )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

        kernal_bit_fp32_StochasticLut6_Backward<MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_W,
                dev_dW,
                output_node_size,
                frame_size,
                frame_stride,
                bin_frame_stride,
                lut_binarize
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = frame_size;
        while ( block_x > 1024 ) { block_x /= 2; }

        dim3    grid((frame_size + block_x - 1) /block_x, 1);
        dim3    block(block_x, 1, 1);
        kernal_fp32_StochasticLut6_BackwardMarge<<<grid, block>>>(
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}




// end of file
