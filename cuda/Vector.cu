#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// -------------------------------------------------
//  add_ex
// -------------------------------------------------

__global__ void kernal_Vector_add_ex
		(
			float*			dst,
			const float*	src0,
			const float*	src1,
			float			a,
			float			b,
			float			c,
			int				size
		)
{
    int	index = threadIdx.x;
	while ( index < size ) {
		dst[index] = a * src0[index] + b * src1[index] + c;
		index += blockDim.x;
	}
}

CUBB_DLL_EXPORT int bbcu_Vector_add_ex(
			float*			dev_dst,
			const float*	dev_src0,
			const float*	dev_src1,
			float			a,
			float			b,
			float			c,
			int				size,
            hipStream_t	streamId
		)
{
	kernal_Vector_add_ex<<<1, 1024, 0, streamId>>>
        (
			dev_dst,
			dev_src0,
			dev_src1,
			a,
			b,
			c,
			size
		);
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}



// -------------------------------------------------
//  mul_ex
// -------------------------------------------------

__global__ void kernal_Vector_mul_ex(
			float*			dst,
			const float*	src0,
			const float*	src1,
			float			a,
			float			b,
			int				size)
{
	int	index = threadIdx.x;
	while ( index < size ) {
		dst[index] = a * src0[index] * src1[index] + b;
		index += blockDim.x;
	}
}


CUBB_DLL_EXPORT int bbcu_Vector_mul_ex
        (
			float*			dev_dst,
			const float*	dev_src0,
			const float*	dev_src1,
			float			a,
			float			b,
			int				size,
            hipStream_t	streamId
		)
{
	kernal_Vector_mul_ex<<<1, 1024, 0, streamId>>>
        (
			dev_dst,
			dev_src0,
			dev_src1,
			a,
			b,
			size
		);
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}


// -------------------------------------------------
//  div_ex
// -------------------------------------------------

__global__ void kernal_Vector_div_ex(
			float*			dst,
			const float*	src0,
			const float*	src1,
			float			a,
			float			b,
			float			c,
			float			d,
			int				size)
{
	int	index = threadIdx.x;
	while ( index < size ) {
		dst[index] = (a * src0[index] + b) / (c * src1[index] + d);
		index += blockDim.x;
	}
}


CUBB_DLL_EXPORT int bbcu_Vector_div_ex(
            float           *dev_dst,
            float const     *dev_src0,
            float const     *dev_src1,
            float	        a,
            float	        b,
            float	        c,
            float	        d,
			int				size,
            hipStream_t	streamId
		)
{
	kernal_Vector_div_ex<<<1, 1024, 0, streamId>>>
        (
			dev_dst,
			dev_src0,
			dev_src1,
			a,
			b,
			c,
			d,
			size
		);
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}



// -------------------------------------------------
//  sqrt
// -------------------------------------------------

__global__ void kernal_Vector_sqrt(
			float*			dst,
			const float*	src,
			int				size)
{
	int	index = threadIdx.x;
	while ( index < size ) {
		dst[index] = sqrt(src[index]);
		index += blockDim.x;
	}
}


CUBB_DLL_EXPORT int bbcu_Vector_sqrt(
            float           *dev_dst,
            float const     *dev_src,
			int				size,
            hipStream_t	streamId
		)
{
	kernal_Vector_sqrt<<<1, 1024, 0, streamId>>>
        (
			dev_dst,
			dev_src,
			size
		);
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}




// -------------------------------------------------
//  exp
// -------------------------------------------------

__global__ void kernal_Vector_exp(
			float*			dst,
			const float*	src,
			int				size)
{
	int	index = threadIdx.x;
	while ( index < size ) {
		dst[index] = exp(src[index]);
		index += blockDim.x;
	}
}


CUBB_DLL_EXPORT int bbcu_Vector_exp(
            float           *dev_dst,
            float const     *dev_src,
			int				size,
            hipStream_t	streamId
		)
{
	kernal_Vector_exp<<<1, 1024, 0, streamId>>>
        (
			dev_dst,
			dev_src,
			size
		);
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}



// end of file
