#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// -------------------------------------------------
//  Forward
// -------------------------------------------------

template <int N=6, int M=16>
__global__ void kernal_MicroMlp_Forward(
			const float*	in_sig_buf,
			float*			out_sig_buf,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b)
{
	int frame_step = blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
	__shared__	 float b1;

	// �W���ǂݍ���
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}

		b0[i] = hidden_b[node * M + i];
		W1[i] = output_W[node * M + i];
	}
	if (threadIdx.x == 0) {
		b1 = output_b[node];
	}

	__syncthreads();

	const float *in_sig_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_sig_ptr[i] = &in_sig_buf[frame_size * in_idx];
	}

	float *out_sig_ptr = &out_sig_buf[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_sig[N];
		for ( int i = 0; i < N; ++i ) {
			in_sig[i] = in_sig_ptr[i][frame];
		}

		// �v�Z
		float sig1 = b1;
		for ( int i = 0; i < M; ++i ) {
			float sig0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				sig0 += in_sig[j] * W0[i][j];
			}
		
			sig0 = fmaxf(sig0, 0);	// ReLU
		
			sig1 += sig0 * W1[i];
		}

		// �o��
		out_sig_ptr[frame] = sig1;

		frame += frame_step;
	}
}


template <int N=6, int M=16>
int bbcu_MicroMlp_Forward
		(
			const float*	dev_in_sig,
			float*			dev_out_sig,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			const float*	dev_output_W,
			const float*	dev_output_b,
			hipStream_t	streamId = 0
		)
{
	dim3	grid(output_node_size);
	dim3	block(512, 1, 1);
	
	kernal_MicroMlp_Forward<N, M><<<grid, block, 0, streamId>>>(
			dev_in_sig,
			dev_out_sig,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);
	BB_CUDA_CHECK_LAST_ERROR();
	
	return 0;
}


int bbcu_MicroMlp6x16_Forward
		(
			const float*	dev_in_sig,
			float*			dev_out_sig,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			const float*	dev_output_W,
			const float*	dev_output_b,
			hipStream_t	streamId
		)
{
	return bbcu_MicroMlp_Forward<6, 16>(
			dev_in_sig,
			dev_out_sig,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b,
			streamId
		);
}



template <int N=6, int M=16>
int bbcu_eva_MicroMlp_Forward
		(
			const float*	in_sig_buf,
			float*			out_sig_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b
		)
{
	hipDeviceSynchronize();
	auto time0 = std::chrono::system_clock::now();

	float* dev_in_sig;
	float* dev_out_sig;
	int*   dev_input_index;
	float* dev_hidden_W;
	float* dev_hidden_b;
	float* dev_output_W;
	float* dev_output_b;

	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_sig,   input_node_size * frame_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_out_sig,  output_node_size * frame_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_input_index, output_node_size * N * sizeof(int)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_W, output_node_size * M * N * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_b, output_node_size * M * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_W, output_node_size * M * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_b, output_node_size * sizeof(float)));
	
	hipDeviceSynchronize();
	auto time1 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipMemcpy(dev_input_index, input_index, output_node_size * N * sizeof(int), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_hidden_W, hidden_W, output_node_size * M * N * sizeof(float), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_hidden_b, hidden_b, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_output_W, output_W, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_output_b, output_b, output_node_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time2 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipMemcpy(dev_in_sig, in_sig_buf, input_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time3 = std::chrono::system_clock::now();
	
	bbcu_MicroMlp_Forward<N, M>(
			dev_in_sig,
			dev_out_sig,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);
	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
    }


	hipDeviceSynchronize();
	auto time4 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipMemcpy(out_sig_buf, dev_out_sig, output_node_size * frame_size * sizeof(float), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
	auto time5 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipFree(dev_in_sig));
	BB_CUDA_SAFE_CALL(hipFree(dev_out_sig));
	BB_CUDA_SAFE_CALL(hipFree(dev_hidden_W));
	BB_CUDA_SAFE_CALL(hipFree(dev_hidden_b));
	BB_CUDA_SAFE_CALL(hipFree(dev_output_W));
	BB_CUDA_SAFE_CALL(hipFree(dev_output_b));

	hipDeviceSynchronize();
	auto time6 = std::chrono::system_clock::now();

	double elapsed_malloc       = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time1-time0).count();
	double elapsed_cpu_to_gpu_p = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time2-time1).count();
	double elapsed_cpu_to_gpu   = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time3-time2).count();
	double elapsed_kernel       = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time4-time3).count();
	double elapsed_gpu_to_cpu   = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time5-time4).count();
	double elapsed_free         = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time6-time5).count();

	double kernel_flops = (double)output_node_size *(double) frame_size * (M*N+M+M)*2.0 / elapsed_kernel / 1000000.0;

	std::cout << "malloc               : " << elapsed_malloc       << " [msec]" << std::endl;
	std::cout << "param copy(cpu->gpu) : " << elapsed_cpu_to_gpu_p << " [msec]" << std::endl;
	std::cout << "data copy(cpu->gpu)  : " << elapsed_cpu_to_gpu   << " [msec]" << std::endl;
	std::cout << "kernel               : " << elapsed_kernel       << " [msec]  " << kernel_flops << " [GFLOPS]" << std::endl;
	std::cout << "data copy(gpu->cpu)  : " << elapsed_gpu_to_cpu   << " [msec]" << std::endl;
	std::cout << "free                 : " << elapsed_free         << " [msec]" << std::endl;
	
	return 0;
}


int bbcu_eva_MicroMlp6x16_Forward
		(
			const float*	in_sig_buf,
			float*			out_sig_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b
		)
{
	return bbcu_eva_MicroMlp_Forward
		(
			in_sig_buf,
			out_sig_buf,
			input_node_size,
			output_node_size,
			frame_size,
			input_index,
			hidden_W,
			hidden_b,
			output_W,
			output_b
		);
}



// -------------------------------------------------
//  Backward
// -------------------------------------------------

// kernel
template <int N=6, int M=16, int H=16>
__global__ void kernal_MicroMlp_Backward(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db)
{
	int	id         = threadIdx.x;
	int frame_step = H;	// blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
//				 float b1;

 	__shared__   float dW0[M][N][H];
	__shared__   float db0[M][H];
	__shared__   float dW1[M][H];
	__shared__	 float db1[H];

	// �W���ǂݍ���
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}

		b0[i] = hidden_b[node * M + i];
		W1[i] = output_W[node * M + i];
	}
//	if (threadIdx.x == 0) {
//		b1 = output_b[node];
//	}
	
	// ���z������
	for ( int i = 0; i < M; ++ i ) {
		for ( int j = 0; j < N; ++j ) {
			dW0[i][j][id] = 0; // hidden_dW[(node * M + i) * N + j];
		}
	}
	for ( int i = 0; i < M; ++i ) {
		db0[i][id] = 0; // hidden_db[node * M + i];
	}
	for ( int i = 0; i < M; ++i ) {
		dW1[i][id] = 0; // output_dW[node * M + i];
	}
	db1[id] = 0; // output_db[node];

	__syncthreads();

	const float *in_sig_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_sig_ptr[i] = &in_sig_buf[frame_size * in_idx];
	}

	float	*out_err_ptr = &out_err_buf[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_sig[N];
		for ( int i = 0; i < N; ++i ) {
			in_sig[i] = in_sig_ptr[i][frame];
		}
		
		// 1�i�ڍČv�Z����2�i�ڋt�`�d
		float	err1 = out_err_ptr[frame];
		float	err0[M];
		db1[id] += err1;
		for ( int i = 0; i < M; ++i ) {
			float sig0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				sig0 += in_sig[j] * W0[i][j];
			}
		
			sig0 = fmaxf(sig0, 0);	// ReLU

			dW1[i][id] += err1 * sig0;

			if ( sig0 > 0 ) {		// ReLU
				err0[i] = err1 * W1[i];
			}
			else {
				err0[i] = 0;
			}
		}
		
		// 1�i�ڋt�`�d
		float *in_err_ptr  = &in_err_buf[frame_size * N * node];
		float	in_err[N];
		for ( int i = 0; i < N; ++i ) {
			in_err[i] = 0;	// in_err_ptr[frame_size * i + frame];
		}

		for ( int i = 0; i < M; ++i ) {
			db0[i][id] += err0[i];
			for ( int j = 0; j < N; ++j ) {
				dW0[i][j][id] += err0[i] * in_sig[j];
				in_err[j] += err0[i] * W0[i][j];
			}
		}
		
		// �덷��������
		for ( int i = 0; i < N; ++i ) {
			in_err_ptr[frame_size * i + frame] = in_err[i];
		}

		frame += frame_step;
	}
	
	__syncthreads();

	int comb = 1;
	while ( comb < H ) {
		int next = comb * 2;
		int mask = next - 1;
		if ( (threadIdx.x & mask) == 0 && id + comb < H ) {
			for ( int i = 0; i < M; ++ i ) {
				for ( int j = 0; j < N; ++j ) {
					dW0[i][j][id] += dW0[i][j][id + comb];
				}
			}
			for ( int i = 0; i < M; ++i ) {
				db0[i][id] += db0[i][id + comb];
			}
			for ( int i = 0; i < M; ++i ) {
				dW1[i][id] += dW1[i][id + comb];
			}
			db1[id] += db1[id + comb];
		}
		comb = next;
		__syncthreads();
	}

	// ���z�o��(��ŕ��񉻂���)
	if ( threadIdx.x == 0 ) {
		for ( int i = 0; i < M; ++i ) {
			for ( int j = 0; j < N; ++j ) {
				hidden_dW[(node * M + i) * N + j] = dW0[i][j][0];
			}
		}
		for ( int i = 0; i < M; ++i ) {
			hidden_db[node * M + i] = db0[i][0];
		}
		for ( int i = 0; i < M; ++i ) {
			output_dW[node * M + i] = dW1[i][0];
		}
		output_db[node] = db1[0];
	}
}


template <int N=6>
__global__ void kernal_MicroMlp_BackwardMarge(
			float*			dst_buf,
			const float*	src_buf,
			int				frame_size,
			int				node_size,
			const int*		input_index
		)
{
	int n          = blockDim.y * blockIdx.y + threadIdx.y;
	int frame      = blockDim.x * blockIdx.x + threadIdx.x;
	
	for ( int node = 0; node < node_size; ++node ) {
		int in_idx = input_index[node*N + n];
		float*		 dst_buf_ptr = &dst_buf[frame_size * in_idx];
		const float* src_buf_ptr = &src_buf[(N * node + n) * frame_size];
		
		dst_buf_ptr[frame] += src_buf_ptr[frame];

		__syncthreads();
	}
}


template <int N=6, int M=16>
int bbcu_MicroMlp_Backward(
			const float*	dev_in_sig_buf,
			float*			dev_in_err_buf,
			float*			dev_in_err_tmp,
			float*			dev_out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			float*			dev_hidden_dW,
			float*			dev_hidden_db,
			const float*	dev_output_W,
			const float*	dev_output_b,
			float*			dev_output_dW,
			float*			dev_output_db,
			hipStream_t	streamId = 0
	)
{
	{
		const int x_size = (8192 / (N*M));

		dim3	grid(output_node_size);
		dim3	block(x_size, 1, 1);
		
		kernal_MicroMlp_Backward<N, M, x_size><<<grid, block, 0, streamId>>>(
				dev_in_sig_buf,
				dev_in_err_tmp,
				dev_out_err_buf,
				frame_size,
				dev_input_index,
				dev_hidden_W,
				dev_hidden_b,
				dev_hidden_dW,
				dev_hidden_db,
				dev_output_W,
				dev_output_b,
				dev_output_dW,
				dev_output_db
			);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
	}

	{
		int block_x = frame_size;
		while ( block_x > 1024 ) { block_x /= 2; }

		dim3	grid(frame_size/block_x, N);
		dim3	block(block_x, 1, 1);

		kernal_MicroMlp_BackwardMarge<N><<<grid, block>>>(
				dev_in_err_buf,
				dev_in_err_tmp,
				frame_size,
				output_node_size,
				dev_input_index
			);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
	}

	return 0;
}


CUBB_DLL_EXPORT int bbcu_MicroMlp6x16_Backward(
			const float*	dev_in_sig_buf,
			float*			dev_in_err_buf,
			float*			dev_in_err_tmp,
			float*			dev_out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			float*			dev_hidden_dW,
			float*			dev_hidden_db,
			const float*	dev_output_W,
			const float*	dev_output_b,
			float*			dev_output_dW,
			float*			dev_output_db,
			hipStream_t	streamId
		)
{
	return bbcu_MicroMlp_Backward<6, 16>(
			dev_in_sig_buf,
			dev_in_err_buf,
			dev_in_err_tmp,
			dev_out_err_buf,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_hidden_dW,
			dev_hidden_db,
			dev_output_W,
			dev_output_b,
			dev_output_dW,
			dev_output_db,
			streamId
		);
}


template <int N=6, int M=16>
int bbcu_eva_MicroMlp_Backward
		(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db
		)
{
	hipDeviceProp_t dev;
	BB_CUDA_SAFE_CALL(hipGetDeviceProperties(&dev, 0));

	hipError_t cudaStatus0 = hipGetLastError();
    if (cudaStatus0 != hipSuccess) {
        fprintf(stderr, "start failed: %s\n", hipGetErrorString(cudaStatus0));
		exit(1);
    }

	hipDeviceSynchronize();
	auto time0 = std::chrono::system_clock::now();

	float* dev_in_sig_buf;
	float* dev_in_err_buf;
	float* dev_in_err_tmp;
	float* dev_out_err_buf;

	int*   dev_input_index;
	float* dev_hidden_W;
	float* dev_hidden_b;
	float* dev_output_W;
	float* dev_output_b;
	float* dev_hidden_dW;
	float* dev_hidden_db;
	float* dev_output_dW;
	float* dev_output_db;

	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_sig_buf,  input_node_size * frame_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_err_buf,  input_node_size * frame_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_err_tmp,  output_node_size * N * frame_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_out_err_buf, output_node_size * frame_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_input_index, output_node_size * N * sizeof(int)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_W, output_node_size * M * N * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_b, output_node_size * M * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_W, output_node_size * M * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_b, output_node_size * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_dW, output_node_size * M * N * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_db, output_node_size * M * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_dW, output_node_size * M * sizeof(float)));
	BB_CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_db, output_node_size * sizeof(float)));
	
	hipDeviceSynchronize();
	auto time1 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipMemcpy(dev_input_index, input_index, output_node_size * N * sizeof(int), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_hidden_W, hidden_W, output_node_size * M * N * sizeof(float), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_hidden_b, hidden_b, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_output_W, output_W, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_output_b, output_b, output_node_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time2 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipMemcpy(dev_in_sig_buf,  in_sig_buf,  input_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));
	BB_CUDA_SAFE_CALL(hipMemcpy(dev_out_err_buf, out_err_buf, output_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time3 = std::chrono::system_clock::now();

	bbcu_MicroMlp_Backward<N, M>(
			dev_in_sig_buf,
			dev_in_err_buf,
			dev_in_err_tmp,
			dev_out_err_buf,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_hidden_dW,
			dev_hidden_db,
			dev_output_W,
			dev_output_b,
			dev_output_dW,
			dev_output_db
		);


	hipDeviceSynchronize();
	auto time4 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipMemcpy(in_err_buf, dev_in_err_buf, input_node_size * frame_size * sizeof(float), hipMemcpyDeviceToHost));

	BB_CUDA_SAFE_CALL(hipMemcpy(hidden_dW, dev_hidden_dW, output_node_size * M * N * sizeof(float), hipMemcpyDeviceToHost));
	BB_CUDA_SAFE_CALL(hipMemcpy(hidden_db, dev_hidden_db, output_node_size * M * sizeof(float), hipMemcpyDeviceToHost));
	BB_CUDA_SAFE_CALL(hipMemcpy(output_dW, dev_output_dW, output_node_size * M * sizeof(float), hipMemcpyDeviceToHost));
	BB_CUDA_SAFE_CALL(hipMemcpy(output_db, dev_output_db, output_node_size * sizeof(float), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
	auto time5 = std::chrono::system_clock::now();

	BB_CUDA_SAFE_CALL(hipFree(dev_in_sig_buf));
	BB_CUDA_SAFE_CALL(hipFree(dev_in_err_buf));
	BB_CUDA_SAFE_CALL(hipFree(dev_in_err_tmp));
	BB_CUDA_SAFE_CALL(hipFree(dev_out_err_buf));
	BB_CUDA_SAFE_CALL(hipFree(dev_input_index));
	BB_CUDA_SAFE_CALL(hipFree(dev_hidden_W));
	BB_CUDA_SAFE_CALL(hipFree(dev_hidden_b));
	BB_CUDA_SAFE_CALL(hipFree(dev_output_W));
	BB_CUDA_SAFE_CALL(hipFree(dev_output_b));
	BB_CUDA_SAFE_CALL(hipFree(dev_hidden_dW));
	BB_CUDA_SAFE_CALL(hipFree(dev_hidden_db));
	BB_CUDA_SAFE_CALL(hipFree(dev_output_dW));
	BB_CUDA_SAFE_CALL(hipFree(dev_output_db));

	hipDeviceSynchronize();
	auto time6 = std::chrono::system_clock::now();

	double elapsed_malloc       = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time1-time0).count();
	double elapsed_cpu_to_gpu_p = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time2-time1).count();
	double elapsed_cpu_to_gpu   = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time3-time2).count();
	double elapsed_kernel       = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time4-time3).count();
	double elapsed_gpu_to_cpu   = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time5-time4).count();
	double elapsed_free         = (double)std::chrono::duration_cast<std::chrono::milliseconds>(time6-time5).count();
//	double kernel_flops = (double)output_node_size *(double) frame_size * (16.0*6.0+16.0+16.0)*2.0 / elapsed_kernel / 1000000.0;
	std::cout << "malloc               : " << elapsed_malloc       << " [msec]" << std::endl;
	std::cout << "param copy(cpu->gpu) : " << elapsed_cpu_to_gpu_p << " [msec]" << std::endl;
	std::cout << "data copy(cpu->gpu)  : " << elapsed_cpu_to_gpu   << " [msec]" << std::endl;
	std::cout << "kernel               : " << elapsed_kernel       << " [msec]" << std::endl;
//	 << kernel_flops << " [GFLOPS])" << std::endl;
	std::cout << "data copy(gpu->cpu)  : " << elapsed_gpu_to_cpu   << " [msec]" << std::endl;
	std::cout << "free                 : " << elapsed_free         << " [msec]" << std::endl;
	
	return 0;
}



CUBB_DLL_EXPORT int bbcu_eva_MicroMlp6x16_Backward
		(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db
		)
{

	return bbcu_eva_MicroMlp_Backward<6, 16>
		(
			in_sig_buf,
			in_err_buf,
			out_err_buf,
			input_node_size,
			output_node_size,
			frame_size,
			input_index,
			hidden_W,
			hidden_b,
			hidden_dW,
			hidden_db,
			output_W,
			output_b,
			output_dW,
			output_db
		);
}

