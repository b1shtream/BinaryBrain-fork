#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



// -------------------------------------------------
//  Forward
// -------------------------------------------------

template <int N=6, int M=16>
__global__ void kernal_MicroMlp_Forward(
			const float*	in_sig_buf,
			float*			out_sig_buf,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b)
{
	int frame_step = blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
	__shared__	 float b1;

	// �W���ǂݍ���
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}

		b0[i] = hidden_b[node * M + i];
		W1[i] = output_W[node * M + i];
	}
	if (threadIdx.x == 0) {
		b1 = output_b[node];
	}

	__syncthreads();

	const float *in_sig_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_sig_ptr[i] = &in_sig_buf[frame_size * in_idx];
	}

	float *out_sig_ptr = &out_sig_buf[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_sig[N];
		for ( int i = 0; i < N; ++i ) {
			in_sig[i] = in_sig_ptr[i][frame];
		}

		// �v�Z
		float sig1 = b1;
		for ( int i = 0; i < M; ++i ) {
			float sig0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				sig0 += in_sig[j] * W0[i][j];
			}
		
			sig0 = fmaxf(sig0, 0);	// ReLU
		
			sig1 += sig0 * W1[i];
		}

		// �o��
		out_sig_ptr[frame] = sig1;

		frame += frame_step;
	}
}


template <int N=6, int M=16>
int bbcu_MicroMlp_Forward
		(
			const float*	dev_in_sig,
			float*			dev_out_sig,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			const float*	dev_output_W,
			const float*	dev_output_b,
			hipStream_t	streamId = 0
		)
{
	dim3	grid(output_node_size);
	dim3	block(512, 1, 1);
	
	kernal_MicroMlp_Forward<N, M><<<grid, block, 0, streamId>>>(
			dev_in_sig,
			dev_out_sig,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);
	BB_CUDA_CHECK_LAST_ERROR();
	
	return 0;
}


int bbcu_MicroMlp6x16_Forward
		(
			float const*	dev_in_sig,
			float*	        dev_out_sig,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			const float*	dev_output_W,
			const float*	dev_output_b,
			hipStream_t	streamId
		)
{
	return bbcu_MicroMlp_Forward<6, 16>(
			dev_in_sig,
			dev_out_sig,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b,
			streamId
		);
}





// -------------------------------------------------
//  Backward
// -------------------------------------------------

// kernel
template <int N=6, int M=16, int H=16>
__global__ void kernal_MicroMlp_Backward(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db)
{
	int	id         = threadIdx.x;
	int frame_step = H;	// blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
//				 float b1;

 	__shared__   float dW0[M][N][H];
	__shared__   float db0[M][H];
	__shared__   float dW1[M][H];
	__shared__	 float db1[H];

	// �W���ǂݍ���
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}

		b0[i] = hidden_b[node * M + i];
		W1[i] = output_W[node * M + i];
	}
//	if (threadIdx.x == 0) {
//		b1 = output_b[node];
//	}
	
	// ���z������
	for ( int i = 0; i < M; ++ i ) {
		for ( int j = 0; j < N; ++j ) {
			dW0[i][j][id] = 0; // hidden_dW[(node * M + i) * N + j];
		}
	}
	for ( int i = 0; i < M; ++i ) {
		db0[i][id] = 0; // hidden_db[node * M + i];
	}
	for ( int i = 0; i < M; ++i ) {
		dW1[i][id] = 0; // output_dW[node * M + i];
	}
	db1[id] = 0; // output_db[node];

	__syncthreads();

	const float *in_sig_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_sig_ptr[i] = &in_sig_buf[frame_size * in_idx];
	}

	float	*out_err_ptr = &out_err_buf[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_sig[N];
		for ( int i = 0; i < N; ++i ) {
			in_sig[i] = in_sig_ptr[i][frame];
		}
		
		// 1�i�ڍČv�Z����2�i�ڋt�`�d
		float	err1 = out_err_ptr[frame];
		float	err0[M];
		db1[id] += err1;
		for ( int i = 0; i < M; ++i ) {
			float sig0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				sig0 += in_sig[j] * W0[i][j];
			}
		
			sig0 = fmaxf(sig0, 0);	// ReLU

			dW1[i][id] += err1 * sig0;

			if ( sig0 > 0 ) {		// ReLU
				err0[i] = err1 * W1[i];
			}
			else {
				err0[i] = 0;
			}
		}
		
		// 1�i�ڋt�`�d
		float *in_err_ptr  = &in_err_buf[frame_size * N * node];
		float	in_err[N];
		for ( int i = 0; i < N; ++i ) {
			in_err[i] = 0;	// in_err_ptr[frame_size * i + frame];
		}

		for ( int i = 0; i < M; ++i ) {
			db0[i][id] += err0[i];
			for ( int j = 0; j < N; ++j ) {
				dW0[i][j][id] += err0[i] * in_sig[j];
				in_err[j] += err0[i] * W0[i][j];
			}
		}
		
		// �덷��������
		for ( int i = 0; i < N; ++i ) {
			in_err_ptr[frame_size * i + frame] = in_err[i];
		}

		frame += frame_step;
	}
	
	__syncthreads();

	int comb = 1;
	while ( comb < H ) {
		int next = comb * 2;
		int mask = next - 1;
		if ( (threadIdx.x & mask) == 0 && id + comb < H ) {
			for ( int i = 0; i < M; ++ i ) {
				for ( int j = 0; j < N; ++j ) {
					dW0[i][j][id] += dW0[i][j][id + comb];
				}
			}
			for ( int i = 0; i < M; ++i ) {
				db0[i][id] += db0[i][id + comb];
			}
			for ( int i = 0; i < M; ++i ) {
				dW1[i][id] += dW1[i][id + comb];
			}
			db1[id] += db1[id + comb];
		}
		comb = next;
		__syncthreads();
	}

	// ���z�o��(��ŕ��񉻂���)
	if ( threadIdx.x == 0 ) {
		for ( int i = 0; i < M; ++i ) {
			for ( int j = 0; j < N; ++j ) {
				hidden_dW[(node * M + i) * N + j] = dW0[i][j][0];
			}
		}
		for ( int i = 0; i < M; ++i ) {
			hidden_db[node * M + i] = db0[i][0];
		}
		for ( int i = 0; i < M; ++i ) {
			output_dW[node * M + i] = dW1[i][0];
		}
		output_db[node] = db1[0];
	}
}


template <int N=6>
__global__ void kernal_MicroMlp_BackwardMarge(
			float*			dst_buf,
			const float*	src_buf,
			int				frame_size,
			int				node_size,
			const int*		input_index
		)
{
	int n          = blockDim.y * blockIdx.y + threadIdx.y;
	int frame      = blockDim.x * blockIdx.x + threadIdx.x;
	
	for ( int node = 0; node < node_size; ++node ) {
		int in_idx = input_index[node*N + n];
		float*		 dst_buf_ptr = &dst_buf[frame_size * in_idx];
		const float* src_buf_ptr = &src_buf[(N * node + n) * frame_size];
		
		dst_buf_ptr[frame] += src_buf_ptr[frame];

		__syncthreads();
	}
}


template <int N=6, int M=16>
int bbcu_MicroMlp_Backward(
			const float*	dev_in_sig_buf,
			float*			dev_in_err_buf,
			float*			dev_in_err_tmp,
			float*			dev_out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			float*			dev_hidden_dW,
			float*			dev_hidden_db,
			const float*	dev_output_W,
			const float*	dev_output_b,
			float*			dev_output_dW,
			float*			dev_output_db,
			hipStream_t	streamId = 0
	)
{
	{
		const int x_size = (8192 / (N*M));

		dim3	grid(output_node_size);
		dim3	block(x_size, 1, 1);
		
		kernal_MicroMlp_Backward<N, M, x_size><<<grid, block, 0, streamId>>>(
				dev_in_sig_buf,
				dev_in_err_tmp,
				dev_out_err_buf,
				frame_size,
				dev_input_index,
				dev_hidden_W,
				dev_hidden_b,
				dev_hidden_dW,
				dev_hidden_db,
				dev_output_W,
				dev_output_b,
				dev_output_dW,
				dev_output_db
			);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
	}

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_in_err_buf, 0, input_node_size * frame_size * sizeof(float)));

	    int block_x = frame_size;
	    while ( block_x > 1024 ) { block_x /= 2; }

	    dim3	grid(frame_size/block_x, N);
	    dim3	block(block_x, 1, 1);

	    kernal_MicroMlp_BackwardMarge<N><<<grid, block>>>(
			    dev_in_err_buf,
			    dev_in_err_tmp,
			    frame_size,
			    output_node_size,
			    dev_input_index
		    );

	    hipError_t cudaStatus = hipGetLastError();
	    if (cudaStatus != hipSuccess) {
		    fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		    return 1;
	    }
    }

	return 0;
}


CUBB_DLL_EXPORT int bbcu_MicroMlp6x16_Backward(
			const float*	dev_in_sig_buf,
			float*			dev_in_err_buf,
			float*			dev_in_err_tmp,
			float*			dev_out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			float*			dev_hidden_dW,
			float*			dev_hidden_db,
			const float*	dev_output_W,
			const float*	dev_output_b,
			float*			dev_output_dW,
			float*			dev_output_db,
			hipStream_t	streamId
		)
{
	return bbcu_MicroMlp_Backward<6, 16>(
			dev_in_sig_buf,
			dev_in_err_buf,
			dev_in_err_tmp,
			dev_out_err_buf,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_hidden_dW,
			dev_hidden_db,
			dev_output_W,
			dev_output_b,
			dev_output_dW,
			dev_output_db,
			streamId
		);
}



// end of file
