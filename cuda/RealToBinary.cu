#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"




//////////////////////////////
// forward
//////////////////////////////

__global__ void kernal_fp32_RealToBinary_Forward(
            const float*    x_buf,
            float*          y_buf,
            float           th_offset,
            float           th_step,
            int             modulation_size,
            int             node_size,
            int             x_frame_size,
            int             x_frame_stride,
            int             y_frame_stride
        )
{
    int x_frame = blockDim.x * blockIdx.x + threadIdx.x;
    int node    = blockDim.y * blockIdx.y + threadIdx.y;
    
    float const *x_ptr = &x_buf[node * x_frame_stride];
    float       *y_ptr = &y_buf[node * y_frame_stride];
    
    if ( x_frame < x_frame_size && node < node_size) {
        float x       = x_ptr[x_frame];
        int   y_frame = x_frame * modulation_size;
        float th      = th_offset;
        for ( int i = 0; i < modulation_size; ++i ) {
            y_ptr[y_frame + i] = (x > th) ? 1.0 : 0.0;
            th += th_step;
        }
    }
}


BBCU_DLL_EXPORT int bbcu_fp32_RealToBinary_Forward
        (
            float const     *dev_x_buf,
            float           *dev_y_buf,
            float           th_offset,
            float           th_step,
            int             modulation_size,
            int             node_size,
            int             x_frame_size,
            int             x_frame_stride,
            int             y_frame_stride,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 1024;
    unsigned int const MAX_FRAME_UNIT = 1024;
    unsigned int const MAX_NODE_UNIT  = 1024;

#if 1
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= x_frame_size )     { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size    ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= x_frame_size ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= node_size    ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid((x_frame_size + (block.x - 1)) / block.x, (node_size + (block.y - 1)) / block.y);
    
    kernal_fp32_RealToBinary_Forward<<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            th_offset,
            th_step,
            modulation_size,
            node_size,
            x_frame_size,
            x_frame_stride,
            y_frame_stride
        );
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}



// end of file
